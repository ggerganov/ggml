#include "hip/hip_runtime.h"
#include "rope.cuh"

struct rope_corr_dims {
    float v[4];
};

static __device__ float rope_yarn_ramp(const float low, const float high, const int i0) {
    const float y = (i0 / 2 - low) / max(0.001f, high - low);
    return 1.0f - min(1.0f, max(0.0f, y));
}

// YaRN algorithm based on LlamaYaRNScaledRotaryEmbedding.py from https://github.com/jquesnelle/yarn
// MIT licensed. Copyright (c) 2023 Jeffrey Quesnelle and Bowen Peng.
static __device__ void rope_yarn(
    float theta_extrap, float freq_scale, rope_corr_dims corr_dims, int64_t i0, float ext_factor, float mscale,
    float * cos_theta, float * sin_theta
) {
    // Get n-d rotational scaling corrected for extrapolation
    float theta_interp = freq_scale * theta_extrap;
    float theta = theta_interp;
    if (ext_factor != 0.0f) {
        float ramp_mix = rope_yarn_ramp(corr_dims.v[0], corr_dims.v[1], i0) * ext_factor;
        theta = theta_interp * (1 - ramp_mix) + theta_extrap * ramp_mix;

        // Get n-d magnitude scaling corrected for interpolation
        mscale *= 1.0f + 0.1f * logf(1.0f / freq_scale);
    }
    *cos_theta = cosf(theta) * mscale;
    *sin_theta = sinf(theta) * mscale;
}

// rope == RoPE == rotary positional embedding
template<typename T, bool has_pos>
static __global__ void rope(
    const T * x, T * dst, int ncols, const int32_t * pos, float freq_scale, int p_delta_rows, float freq_base,
    float ext_factor, float attn_factor, rope_corr_dims corr_dims
) {
    const int col = 2*(blockDim.y*blockIdx.y + threadIdx.y);

    if (col >= ncols) {
        return;
    }

    const int row = blockDim.x*blockIdx.x + threadIdx.x;
    const int i = row*ncols + col;
    const int i2 = row/p_delta_rows;

    const int p = has_pos ? pos[i2] : 0;
    const float theta_base = p*powf(freq_base, -float(col)/ncols);

    float cos_theta, sin_theta;
    rope_yarn(theta_base, freq_scale, corr_dims, col, ext_factor, attn_factor, &cos_theta, &sin_theta);

    const float x0 = x[i + 0];
    const float x1 = x[i + 1];

    dst[i + 0] = x0*cos_theta - x1*sin_theta;
    dst[i + 1] = x0*sin_theta + x1*cos_theta;
}

template<typename T, bool has_pos, bool has_freq_facs>
static __global__ void rope_neox(
    const T * x, T * dst, int ncols, int n_dims, const int32_t * pos, float freq_scale, int p_delta_rows,
    float ext_factor, float attn_factor, rope_corr_dims corr_dims, float theta_scale, float inv_ndims, const float * freq_factors
) {
    const int col = 2*(blockDim.y*blockIdx.y + threadIdx.y);

    if (col >= ncols) {
        return;
    }

    const int row = blockDim.x*blockIdx.x + threadIdx.x;
    const int ib = col / n_dims;
    const int ic = col % n_dims;

    if (ib > 0) {
        const int i = row*ncols + ib*n_dims + ic;

        dst[i + 0] = x[i + 0];
        dst[i + 1] = x[i + 1];

        return;
    }

    const int i  = row*ncols + ib*n_dims + ic/2;
    const int i2 = row/p_delta_rows;

    float cur_rot = inv_ndims * ic - ib;

    const int p = has_pos ? pos[i2] : 0;
    const float freq_factor = has_freq_facs ? freq_factors[ic/2] : 1.0f;

    const float theta_base = p*freq_scale*powf(theta_scale, col/2.0f)/freq_factor;

    float cos_theta, sin_theta;
    rope_yarn(theta_base, freq_scale, corr_dims, cur_rot, ext_factor, attn_factor, &cos_theta, &sin_theta);

    const float x0 = x[i + 0];
    const float x1 = x[i + n_dims/2];

    dst[i + 0]        = x0*cos_theta - x1*sin_theta;
    dst[i + n_dims/2] = x0*sin_theta + x1*cos_theta;
}

static __global__ void rope_glm_f32(
    const float * x, float * dst, int ncols, const int32_t * pos, float freq_scale, int p_delta_rows, float freq_base,
    int n_ctx
) {
    const int col = blockDim.x*blockIdx.x + threadIdx.x;
    const int half_n_dims = ncols/4;

    if (col >= half_n_dims) {
        return;
    }

    const int row = blockDim.y*blockIdx.y + threadIdx.y;
    const int i = row*ncols + col;
    const int i2 = row/p_delta_rows;

    const float col_theta_scale = powf(freq_base, -2.0f*col/ncols);
     // FIXME: this is likely wrong
    const int p = pos != nullptr ? pos[i2] : 0;

    const float theta = min(p, n_ctx - 2)*freq_scale*col_theta_scale;
    const float sin_theta = sinf(theta);
    const float cos_theta = cosf(theta);

    const float x0 = x[i + 0];
    const float x1 = x[i + half_n_dims];

    dst[i + 0]           = x0*cos_theta - x1*sin_theta;
    dst[i + half_n_dims] = x0*sin_theta + x1*cos_theta;

    const float block_theta = ((float)max(p - n_ctx - 2, 0))*col_theta_scale;
    const float sin_block_theta = sinf(block_theta);
    const float cos_block_theta = cosf(block_theta);

    const float x2 = x[i + half_n_dims * 2];
    const float x3 = x[i + half_n_dims * 3];

    dst[i + half_n_dims * 2] = x2*cos_block_theta - x3*sin_block_theta;
    dst[i + half_n_dims * 3] = x2*sin_block_theta + x3*cos_block_theta;
}


template<typename T>
static void rope_cuda(
    const T * x, T * dst, int ncols, int nrows, const int32_t * pos, float freq_scale, int p_delta_rows,
    float freq_base, float ext_factor, float attn_factor, rope_corr_dims corr_dims, hipStream_t stream
) {
    GGML_ASSERT(ncols % 2 == 0);
    const dim3 block_dims(1, CUDA_ROPE_BLOCK_SIZE, 1);
    const int num_blocks_x = (ncols + 2*CUDA_ROPE_BLOCK_SIZE - 1) / (2*CUDA_ROPE_BLOCK_SIZE);
    const dim3 block_nums(nrows, num_blocks_x, 1);
    if (pos == nullptr) {
        rope<T, false><<<block_nums, block_dims, 0, stream>>>(
            x, dst, ncols, pos, freq_scale, p_delta_rows, freq_base, ext_factor, attn_factor, corr_dims
        );
    } else {
        rope<T, true><<<block_nums, block_dims, 0, stream>>>(
            x, dst, ncols, pos, freq_scale, p_delta_rows, freq_base, ext_factor, attn_factor, corr_dims
        );
    }
}

template<typename T>
static void rope_neox_cuda(
    const T * x, T * dst, int ncols, int n_dims, int nrows, const int32_t * pos, float freq_scale, int p_delta_rows,
    float freq_base, float ext_factor, float attn_factor, rope_corr_dims corr_dims, const float * freq_factors, hipStream_t stream
) {
    GGML_ASSERT(ncols % 2 == 0);
    const dim3 block_dims(1, CUDA_ROPE_BLOCK_SIZE, 1);
    const int num_blocks_x = (ncols + 2*CUDA_ROPE_BLOCK_SIZE - 1) / (2*CUDA_ROPE_BLOCK_SIZE);
    const dim3 block_nums(nrows, num_blocks_x, 1);

    const float theta_scale = powf(freq_base, -2.0f/n_dims);
    const float inv_ndims = -1.0f / n_dims;

    if (pos == nullptr) {
        if (freq_factors == nullptr) {
            rope_neox<T, false, false><<<block_nums, block_dims, 0, stream>>>(
                x, dst, ncols, n_dims, pos, freq_scale, p_delta_rows, ext_factor, attn_factor, corr_dims,
                theta_scale, inv_ndims, freq_factors
                );
        } else {
            rope_neox<T, false, true><<<block_nums, block_dims, 0, stream>>>(
                x, dst, ncols, n_dims, pos, freq_scale, p_delta_rows, ext_factor, attn_factor, corr_dims,
                theta_scale, inv_ndims, freq_factors
                );
        }
    } else {
        if (freq_factors == nullptr) {
            rope_neox<T, true, false><<<block_nums, block_dims, 0, stream>>>(
                x, dst, ncols, n_dims, pos, freq_scale, p_delta_rows, ext_factor, attn_factor, corr_dims,
                theta_scale, inv_ndims, freq_factors
                );
        } else {
            rope_neox<T, true, true><<<block_nums, block_dims, 0, stream>>>(
                x, dst, ncols, n_dims, pos, freq_scale, p_delta_rows, ext_factor, attn_factor, corr_dims,
                theta_scale, inv_ndims, freq_factors
                );
        }
    }
}

static void rope_glm_f32_cuda(
    const float * x, float * dst, int ncols, int nrows, const int32_t * pos, float freq_scale, int p_delta_rows,
    float freq_base, int n_ctx, hipStream_t stream
) {
    GGML_ASSERT(ncols % 4 == 0);
    const dim3 block_dims(CUDA_ROPE_BLOCK_SIZE/4, 1, 1);
    const int num_blocks_x = (ncols + CUDA_ROPE_BLOCK_SIZE - 1) / CUDA_ROPE_BLOCK_SIZE;
    const dim3 block_nums(num_blocks_x, nrows, 1);
    rope_glm_f32<<<block_nums, block_dims, 0, stream>>>(x, dst, ncols, pos, freq_scale, p_delta_rows, freq_base, n_ctx);
}

static void rope_cuda_f16(
    const half * x, half * dst, int ncols, int nrows, const int32_t * pos, float freq_scale, int p_delta_rows,
    float freq_base, float ext_factor, float attn_factor, rope_corr_dims corr_dims, hipStream_t stream) {

    rope_cuda<half>(x, dst, ncols, nrows, pos, freq_scale, p_delta_rows, freq_base, ext_factor, attn_factor, corr_dims, stream);
}

static void rope_cuda_f32(
    const float * x, float * dst, int ncols, int nrows, const int32_t * pos, float freq_scale, int p_delta_rows,
    float freq_base, float ext_factor, float attn_factor, rope_corr_dims corr_dims, hipStream_t stream) {

    rope_cuda<float>(x, dst, ncols, nrows, pos, freq_scale, p_delta_rows, freq_base, ext_factor, attn_factor, corr_dims, stream);
}

static void rope_neox_cuda_f16(
    const half * x, half * dst, int ncols, int n_dims, int nrows, const int32_t * pos, float freq_scale, int p_delta_rows,
    float freq_base, float ext_factor, float attn_factor, rope_corr_dims corr_dims, const float * freq_factors, hipStream_t stream) {

    rope_neox_cuda<half>(x, dst, ncols, n_dims, nrows, pos, freq_scale, p_delta_rows, freq_base, ext_factor, attn_factor, corr_dims, freq_factors, stream);
}

static void rope_neox_cuda_f32(
    const float * x, float * dst, int ncols, int n_dims, int nrows, const int32_t * pos, float freq_scale, int p_delta_rows,
    float freq_base, float ext_factor, float attn_factor, rope_corr_dims corr_dims, const float * freq_factors, hipStream_t stream
) {

    rope_neox_cuda<float>(x, dst, ncols, n_dims, nrows, pos, freq_scale, p_delta_rows, freq_base, ext_factor, attn_factor, corr_dims, freq_factors, stream);
}

void ggml_cuda_op_rope(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const ggml_tensor * src1 = dst->src[1];
    const ggml_tensor * src2 = dst->src[2];

    const float * src0_d = (const float *)src0->data;
    const float * src1_d = (const float *)src1->data;

    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    const int64_t ne00 = src0->ne[0];
    const int64_t ne01 = src0->ne[1];
    const int64_t nrows = ggml_nrows(src0);

    //const int n_past      = ((int32_t *) dst->op_params)[0];
    const int n_dims      = ((int32_t *) dst->op_params)[1];
    const int mode        = ((int32_t *) dst->op_params)[2];
    const int n_ctx       = ((int32_t *) dst->op_params)[3];
    const int n_orig_ctx  = ((int32_t *) dst->op_params)[4];

    // RoPE alteration for extended context
    float freq_base, freq_scale, ext_factor, attn_factor, beta_fast, beta_slow;
    memcpy(&freq_base,   (int32_t *) dst->op_params +  5, sizeof(float));
    memcpy(&freq_scale,  (int32_t *) dst->op_params +  6, sizeof(float));
    memcpy(&ext_factor,  (int32_t *) dst->op_params +  7, sizeof(float));
    memcpy(&attn_factor, (int32_t *) dst->op_params +  8, sizeof(float));
    memcpy(&beta_fast,   (int32_t *) dst->op_params +  9, sizeof(float));
    memcpy(&beta_slow,   (int32_t *) dst->op_params + 10, sizeof(float));

    const float * freq_factors = nullptr;
    const int32_t * pos = nullptr;

    const bool is_neox = mode & 2;
    const bool is_glm  = mode & 4;

    pos = (const int32_t *) src1_d;

    if (is_neox) {
        if (src2 != nullptr) {
            freq_factors = (const float *) src2->data;
        }
    } else {
        GGML_ASSERT(src2 == nullptr && "TODO: freq_factors not implemented for !is_neox");
    }

    rope_corr_dims corr_dims;
    ggml_rope_yarn_corr_dims(n_dims, n_orig_ctx, freq_base, beta_fast, beta_slow, corr_dims.v);

    // compute
    if (is_glm) {
        GGML_ASSERT(false);
        rope_glm_f32_cuda(src0_d, dst_d, ne00, nrows, pos, freq_scale, ne01, freq_base, n_ctx, stream);
    } else if (is_neox) {
        if (src0->type == GGML_TYPE_F32) {
            rope_neox_cuda_f32(
                (const float *)src0_d, (float *)dst_d, ne00, n_dims, nrows, pos, freq_scale, ne01, freq_base, ext_factor,
                attn_factor, corr_dims, freq_factors, stream
            );
        } else if (src0->type == GGML_TYPE_F16) {
            rope_neox_cuda_f16(
                (const half *)src0_d, (half *)dst_d, ne00, n_dims, nrows, pos, freq_scale, ne01, freq_base, ext_factor,
                attn_factor, corr_dims, freq_factors, stream
            );
        } else {
            GGML_ASSERT(false);
        }
    } else {
        if (src0->type == GGML_TYPE_F32) {
            rope_cuda_f32(
                (const float *)src0_d, (float *)dst_d, ne00, nrows, pos, freq_scale, ne01, freq_base, ext_factor,
                attn_factor, corr_dims, stream
            );
        } else if (src0->type == GGML_TYPE_F16) {
            rope_cuda_f16(
                (const half *)src0_d, (half *)dst_d, ne00, nrows, pos, freq_scale, ne01, freq_base, ext_factor,
                attn_factor, corr_dims, stream
            );
        } else {
            GGML_ASSERT(false);
        }
    }
}

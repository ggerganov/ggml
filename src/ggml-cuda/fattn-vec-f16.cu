#include "hip/hip_runtime.h"
#include "common.cuh"
#include "fattn-common.cuh"
#include "fattn-vec-f16.cuh"

template<int D, int ncols, int parallel_blocks> // D == head size
#if !(defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__))
__launch_bounds__(D, 1)
#endif // !(defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__))
static __global__ void flash_attn_vec_ext_f16(
        const char * __restrict__ Q,
        const char * __restrict__ K,
        const char * __restrict__ V,
        const char * __restrict__ mask,
        float      * __restrict__ dst,
        float2     * __restrict__ dst_meta,
        const float scale,
        const float max_bias,
        const float m0,
        const float m1,
        const uint32_t n_head_log2,
        const int ne00,
        const int ne01,
        const int ne02,
        const int ne03,
        const int ne10,
        const int ne11,
        const int ne12,
        const int ne13,
        const int ne31,
        const int nb31,
        const int nb01,
        const int nb02,
        const int nb03,
        const int nb11,
        const int nb12,
        const int nb13,
        const int ne0,
        const int ne1,
        const int ne2,
        const int ne3) {
#if FP16_AVAILABLE
    //In this kernel Q, K, V are matrices while i, j, k are matrix indices.

    const int ic0 = (blockIdx.x / parallel_blocks) * ncols; // Index of the Q/QKV column to work on.
    const int ip  =  blockIdx.x % parallel_blocks; // Index in group of blocks running for the same column in parallel.

    const int gqa_ratio = ne02 / ne12; // With grouped query attention there are > 1 Q matrices per K, V matrix.
    const float2 * Q_f2  = (const float2 *) (Q    + nb02* blockIdx.y              + nb01*ic0);
    const half2  * K_h2  = (const half2  *) (K    + nb12*(blockIdx.y / gqa_ratio));
    const half   * V_h   = (const half   *) (V    + nb12*(blockIdx.y / gqa_ratio)); // K and V have same shape
    const half   * maskh = (const half   *)  mask + ne11*ic0;

    const int stride_KV  = nb11 / sizeof(half);
    const int stride_KV2 = nb11 / sizeof(half2);

    half  slopeh = __float2half(1.0f);

    // ALiBi
    if (max_bias > 0.0f) {
        const uint32_t h = blockIdx.y;

        const float base = h < n_head_log2 ? m0 : m1;
        const int   exph = h < n_head_log2 ? h + 1 : 2*(h - n_head_log2) + 1;

        slopeh = __float2half(powf(base, exph));
    }

    static_assert(D % (2*WARP_SIZE) == 0, "D not divisible by 2*WARP_SIZE == 64.");
    constexpr int nwarps = D / WARP_SIZE;
    const int tid = WARP_SIZE*threadIdx.y + threadIdx.x;
    __builtin_assume(tid < D);

    __shared__ half KQ[ncols*D];
#pragma unroll
    for (int j = 0; j < ncols; ++j) {
        KQ[j*D + tid] = -HALF_MAX_HALF;
    }
    half2 * KQ2 = (half2 *) KQ;

    half kqmax[ncols];
#pragma unroll
    for (int j = 0; j < ncols; ++j) {
        kqmax[j] = -HALF_MAX_HALF;
    }
    half kqsum[ncols] = {0.0f};

    __shared__ half kqmax_shared[ncols][WARP_SIZE];
    __shared__ half kqsum_shared[ncols][WARP_SIZE];
#pragma unroll
    for (int j = 0; j < ncols; ++j) {
        if (threadIdx.y == 0) {
            kqmax_shared[j][threadIdx.x] = -HALF_MAX_HALF;
            kqsum_shared[j][threadIdx.x] = 0.0f;
        }
    }
    __syncthreads();

    // Convert Q to half2 and store in registers:
    half2 Q_h2[ncols][D/(2*WARP_SIZE)];
#pragma unroll
    for (int j = 0; j < ncols; ++j) {
#pragma unroll
        for (int i0 = 0; i0 < D/2; i0 += WARP_SIZE) {
            const int i = i0 + threadIdx.x;

            const float2 tmp = Q_f2[j*(nb01/sizeof(float2)) + i];
            Q_h2[j][i0/WARP_SIZE] = make_half2(scale, scale) * make_half2(tmp.x, tmp.y);
        }
    }

    half2 VKQ[ncols] = {{0.0f, 0.0f}};

    const int k_start = parallel_blocks == 1 ? 0 : ip*D;
    for (int k_VKQ_0 = k_start; k_VKQ_0 < ne11; k_VKQ_0 += parallel_blocks*D) {
        // Calculate KQ tile and keep track of new maximum KQ values:

        // For unknown reasons using a half array of size 1 for kqmax_new causes a performance regression,
        // see https://github.com/ggerganov/llama.cpp/pull/7061 .
        // Therefore this variable is defined twice but only used once (so that the compiler can optimize out the unused variable).
        half kqmax_new = kqmax[0];
        half kqmax_new_arr[ncols];
#pragma unroll
        for (int j = 0; j < ncols; ++j) {
            kqmax_new_arr[j] = kqmax[j];
        }

#pragma unroll
        for (int i_KQ_0 = 0; i_KQ_0 < D; i_KQ_0 += nwarps) {
            const int i_KQ = i_KQ_0 + threadIdx.y;

            if ((i_KQ_0 + nwarps > D && i_KQ >= D) || (FATTN_KQ_STRIDE % D != 0 && k_VKQ_0 + i_KQ >= ne11)) {
                break;
            }

            half2 sum2[ncols] = {{0.0f, 0.0f}};
#pragma unroll
            for (int k_KQ_0 = 0; k_KQ_0 < D/2; k_KQ_0 += WARP_SIZE) {
                const int k_KQ = k_KQ_0 + threadIdx.x;

                const half2 K_ik = K_h2[(k_VKQ_0 + i_KQ)*stride_KV2 + k_KQ];
#pragma unroll
                for (int j = 0; j < ncols; ++j) {
                    sum2[j] += K_ik * Q_h2[j][k_KQ_0/WARP_SIZE];
                }
            }

#pragma unroll
            for (int j = 0; j < ncols; ++j) {
                sum2[j] = warp_reduce_sum(sum2[j]);
                half sum = __low2half(sum2[j]) + __high2half(sum2[j]);
                sum += mask ? slopeh*maskh[j*ne11 + k_VKQ_0 + i_KQ] : __float2half(0.0f);

                if (ncols == 1) {
                    kqmax_new        = ggml_cuda_hmax(kqmax_new,        sum);
                } else {
                    kqmax_new_arr[j] = ggml_cuda_hmax(kqmax_new_arr[j], sum);
                }

                if (threadIdx.x == 0) {
                    KQ[j*D + i_KQ] = sum;
                }
            }
        }

#pragma unroll
        for (int j = 0; j < ncols; ++j) {
            half kqmax_new_j = ncols == 1 ? kqmax_new : kqmax_new_arr[j];

            kqmax_new_j = warp_reduce_max(kqmax_new_j);
            if (threadIdx.x == 0) {
                kqmax_shared[j][threadIdx.y] = kqmax_new_j;
            }
        }

        __syncthreads();

#pragma unroll
        for (int j = 0; j < ncols; ++j) {
            half kqmax_new_j = kqmax_shared[j][threadIdx.x];
            kqmax_new_j = warp_reduce_max(kqmax_new_j);

            const half KQ_max_scale = hexp(kqmax[j] - kqmax_new_j);
            kqmax[j] = kqmax_new_j;

            const half val = hexp(KQ[j*D + tid] - kqmax[j]);
            kqsum[j] = kqsum[j]*KQ_max_scale + val;
            KQ[j*D + tid] = val;

            VKQ[j] *= __half2half2(KQ_max_scale);
        }

        __syncthreads();

#pragma unroll
        for (int k0 = 0; k0 < D; k0 += 2) {
            if (FATTN_KQ_STRIDE % D != 0 && k_VKQ_0 + k0 >= ne11) {
                break;
            }

            half2 V_k;
            reinterpret_cast<half&>(V_k.x) = V_h[(k_VKQ_0 + k0 + 0)*stride_KV + tid];
            reinterpret_cast<half&>(V_k.y) = V_h[(k_VKQ_0 + k0 + 1)*stride_KV + tid];
#pragma unroll
            for (int j = 0; j < ncols; ++j) {
                VKQ[j] += V_k*KQ2[j*(D/2) + k0/2];
            }
        }

        __syncthreads();
    }

#pragma unroll
    for (int j = 0; j < ncols; ++j) {
        kqsum[j] = warp_reduce_sum(kqsum[j]);
        if (threadIdx.x == 0) {
            kqsum_shared[j][threadIdx.y] = kqsum[j];
        }
    }

    __syncthreads();

#pragma unroll
    for (int j_VKQ = 0; j_VKQ < ncols; ++j_VKQ) {
        kqsum[j_VKQ] = kqsum_shared[j_VKQ][threadIdx.x];
        kqsum[j_VKQ] = warp_reduce_sum(kqsum[j_VKQ]);

        half dst_val = (__low2half(VKQ[j_VKQ]) + __high2half(VKQ[j_VKQ]));
        if (parallel_blocks == 1) {
            dst_val /= kqsum[j_VKQ];
        }
        const int j_dst = (ic0 + j_VKQ)*parallel_blocks + ip;
        dst[j_dst*D*gridDim.y + D*blockIdx.y + tid] = dst_val;
    }

    if (parallel_blocks != 1 && threadIdx.x < ncols) {
        dst_meta[(ic0 + threadIdx.x)*gridDim.y*parallel_blocks + blockIdx.y*parallel_blocks + ip] = make_float2(kqmax[threadIdx.x], kqsum[threadIdx.x]);
    }
#else
   NO_DEVICE_CODE;
#endif // FP16_AVAILABLE
}

template <int D, int cols_per_block, int parallel_blocks> void launch_fattn_vec_f16(
        const ggml_tensor * Q, const ggml_tensor * K, const ggml_tensor * V, ggml_tensor * KQV, const ggml_tensor * mask,
        ggml_cuda_pool & pool, hipStream_t main_stream
) {
    ggml_cuda_pool_alloc<float>  dst_tmp(pool);
    ggml_cuda_pool_alloc<float2> dst_tmp_meta(pool);

    if (parallel_blocks > 1) {
        dst_tmp.alloc(parallel_blocks*ggml_nelements(KQV));
        dst_tmp_meta.alloc(parallel_blocks*ggml_nrows(KQV));
    }

    constexpr int  nwarps = (D + WARP_SIZE - 1) / WARP_SIZE;
    const     dim3 block_dim(WARP_SIZE, nwarps, 1);
    const     dim3 blocks_num(parallel_blocks*((Q->ne[1] + cols_per_block - 1) / cols_per_block), Q->ne[2], Q->ne[3]);
    const     int  shmem = 0;

    float scale    = 1.0f;
    float max_bias = 0.0f;

    memcpy(&scale,    (float *) KQV->op_params + 0, sizeof(float));
    memcpy(&max_bias, (float *) KQV->op_params + 1, sizeof(float));

    const uint32_t n_head      = Q->ne[2];
    const uint32_t n_head_log2 = 1u << (uint32_t) floorf(log2f((float) n_head));

    const float m0 = powf(2.0f, -(max_bias       ) / n_head_log2);
    const float m1 = powf(2.0f, -(max_bias / 2.0f) / n_head_log2);

    flash_attn_vec_ext_f16<D, cols_per_block, parallel_blocks>
        <<<blocks_num, block_dim, shmem, main_stream>>> (
                (const char *) Q->data,
                (const char *) K->data,
                (const char *) V->data,
                mask ? ((const char *) mask->data) : nullptr,
                parallel_blocks == 1 ? (float *) KQV->data : dst_tmp.ptr, dst_tmp_meta.ptr,
                scale, max_bias, m0, m1, n_head_log2,
                Q->ne[0], Q->ne[1], Q->ne[2], Q->ne[3],
                K->ne[0], K->ne[1], K->ne[2], K->ne[3],
                mask ? mask->ne[1] : 0, mask ?  mask->nb[1] : 0,
                Q->nb[1], Q->nb[2], Q->nb[3],
                K->nb[1], K->nb[2], K->nb[3],
                KQV->ne[0], KQV->ne[1], KQV->ne[2], KQV->ne[3]
                );
    CUDA_CHECK(hipGetLastError());

    if (parallel_blocks == 1) {
        return;
    }

    const dim3 block_dim_combine(D, 1, 1);
    const dim3 blocks_num_combine(Q->ne[1], blocks_num.y, blocks_num.z);
    const int  shmem_combine = 0;

    flash_attn_combine_results<D, parallel_blocks>
        <<<blocks_num_combine, block_dim_combine, shmem_combine, main_stream>>>
        (dst_tmp.ptr, dst_tmp_meta.ptr, (float *) KQV->data);
    CUDA_CHECK(hipGetLastError());
}

void ggml_cuda_flash_attn_ext_vec_f16(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * Q = dst->src[0];
    const ggml_tensor * K = dst->src[1];
    const ggml_tensor * V = dst->src[2];

    const ggml_tensor * mask = dst->src[3];

    ggml_tensor * KQV = dst;

    const int32_t precision = KQV->op_params[2];
    GGML_ASSERT(precision == GGML_PREC_DEFAULT);

    constexpr int cols_per_block = 1;
    constexpr int parallel_blocks = 4;
    switch (Q->ne[0]) {
        case 64:
            launch_fattn_vec_f16< 64, cols_per_block, parallel_blocks>(Q, K, V, KQV, mask, ctx.pool(), ctx.stream());
            break;
        case 128:
            launch_fattn_vec_f16<128, cols_per_block, parallel_blocks>(Q, K, V, KQV, mask, ctx.pool(), ctx.stream());
            break;
        case 256:
            launch_fattn_vec_f16<256, cols_per_block, parallel_blocks>(Q, K, V, KQV, mask, ctx.pool(), ctx.stream());
            break;
        default:
            GGML_ASSERT(false);
            break;
    }
}

void ggml_cuda_flash_attn_ext_vec_f16_no_mma(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * Q = dst->src[0];
    const ggml_tensor * K = dst->src[1];
    const ggml_tensor * V = dst->src[2];

    const ggml_tensor * mask = dst->src[3];

    ggml_tensor * KQV = dst;

    const int32_t precision = KQV->op_params[2];
    GGML_ASSERT(precision == GGML_PREC_DEFAULT);
    GGML_ASSERT(Q->ne[0] == 64 || Q->ne[0] == 128 && "FlashAttention without tensor cores only supports head sizes 64 and 128.");

    if (Q->ne[1] == 1) {
        constexpr int cols_per_block = 1;
        constexpr int parallel_blocks = 4;
        switch (Q->ne[0]) {
            case 64:
                launch_fattn_vec_f16< 64, cols_per_block, parallel_blocks>(Q, K, V, KQV, mask, ctx.pool(), ctx.stream());
                break;
            case 128:
                launch_fattn_vec_f16<128, cols_per_block, parallel_blocks>(Q, K, V, KQV, mask, ctx.pool(), ctx.stream());
                break;
            default:
                GGML_ASSERT(false);
                break;
        }
        return;
    }

    if (Q->ne[1] == 2) {
        constexpr int cols_per_block = 2;
        constexpr int parallel_blocks = 4;
        switch (Q->ne[0]) {
            case 64:
                launch_fattn_vec_f16< 64, cols_per_block, parallel_blocks>(Q, K, V, KQV, mask, ctx.pool(), ctx.stream());
                break;
            case 128:
                launch_fattn_vec_f16<128, cols_per_block, parallel_blocks>(Q, K, V, KQV, mask, ctx.pool(), ctx.stream());
                break;
            default:
                GGML_ASSERT(false);
                break;
        }
        return;
    }

    if (Q->ne[1] <= 4) {
        constexpr int cols_per_block = 4;
        constexpr int parallel_blocks = 4;
        switch (Q->ne[0]) {
            case 64:
                launch_fattn_vec_f16< 64, cols_per_block, parallel_blocks>(Q, K, V, KQV, mask, ctx.pool(), ctx.stream());
                break;
            case 128:
                launch_fattn_vec_f16<128, cols_per_block, parallel_blocks>(Q, K, V, KQV, mask, ctx.pool(), ctx.stream());
                break;
            default:
                GGML_ASSERT(false);
                break;
        }
        return;
    }

    if (Q->ne[1] <= 8) {
        constexpr int cols_per_block = 8;
        constexpr int parallel_blocks = 4;
        switch (Q->ne[0]) {
            case 64:
                launch_fattn_vec_f16< 64, cols_per_block, parallel_blocks>(Q, K, V, KQV, mask, ctx.pool(), ctx.stream());
                break;
            case 128:
                launch_fattn_vec_f16<128, cols_per_block, parallel_blocks>(Q, K, V, KQV, mask, ctx.pool(), ctx.stream());
                break;
            default:
                GGML_ASSERT(false);
                break;
        }
        return;
    }

    constexpr int cols_per_block = 8;
    constexpr int parallel_blocks = 1;
    switch (Q->ne[0]) {
        case 64:
            launch_fattn_vec_f16< 64, cols_per_block, parallel_blocks>(Q, K, V, KQV, mask, ctx.pool(), ctx.stream());
            break;
        case 128:
            launch_fattn_vec_f16<128, cols_per_block, parallel_blocks>(Q, K, V, KQV, mask, ctx.pool(), ctx.stream());
            break;
        default:
            GGML_ASSERT(false);
            break;
    }
}

#include "hip/hip_runtime.h"
#include "common.cuh"
#include "fattn-common.cuh"
#include "fattn-vec-f16.cuh"

template<int D, int ncols, int parallel_blocks> // D == head size
#if !(defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__))
__launch_bounds__(D, 1)
#endif // !(defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__))
static __global__ void flash_attn_vec_ext_f16(
        const char * __restrict__ Q,
        const char * __restrict__ K,
        const char * __restrict__ V,
        const char * __restrict__ mask,
        float      * __restrict__ dst,
        float2     * __restrict__ dst_meta,
        const float scale,
        const float max_bias,
        const float m0,
        const float m1,
        const uint32_t n_head_log2,
        const int ne00,
        const int ne01,
        const int ne02,
        const int ne03,
        const int ne10,
        const int ne11,
        const int ne12,
        const int ne13,
        const int ne31,
        const int nb31,
        const int nb01,
        const int nb02,
        const int nb03,
        const int nb11,
        const int nb12,
        const int nb13,
        const int ne0,
        const int ne1,
        const int ne2,
        const int ne3) {
#if FP16_AVAILABLE
    //In this kernel Q, K, V are matrices while i, j, k are matrix indices.

    const int ic0 = (blockIdx.x / parallel_blocks) * ncols; // Index of the Q/QKV column to work on.
    const int ip  =  blockIdx.x % parallel_blocks; // Index in group of blocks running for the same column in parallel.

    const int gqa_ratio = ne02 / ne12; // With grouped query attention there are > 1 Q matrices per K, V matrix.
    const float2 * Q_f2  = (const float2 *) (Q    + nb02* blockIdx.y              + nb01*ic0);
    const half2  * K_h2  = (const half2  *) (K    + nb12*(blockIdx.y / gqa_ratio));
    const half   * V_h   = (const half   *) (V    + nb12*(blockIdx.y / gqa_ratio)); // K and V have same shape
    const half   * maskh = (const half   *)  mask + ne11*ic0;

    const int stride_KV  = nb11 / sizeof(half);
    const int stride_KV2 = nb11 / sizeof(half2);

    const float slopef = get_alibi_slope(max_bias, blockIdx.y, n_head_log2, m0, m1);
    const half  slopeh = __float2half(slopef);

    static_assert(D % (2*WARP_SIZE) == 0, "D not divisible by 2*WARP_SIZE == 64.");
    constexpr int nwarps = D / WARP_SIZE;
    const int tid = WARP_SIZE*threadIdx.y + threadIdx.x;
    __builtin_assume(tid < D);

    __shared__ half KQ[ncols*D];
#pragma unroll
    for (int j = 0; j < ncols; ++j) {
        KQ[j*D + tid] = -HALF_MAX_HALF;
    }
    half2 * KQ2 = (half2 *) KQ;

    half kqmax[ncols];
#pragma unroll
    for (int j = 0; j < ncols; ++j) {
        kqmax[j] = -HALF_MAX_HALF;
    }
    half kqsum[ncols] = {0.0f};

    __shared__ half kqmax_shared[ncols][WARP_SIZE];
    __shared__ half kqsum_shared[ncols][WARP_SIZE];
#pragma unroll
    for (int j = 0; j < ncols; ++j) {
        if (threadIdx.y == 0) {
            kqmax_shared[j][threadIdx.x] = -HALF_MAX_HALF;
            kqsum_shared[j][threadIdx.x] = 0.0f;
        }
    }
    __syncthreads();

    // Convert Q to half2 and store in registers:
    half2 Q_h2[ncols][D/(2*WARP_SIZE)];
#pragma unroll
    for (int j = 0; j < ncols; ++j) {
#pragma unroll
        for (int i0 = 0; i0 < D/2; i0 += WARP_SIZE) {
            const int i = i0 + threadIdx.x;

            const float2 tmp = ncols <= 2 || ic0 + j < ne01 ? Q_f2[j*(nb01/sizeof(float2)) + i] : make_float2(0.0f, 0.0f);
            Q_h2[j][i0/WARP_SIZE] = make_half2(scale, scale) * make_half2(tmp.x, tmp.y);
        }
    }

    half2 VKQ[ncols] = {{0.0f, 0.0f}};

    const int k_start = parallel_blocks == 1 ? 0 : ip*D;
    for (int k_VKQ_0 = k_start; k_VKQ_0 < ne11; k_VKQ_0 += parallel_blocks*D) {
        // Calculate KQ tile and keep track of new maximum KQ values:

        // For unknown reasons using a half array of size 1 for kqmax_new causes a performance regression,
        // see https://github.com/ggerganov/llama.cpp/pull/7061 .
        // Therefore this variable is defined twice but only used once (so that the compiler can optimize out the unused variable).
        half kqmax_new = kqmax[0];
        half kqmax_new_arr[ncols];
#pragma unroll
        for (int j = 0; j < ncols; ++j) {
            kqmax_new_arr[j] = kqmax[j];
        }

#pragma unroll
        for (int i_KQ_0 = 0; i_KQ_0 < D; i_KQ_0 += nwarps) {
            const int i_KQ = i_KQ_0 + threadIdx.y;

            if ((i_KQ_0 + nwarps > D && i_KQ >= D) || (FATTN_KQ_STRIDE % D != 0 && k_VKQ_0 + i_KQ >= ne11)) {
                break;
            }

            half2 sum2[ncols] = {{0.0f, 0.0f}};
#pragma unroll
            for (int k_KQ_0 = 0; k_KQ_0 < D/2; k_KQ_0 += WARP_SIZE) {
                const int k_KQ = k_KQ_0 + threadIdx.x;

                const half2 K_ik = K_h2[(k_VKQ_0 + i_KQ)*stride_KV2 + k_KQ];
#pragma unroll
                for (int j = 0; j < ncols; ++j) {
                    sum2[j] += K_ik * Q_h2[j][k_KQ_0/WARP_SIZE];
                }
            }

#pragma unroll
            for (int j = 0; j < ncols; ++j) {
                sum2[j] = warp_reduce_sum(sum2[j]);
                half sum = __low2half(sum2[j]) + __high2half(sum2[j]);
                sum += mask ? slopeh*maskh[j*ne11 + k_VKQ_0 + i_KQ] : __float2half(0.0f);

                if (ncols == 1) {
                    kqmax_new        = ggml_cuda_hmax(kqmax_new,        sum);
                } else {
                    kqmax_new_arr[j] = ggml_cuda_hmax(kqmax_new_arr[j], sum);
                }

                if (threadIdx.x == 0) {
                    KQ[j*D + i_KQ] = sum;
                }
            }
        }

#pragma unroll
        for (int j = 0; j < ncols; ++j) {
            half kqmax_new_j = ncols == 1 ? kqmax_new : kqmax_new_arr[j];

            kqmax_new_j = warp_reduce_max(kqmax_new_j);
            if (threadIdx.x == 0) {
                kqmax_shared[j][threadIdx.y] = kqmax_new_j;
            }
        }

        __syncthreads();

#pragma unroll
        for (int j = 0; j < ncols; ++j) {
            half kqmax_new_j = kqmax_shared[j][threadIdx.x];
            kqmax_new_j = warp_reduce_max(kqmax_new_j);

            const half KQ_max_scale = hexp(kqmax[j] - kqmax_new_j);
            kqmax[j] = kqmax_new_j;

            const half val = hexp(KQ[j*D + tid] - kqmax[j]);
            kqsum[j] = kqsum[j]*KQ_max_scale + val;
            KQ[j*D + tid] = val;

            VKQ[j] *= __half2half2(KQ_max_scale);
        }

        __syncthreads();

#pragma unroll
        for (int k0 = 0; k0 < D; k0 += 2) {
            if (FATTN_KQ_STRIDE % D != 0 && k_VKQ_0 + k0 >= ne11) {
                break;
            }

            half2 V_k;
            reinterpret_cast<half&>(V_k.x) = V_h[(k_VKQ_0 + k0 + 0)*stride_KV + tid];
            reinterpret_cast<half&>(V_k.y) = V_h[(k_VKQ_0 + k0 + 1)*stride_KV + tid];
#pragma unroll
            for (int j = 0; j < ncols; ++j) {
                VKQ[j] += V_k*KQ2[j*(D/2) + k0/2];
            }
        }

        __syncthreads();
    }

#pragma unroll
    for (int j = 0; j < ncols; ++j) {
        kqsum[j] = warp_reduce_sum(kqsum[j]);
        if (threadIdx.x == 0) {
            kqsum_shared[j][threadIdx.y] = kqsum[j];
        }
    }

    __syncthreads();

#pragma unroll
    for (int j_VKQ = 0; j_VKQ < ncols; ++j_VKQ) {
        if (ncols > 2 && ic0 + j_VKQ >= ne01) {
            break;
        }

        kqsum[j_VKQ] = kqsum_shared[j_VKQ][threadIdx.x];
        kqsum[j_VKQ] = warp_reduce_sum(kqsum[j_VKQ]);

        half dst_val = (__low2half(VKQ[j_VKQ]) + __high2half(VKQ[j_VKQ]));
        if (parallel_blocks == 1) {
            dst_val /= kqsum[j_VKQ];
        }
        const int j_dst = (ic0 + j_VKQ)*parallel_blocks + ip;
        dst[j_dst*D*gridDim.y + D*blockIdx.y + tid] = dst_val;
    }

    if (parallel_blocks != 1 && tid < ncols && (ncols <= 2 || ic0 + tid < ne01)) {
        dst_meta[(ic0 + tid)*gridDim.y*parallel_blocks + blockIdx.y*parallel_blocks + ip] = make_float2(kqmax[tid], kqsum[tid]);
    }
#else
   NO_DEVICE_CODE;
#endif // FP16_AVAILABLE
}

void ggml_cuda_flash_attn_ext_vec_f16(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_tensor * KQV = dst;
    ggml_tensor * Q   = dst->src[0];

    const int32_t precision = KQV->op_params[2];
    GGML_ASSERT(precision == GGML_PREC_DEFAULT);

    constexpr int cols_per_block  = 1;
    constexpr int parallel_blocks = 4;
    switch (Q->ne[0]) {
        case  64: {
            constexpr int      D = 64;
            constexpr int nwarps = D/WARP_SIZE;
            fattn_kernel_t fattn_kernel = flash_attn_vec_ext_f16<D, cols_per_block, parallel_blocks>;
            launch_fattn<D, parallel_blocks>(ctx, dst, fattn_kernel, nwarps, cols_per_block);
        } break;
        case 128: {
            constexpr int      D = 128;
            constexpr int nwarps = D/WARP_SIZE;
            fattn_kernel_t fattn_kernel = flash_attn_vec_ext_f16<D, cols_per_block, parallel_blocks>;
            launch_fattn<D, parallel_blocks>(ctx, dst, fattn_kernel, nwarps, cols_per_block);
        } break;
        case 256: {
            constexpr int      D = 256;
            constexpr int nwarps = D/WARP_SIZE;
            fattn_kernel_t fattn_kernel = flash_attn_vec_ext_f16<D, cols_per_block, parallel_blocks>;
            launch_fattn<D, parallel_blocks>(ctx, dst, fattn_kernel, nwarps, cols_per_block);
        } break;
        default:
            GGML_ASSERT(false);
            break;
    }
}

template <int cols_per_block, int parallel_blocks>
void launch_fattn_vec_f16_64_128(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * Q = dst->src[0];
    switch (Q->ne[0]) {
        case  64: {
            constexpr int      D = 64;
            constexpr int nwarps = D/WARP_SIZE;
            fattn_kernel_t fattn_kernel = flash_attn_vec_ext_f16<D, cols_per_block, parallel_blocks>;
            launch_fattn<D, parallel_blocks>(ctx, dst, fattn_kernel, nwarps, cols_per_block);
        } break;
        case 128: {
            constexpr int      D = 128;
            constexpr int nwarps = D/WARP_SIZE;
            fattn_kernel_t fattn_kernel = flash_attn_vec_ext_f16<D, cols_per_block, parallel_blocks>;
            launch_fattn<D, parallel_blocks>(ctx, dst, fattn_kernel, nwarps, cols_per_block);
        } break;
        default: {
            GGML_ASSERT(false && "FlashAttention without tensor cores only supports head sizes 64 and 128.");
        } break;
    }
}

void ggml_cuda_flash_attn_ext_vec_f16_no_mma(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * KQV = dst;
    const ggml_tensor * Q   = dst->src[0];

    const int32_t precision = KQV->op_params[2];
    GGML_ASSERT(precision == GGML_PREC_DEFAULT);

    if (Q->ne[1] == 1) {
        ggml_cuda_flash_attn_ext_vec_f16(ctx, dst);
        return;
    }

    if (Q->ne[1] == 2) {
        constexpr int cols_per_block  = 2;
        constexpr int parallel_blocks = 4;
        launch_fattn_vec_f16_64_128<cols_per_block, parallel_blocks>(ctx, dst);
        return;
    }

    if (Q->ne[1] <= 4) {
        constexpr int cols_per_block  = 4;
        constexpr int parallel_blocks = 4;
        launch_fattn_vec_f16_64_128<cols_per_block, parallel_blocks>(ctx, dst);
        return;
    }

    if (Q->ne[1] <= 8) {
        constexpr int cols_per_block  = 8;
        constexpr int parallel_blocks = 4;
        launch_fattn_vec_f16_64_128<cols_per_block, parallel_blocks>(ctx, dst);
        return;
    }

    constexpr int cols_per_block  = 8;
    constexpr int parallel_blocks = 1;
    launch_fattn_vec_f16_64_128<cols_per_block, parallel_blocks>(ctx, dst);
}

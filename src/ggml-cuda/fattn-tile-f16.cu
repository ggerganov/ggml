#include "hip/hip_runtime.h"
#include "common.cuh"
#include "fattn-common.cuh"
#include "fattn-tile-f16.cuh"

#define FATTN_KQ_STRIDE_TILE_F16 64

template<int D, int ncols, int nwarps, int parallel_blocks> // D == head size
#if !(defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__))
__launch_bounds__(nwarps*WARP_SIZE, 1)
#endif // !(defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__))
static __global__ void flash_attn_tile_ext_f16(
        const char * __restrict__ Q,
        const char * __restrict__ K,
        const char * __restrict__ V,
        const char * __restrict__ mask,
        float      * __restrict__ dst,
        float2     * __restrict__ dst_meta,
        const float scale,
        const float max_bias,
        const float m0,
        const float m1,
        const uint32_t n_head_log2,
        const int ne00,
        const int ne01,
        const int ne02,
        const int ne03,
        const int ne10,
        const int ne11,
        const int ne12,
        const int ne13,
        const int ne31,
        const int nb31,
        const int nb01,
        const int nb02,
        const int nb03,
        const int nb11,
        const int nb12,
        const int nb13,
        const int ne0,
        const int ne1,
        const int ne2,
        const int ne3) {
#if FP16_AVAILABLE
    //In this kernel Q, K, V are matrices while i, j, k are matrix indices.

    const int ic0 = (blockIdx.x / parallel_blocks) * ncols; // Index of the Q/QKV column to work on.
    const int ip  =  blockIdx.x % parallel_blocks; // Index in group of blocks running for the same column in parallel.

    const int gqa_ratio = ne02 / ne12; // With grouped query attention there are > 1 Q matrices per K, V matrix.
    const float2 * Q_f2  = (const float2 *) (Q    + nb02* blockIdx.y              + nb01*ic0);
    const half2  * K_h2  = (const half2  *) (K    + nb12*(blockIdx.y / gqa_ratio));
    const half2  * V_h2  = (const half2  *) (V    + nb12*(blockIdx.y / gqa_ratio)); // K and V have same shape
    const half   * maskh = (const half   *)  mask + ne11*ic0;

    const int stride_KV2 = nb11 / sizeof(half2);

    half slopeh = __float2half(1.0f);

    // ALiBi
    if (max_bias > 0.0f) {
        const uint32_t h = blockIdx.y;

        const float base = h < n_head_log2 ? m0 : m1;
        const int   exph = h < n_head_log2 ? h + 1 : 2*(h - n_head_log2) + 1;

        slopeh = __float2half(powf(base, exph));
    }

    static_assert(D % (2*WARP_SIZE) == 0, "D not divisible by 2*WARP_SIZE == 64.");

    __shared__ half KQ[ncols*FATTN_KQ_STRIDE_TILE_F16];
    half2 * KQ2 = (half2 *) KQ;

    __shared__ half2 KV_tmp[FATTN_KQ_STRIDE_TILE_F16][D/2 + 1]; // Pad D to avoid memory bank conflicts.

    half kqmax[ncols/nwarps];
#pragma unroll
    for (int j0 = 0; j0 < ncols; j0 += nwarps) {
        kqmax[j0/nwarps] = -HALF_MAX_HALF;
    }
    half2 kqsum[ncols/nwarps] = {{0.0f, 0.0f}};

    half2 VKQ[ncols/nwarps][(D/2)/WARP_SIZE] = {{{0.0f, 0.0f}}};

    // Convert Q to half2 and store in registers:
    __shared__ half2 Q_h2[ncols][D/2];
#pragma unroll
    for (int j0 = 0; j0 < ncols; j0 += nwarps) {
        const int j = j0 + threadIdx.y;

#pragma unroll
        for (int i0 = 0; i0 < D/2; i0 += WARP_SIZE) {
            const int i = i0 + threadIdx.x;

            const float2 tmp = Q_f2[j*(nb01/sizeof(float2)) + i];
            Q_h2[j][i] = make_half2(scale, scale) * make_half2(tmp.x, tmp.y);
        }
    }

    __syncthreads();

    const int k_start = parallel_blocks == 1 ? 0 : ip*FATTN_KQ_STRIDE_TILE_F16;
    for (int k_VKQ_0 = k_start; k_VKQ_0 < ne11; k_VKQ_0 += parallel_blocks*FATTN_KQ_STRIDE_TILE_F16) {
        // Calculate KQ tile and keep track of new maximum KQ values:

        half kqmax_new[ncols/nwarps];
#pragma unroll
        for (int j = 0; j < ncols/nwarps; ++j) {
            kqmax_new[j] = kqmax[j];
        }

#pragma unroll
        for (int i_KQ_0 = 0; i_KQ_0 < FATTN_KQ_STRIDE_TILE_F16; i_KQ_0 += nwarps) {
            const int i_KQ = i_KQ_0 + threadIdx.y;

#pragma unroll
            for (int k_KQ_0 = 0; k_KQ_0 < D/2; k_KQ_0 += WARP_SIZE) {
                const int k_KQ = k_KQ_0 + threadIdx.x;

                KV_tmp[i_KQ][k_KQ] = K_h2[(k_VKQ_0 + i_KQ)*stride_KV2 + k_KQ];
            }
        }

        __syncthreads();

        half2 sum2[FATTN_KQ_STRIDE_TILE_F16/WARP_SIZE][ncols/nwarps] = {{{0.0f, 0.0f}}};

#pragma unroll
        for (int k_KQ = 0; k_KQ < D/2; ++k_KQ) {
            half2 K_k[FATTN_KQ_STRIDE_TILE_F16/WARP_SIZE];
            half2 Q_k[ncols/nwarps];

#pragma unroll
            for (int i_KQ_0 = 0; i_KQ_0 < FATTN_KQ_STRIDE_TILE_F16; i_KQ_0 += WARP_SIZE) {
                const int i_KQ = i_KQ_0 + threadIdx.x;

                K_k[i_KQ_0/WARP_SIZE] = KV_tmp[i_KQ][k_KQ];
            }
#pragma unroll
            for (int j_KQ_0 = 0; j_KQ_0 < ncols; j_KQ_0 += nwarps) {
                const int j_KQ = j_KQ_0 + threadIdx.y;

                Q_k[j_KQ_0/nwarps] = Q_h2[j_KQ][k_KQ];
            }

#pragma unroll
            for (int i_KQ_0 = 0; i_KQ_0 < FATTN_KQ_STRIDE_TILE_F16; i_KQ_0 += WARP_SIZE) {
#pragma unroll
                for (int j_KQ_0 = 0; j_KQ_0 < ncols; j_KQ_0 += nwarps) {
                    sum2[i_KQ_0/WARP_SIZE][j_KQ_0/nwarps] += K_k[i_KQ_0/WARP_SIZE]*Q_k[j_KQ_0/nwarps];
                }
            }
        }

#pragma unroll
        for (int i_KQ_0 = 0; i_KQ_0 < FATTN_KQ_STRIDE_TILE_F16; i_KQ_0 += WARP_SIZE) {
            const int i_KQ = i_KQ_0 + threadIdx.x;

#pragma unroll
            for (int j_KQ_0 = 0; j_KQ_0 < ncols; j_KQ_0 += nwarps) {
                const int j_KQ = j_KQ_0 + threadIdx.y;

                half sum = __low2half(sum2[i_KQ_0/WARP_SIZE][j_KQ_0/nwarps]) + __high2half(sum2[i_KQ_0/WARP_SIZE][j_KQ_0/nwarps]);
                sum += mask ? slopeh*maskh[j_KQ*ne11 + k_VKQ_0 + i_KQ] : __float2half(0.0f);

                kqmax_new[j_KQ_0/nwarps] = ggml_cuda_hmax(kqmax_new[j_KQ_0/nwarps], sum);

                KQ[j_KQ*FATTN_KQ_STRIDE_TILE_F16 + i_KQ] = sum;
            }
        }

        __syncthreads();

#pragma unroll
        for (int j0 = 0; j0 < ncols; j0 += nwarps) {
            const int j = j0 + threadIdx.y;

            kqmax_new[j0/nwarps] = warp_reduce_max(kqmax_new[j0/nwarps]);
            const half2 KQ_max_scale = __half2half2(hexp(kqmax[j0/nwarps] - kqmax_new[j0/nwarps]));
            kqmax[j0/nwarps] = kqmax_new[j0/nwarps];

#pragma unroll
            for (int i0 = 0; i0 < FATTN_KQ_STRIDE_TILE_F16/2; i0 += WARP_SIZE) {
                const int i = i0 + threadIdx.x;

                const half2 diff = KQ2[j*(FATTN_KQ_STRIDE_TILE_F16/2) + i] - __half2half2(kqmax[j0/nwarps]);
                const half2 val = h2exp(diff);
                kqsum[j0/nwarps] = kqsum[j0/nwarps]*KQ_max_scale + val;
                KQ2[j*(FATTN_KQ_STRIDE_TILE_F16/2) + i] = val;
            }

#pragma unroll
            for (int i0 = 0; i0 < D/2; i0 += WARP_SIZE) {
                VKQ[j0/nwarps][i0/WARP_SIZE] *= KQ_max_scale;
            }
        }

        __syncthreads();

#pragma unroll
        for (int k0 = 0; k0 < FATTN_KQ_STRIDE_TILE_F16; k0 += nwarps) {
            const int k = k0 + threadIdx.y;

#pragma unroll
            for (int i0 = 0; i0 < D/2; i0 += WARP_SIZE) {
                const int i = i0 + threadIdx.x;

                KV_tmp[k][i] = V_h2[(k_VKQ_0 + k)*stride_KV2 + i];
            }
        }

        __syncthreads();

#pragma unroll
        for (int k0 = 0; k0 < FATTN_KQ_STRIDE_TILE_F16; k0 += 2) {
            half2  V_k[(D/2)/WARP_SIZE][2];
            half2 KQ_k[ncols/nwarps];

#pragma unroll
            for (int i0 = 0; i0 < D/2; i0 += WARP_SIZE) {
                const int i = i0 + threadIdx.x;

                V_k[i0/WARP_SIZE][0] = KV_tmp[k0 + 0][i];
                V_k[i0/WARP_SIZE][1] = KV_tmp[k0 + 1][i];
            }
#pragma unroll
            for (int j0 = 0; j0 < ncols; j0 += nwarps) {
                const int j = j0 + threadIdx.y;

                KQ_k[j0/nwarps] = KQ2[j*(FATTN_KQ_STRIDE_TILE_F16/2) + k0/2];
            }

#pragma unroll
            for (int i0 = 0; i0 < D/2; i0 += WARP_SIZE) {
#pragma unroll
                for (int j0 = 0; j0 < ncols; j0 += nwarps) {
                    VKQ[j0/nwarps][i0/WARP_SIZE] += V_k[i0/WARP_SIZE][0]* __low2half2(KQ_k[j0/nwarps]);
                    VKQ[j0/nwarps][i0/WARP_SIZE] += V_k[i0/WARP_SIZE][1]*__high2half2(KQ_k[j0/nwarps]);
                }
            }
        }

        __syncthreads();
    }

#pragma unroll
    for (int j_VKQ_0 = 0; j_VKQ_0 < ncols; j_VKQ_0 += nwarps) {
        const int j_VKQ = j_VKQ_0 + threadIdx.y;

        half kqsum_j = __low2half(kqsum[j_VKQ_0/nwarps]) + __high2half(kqsum[j_VKQ_0/nwarps]);
        kqsum_j = warp_reduce_sum(kqsum_j);

#pragma unroll
        for (int i00 = 0; i00 < D; i00 += 2*WARP_SIZE) {
            const int i0 = i00 + 2*threadIdx.x;

            half2 dst_val = VKQ[j_VKQ_0/nwarps][i0/(2*WARP_SIZE)];
            if (parallel_blocks == 1) {
                dst_val /= __half2half2(kqsum_j);
            }
            const int j_dst = (ic0 + j_VKQ)*parallel_blocks + ip;
            dst[j_dst*D*gridDim.y + D*blockIdx.y + i0 + 0] =  __low2float(dst_val);
            dst[j_dst*D*gridDim.y + D*blockIdx.y + i0 + 1] = __high2float(dst_val);
        }

        if (parallel_blocks != 1 && threadIdx.x == 0) {
            dst_meta[(ic0 + j_VKQ)*gridDim.y*parallel_blocks + blockIdx.y*parallel_blocks + ip] = make_float2(kqmax[j_VKQ_0/nwarps], kqsum_j);
        }
    }
#else
   NO_DEVICE_CODE;
#endif // FP16_AVAILABLE
}

template <int D, int cols_per_block, int parallel_blocks> void launch_fattn_tile_f16(
        const ggml_tensor * Q, const ggml_tensor * K, const ggml_tensor * V, ggml_tensor * KQV, const ggml_tensor * mask,
        ggml_cuda_pool & pool, hipStream_t main_stream
) {
    ggml_cuda_pool_alloc<float>  dst_tmp(pool);
    ggml_cuda_pool_alloc<float2> dst_tmp_meta(pool);

    if (parallel_blocks > 1) {
        dst_tmp.alloc(parallel_blocks*ggml_nelements(KQV));
        dst_tmp_meta.alloc(parallel_blocks*ggml_nrows(KQV));
    }

    constexpr int  nwarps = 8;
    const     dim3 block_dim(WARP_SIZE, nwarps, 1);
    const     dim3 blocks_num(parallel_blocks*((Q->ne[1] + cols_per_block - 1) / cols_per_block), Q->ne[2], Q->ne[3]);
    const     int  shmem = 0;

    float scale    = 1.0f;
    float max_bias = 0.0f;

    memcpy(&scale,    (float *) KQV->op_params + 0, sizeof(float));
    memcpy(&max_bias, (float *) KQV->op_params + 1, sizeof(float));

    const uint32_t n_head      = Q->ne[2];
    const uint32_t n_head_log2 = 1u << (uint32_t) floorf(log2f((float) n_head));

    const float m0 = powf(2.0f, -(max_bias       ) / n_head_log2);
    const float m1 = powf(2.0f, -(max_bias / 2.0f) / n_head_log2);

    flash_attn_tile_ext_f16<D, cols_per_block, nwarps, parallel_blocks>
        <<<blocks_num, block_dim, shmem, main_stream>>> (
                (const char *) Q->data,
                (const char *) K->data,
                (const char *) V->data,
                mask ? ((const char *) mask->data) : nullptr,
                parallel_blocks == 1 ? (float *) KQV->data : dst_tmp.ptr, dst_tmp_meta.ptr,
                scale, max_bias, m0, m1, n_head_log2,
                Q->ne[0], Q->ne[1], Q->ne[2], Q->ne[3],
                K->ne[0], K->ne[1], K->ne[2], K->ne[3],
                mask ? mask->ne[1] : 0, mask ?  mask->nb[1] : 0,
                Q->nb[1], Q->nb[2], Q->nb[3],
                K->nb[1], K->nb[2], K->nb[3],
                KQV->ne[0], KQV->ne[1], KQV->ne[2], KQV->ne[3]
                );
    CUDA_CHECK(hipGetLastError());

    if (parallel_blocks == 1) {
        return;
    }

    const dim3 block_dim_combine(D, 1, 1);
    const dim3 blocks_num_combine(Q->ne[1], blocks_num.y, blocks_num.z);
    const int  shmem_combine = 0;

    flash_attn_combine_results<D, parallel_blocks>
        <<<blocks_num_combine, block_dim_combine, shmem_combine, main_stream>>>
        (dst_tmp.ptr, dst_tmp_meta.ptr, (float *) KQV->data);
    CUDA_CHECK(hipGetLastError());
}

void ggml_cuda_flash_attn_ext_tile_f16(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * Q = dst->src[0];
    const ggml_tensor * K = dst->src[1];
    const ggml_tensor * V = dst->src[2];

    const ggml_tensor * mask = dst->src[3];

    ggml_tensor * KQV = dst;

    const int32_t precision = KQV->op_params[2];
    GGML_ASSERT(precision == GGML_PREC_DEFAULT);
    GGML_ASSERT(Q->ne[0] == 64 || Q->ne[0] == 128 && "FlashAttention without tensor cores only supports head sizes 64 and 128.");

    if (Q->ne[1] <= 16) {
        constexpr int cols_per_block = 16;
        constexpr int parallel_blocks = 4;
        switch (Q->ne[0]) {
            case 64:
                launch_fattn_tile_f16< 64, cols_per_block, parallel_blocks>(Q, K, V, KQV, mask, ctx.pool(), ctx.stream());
                break;
            case 128:
                launch_fattn_tile_f16<128, cols_per_block, parallel_blocks>(Q, K, V, KQV, mask, ctx.pool(), ctx.stream());
                break;
            default:
                GGML_ASSERT(false);
                break;
        }
        return;
    }

    if (Q->ne[1] <= 32) {
        constexpr int cols_per_block = 32;
        constexpr int parallel_blocks = 4;
        switch (Q->ne[0]) {
            case 64:
                launch_fattn_tile_f16< 64, cols_per_block, parallel_blocks>(Q, K, V, KQV, mask, ctx.pool(), ctx.stream());
                break;
            case 128:
                launch_fattn_tile_f16<128, cols_per_block, parallel_blocks>(Q, K, V, KQV, mask, ctx.pool(), ctx.stream());
                break;
            default:
                GGML_ASSERT(false);
                break;
        }
        return;
    }

    constexpr int cols_per_block = 32;
    constexpr int parallel_blocks = 1;
    switch (Q->ne[0]) {
        case 64:
            launch_fattn_tile_f16< 64, cols_per_block, parallel_blocks>(Q, K, V, KQV, mask, ctx.pool(), ctx.stream());
            break;
        case 128:
            launch_fattn_tile_f16<128, cols_per_block, parallel_blocks>(Q, K, V, KQV, mask, ctx.pool(), ctx.stream());
            break;
        default:
            GGML_ASSERT(false);
            break;
    }
}

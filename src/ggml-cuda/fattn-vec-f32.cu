#include "hip/hip_runtime.h"
#include "common.cuh"
#include "fattn-common.cuh"
#include "fattn-vec-f32.cuh"

template<int D, int ncols, int parallel_blocks> // D == head size
#if !(defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__))
__launch_bounds__(D, 1)
#endif // !(defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__))
static __global__ void flash_attn_vec_ext_f32(
        const char * __restrict__ Q,
        const char * __restrict__ K,
        const char * __restrict__ V,
        const char * __restrict__ mask,
        float      * __restrict__ dst,
        float2     * __restrict__ dst_meta,
        const float scale,
        const float max_bias,
        const float m0,
        const float m1,
        const uint32_t n_head_log2,
        const int ne00,
        const int ne01,
        const int ne02,
        const int ne03,
        const int ne10,
        const int ne11,
        const int ne12,
        const int ne13,
        const int ne31,
        const int nb31,
        const int nb01,
        const int nb02,
        const int nb03,
        const int nb11,
        const int nb12,
        const int nb13,
        const int ne0,
        const int ne1,
        const int ne2,
        const int ne3) {
    //In this kernel Q, K, V are matrices while i, j, k are matrix indices.

    const int ic0 = (blockIdx.x / parallel_blocks) * ncols; // Index of the Q/QKV column to work on.
    const int ip  =  blockIdx.x % parallel_blocks; // Index in group of blocks running for the same column in parallel.

    const int gqa_ratio = ne02 / ne12; // With grouped query attention there are > 1 Q matrices per K, V matrix.
    const float2 * Q_f2  = (const float2 *) (Q    + nb02* blockIdx.y              + nb01*ic0);
    const half2  * K_h2  = (const half2  *) (K    + nb12*(blockIdx.y / gqa_ratio));
    const half   * V_h   = (const half   *) (V    + nb12*(blockIdx.y / gqa_ratio)); // K and V have same shape
    const half   * maskh = (const half   *)  mask + ne11*ic0;

    const int stride_KV  = nb11 / sizeof(half);
    const int stride_KV2 = nb11 / sizeof(half2);

    const float slope = get_alibi_slope(max_bias, blockIdx.y, n_head_log2, m0, m1);

    static_assert(D % (2*WARP_SIZE) == 0, "D not divisible by 2*WARP_SIZE == 64.");
    constexpr int nwarps = D / WARP_SIZE;
    const int tid = WARP_SIZE*threadIdx.y + threadIdx.x;
    __builtin_assume(tid < D);

    __shared__ float KQ[ncols*D];
#pragma unroll
    for (int j = 0; j < ncols; ++j) {
        KQ[j*D + tid] = -FLT_MAX/2.0f;
    }

    float kqmax[ncols];
#pragma unroll
    for (int j = 0; j < ncols; ++j) {
        kqmax[j] = -FLT_MAX/2.0f;
    }
    float kqsum[ncols] = {0.0f};

    __shared__ float kqmax_shared[ncols][WARP_SIZE];
    __shared__ float kqsum_shared[ncols][WARP_SIZE];
#pragma unroll
    for (int j = 0; j < ncols; ++j) {
        if (threadIdx.y == 0) {
            kqmax_shared[j][threadIdx.x] = -FLT_MAX/2.0f;
            kqsum_shared[j][threadIdx.x] = 0.0f;
        }
    }
    __syncthreads();

    // Convert Q to half2 and store in registers:
    float2 Q_h2[ncols][D/(2*WARP_SIZE)];
#pragma unroll
    for (int j = 0; j < ncols; ++j) {
#pragma unroll
        for (int i0 = 0; i0 < D/2; i0 += WARP_SIZE) {
            const int i = i0 + threadIdx.x;

            Q_h2[j][i0/WARP_SIZE]    = ncols <= 2 || ic0 + j ? Q_f2[j*(nb01/sizeof(float2)) + i] : make_float2(0.0f, 0.0f);
            Q_h2[j][i0/WARP_SIZE].x *= scale;
            Q_h2[j][i0/WARP_SIZE].y *= scale;
        }
    }

    float VKQ[ncols] = {0.0f};

    const int k_start = parallel_blocks == 1 ? 0 : ip*D;
    for (int k_VKQ_0 = k_start; k_VKQ_0 < ne11; k_VKQ_0 += parallel_blocks*D) {
        // Calculate KQ tile and keep track of new maximum KQ values:

        float kqmax_new_arr[ncols];
#pragma unroll
        for (int j = 0; j < ncols; ++j) {
            kqmax_new_arr[j] = kqmax[j];
        }

#pragma unroll
        for (int i_KQ_0 = 0; i_KQ_0 < D; i_KQ_0 += nwarps) {
            const int i_KQ = i_KQ_0 + threadIdx.y;

            if ((i_KQ_0 + nwarps > D && i_KQ >= D) || (FATTN_KQ_STRIDE % D != 0 && k_VKQ_0 + i_KQ >= ne11)) {
                break;
            }

            float sum[ncols] = {0.0f};
#pragma unroll
            for (int k_KQ_0 = 0; k_KQ_0 < D/2; k_KQ_0 += WARP_SIZE) {
                const int k_KQ = k_KQ_0 + threadIdx.x;

                const half2 K_ik = K_h2[(k_VKQ_0 + i_KQ)*stride_KV2 + k_KQ];
#pragma unroll
                for (int j = 0; j < ncols; ++j) {
                    sum[j] +=  __low2float(K_ik) * Q_h2[j][k_KQ_0/WARP_SIZE].x;
                    sum[j] += __high2float(K_ik) * Q_h2[j][k_KQ_0/WARP_SIZE].y;
                }
            }

#pragma unroll
            for (int j = 0; j < ncols; ++j) {
                sum[j] = warp_reduce_sum(sum[j]);
                sum[j] += mask ? slope*__half2float(maskh[j*ne11 + k_VKQ_0 + i_KQ]) : 0.0f;

                kqmax_new_arr[j] = fmaxf(kqmax_new_arr[j], sum[j]);

                if (threadIdx.x == 0) {
                    KQ[j*D + i_KQ] = sum[j];
                }
            }
        }

#pragma unroll
        for (int j = 0; j < ncols; ++j) {
            float kqmax_new_j = kqmax_new_arr[j];

            kqmax_new_j = warp_reduce_max(kqmax_new_j);
            if (threadIdx.x == 0) {
                kqmax_shared[j][threadIdx.y] = kqmax_new_j;
            }
        }

        __syncthreads();

#pragma unroll
        for (int j = 0; j < ncols; ++j) {
            float kqmax_new_j = kqmax_shared[j][threadIdx.x];
            kqmax_new_j = warp_reduce_max(kqmax_new_j);

            const float KQ_max_scale = expf(kqmax[j] - kqmax_new_j);
            kqmax[j] = kqmax_new_j;

            const float val = expf(KQ[j*D + tid] - kqmax[j]);
            kqsum[j] = kqsum[j]*KQ_max_scale + val;
            KQ[j*D + tid] = val;

            VKQ[j] *= KQ_max_scale;
        }

        __syncthreads();

#pragma unroll
        for (int k = 0; k < D; ++k) {
            if (FATTN_KQ_STRIDE % D != 0 && k_VKQ_0 + k >= ne11) {
                break;
            }

            const float V_ki = __half2float(V_h[(k_VKQ_0 + k)*stride_KV + tid]);
#pragma unroll
            for (int j = 0; j < ncols; ++j) {
                VKQ[j] += V_ki*KQ[j*D + k];
            }
        }

        __syncthreads();
    }

#pragma unroll
    for (int j = 0; j < ncols; ++j) {
        kqsum[j] = warp_reduce_sum(kqsum[j]);
        if (threadIdx.x == 0) {
            kqsum_shared[j][threadIdx.y] = kqsum[j];
        }
    }

    __syncthreads();

#pragma unroll
    for (int j_VKQ = 0; j_VKQ < ncols; ++j_VKQ) {
        if (ncols > 2 && ic0 + j_VKQ >= ne01) {
            break;
        }

        kqsum[j_VKQ] = kqsum_shared[j_VKQ][threadIdx.x];
        kqsum[j_VKQ] = warp_reduce_sum(kqsum[j_VKQ]);

        float dst_val = VKQ[j_VKQ];
        if (parallel_blocks == 1) {
            dst_val /= kqsum[j_VKQ];
        }
        const int j_dst = (ic0 + j_VKQ)*parallel_blocks + ip;
        dst[j_dst*D*gridDim.y + D*blockIdx.y + tid] = dst_val;
    }

    if (parallel_blocks != 1 && tid < ncols && (ncols <= 2 || ic0 + tid < ne01)) {
        dst_meta[(ic0 + tid)*gridDim.y*parallel_blocks + blockIdx.y*parallel_blocks + ip] = make_float2(kqmax[tid], kqsum[tid]);
    }
}

template <int cols_per_block, int parallel_blocks>
void launch_fattn_vec_f32_64_128(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * Q = dst->src[0];
    switch (Q->ne[0]) {
        case  64: {
            constexpr int      D = 64;
            constexpr int nwarps = D/WARP_SIZE;
            fattn_kernel_t fattn_kernel = flash_attn_vec_ext_f32<D, cols_per_block, parallel_blocks>;
            launch_fattn<D, parallel_blocks>(ctx, dst, fattn_kernel, nwarps, cols_per_block);
        } break;
        case 128: {
            constexpr int      D = 128;
            constexpr int nwarps = D/WARP_SIZE;
            fattn_kernel_t fattn_kernel = flash_attn_vec_ext_f32<D, cols_per_block, parallel_blocks>;
            launch_fattn<D, parallel_blocks>(ctx, dst, fattn_kernel, nwarps, cols_per_block);
        } break;
        default: {
            GGML_ASSERT(false && "FlashAttention without tensor cores only supports head sizes 64 and 128.");
        } break;
    }
}

void ggml_cuda_flash_attn_ext_vec_f32(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * Q = dst->src[0];

    if (Q->ne[1] == 1) {
        constexpr int cols_per_block  = 1;
        constexpr int parallel_blocks = 4;
        launch_fattn_vec_f32_64_128<cols_per_block, parallel_blocks>(ctx, dst);
        return;
    }

    if (Q->ne[1] == 2) {
        constexpr int cols_per_block  = 2;
        constexpr int parallel_blocks = 4;
        launch_fattn_vec_f32_64_128<cols_per_block, parallel_blocks>(ctx, dst);
        return;
    }

    if (Q->ne[1] <= 4) {
        constexpr int cols_per_block  = 4;
        constexpr int parallel_blocks = 4;
        launch_fattn_vec_f32_64_128<cols_per_block, parallel_blocks>(ctx, dst);
        return;
    }

    if (Q->ne[1] <= 8) {
        constexpr int cols_per_block  = 8;
        constexpr int parallel_blocks = 4;
        launch_fattn_vec_f32_64_128<cols_per_block, parallel_blocks>(ctx, dst);
        return;
    }

    constexpr int cols_per_block  = 8;
    constexpr int parallel_blocks = 1;
    launch_fattn_vec_f32_64_128<cols_per_block, parallel_blocks>(ctx, dst);
}

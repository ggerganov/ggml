#include "hip/hip_runtime.h"
#include "common.cuh"
#include "fattn-common.cuh"
#include "fattn-vec-f32.cuh"

template<int D, int ncols, int parallel_blocks> // D == head size
#if !(defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__))
__launch_bounds__(D, 1)
#endif // !(defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__))
static __global__ void flash_attn_vec_ext_f32(
        const char * __restrict__ Q,
        const char * __restrict__ K,
        const char * __restrict__ V,
        const char * __restrict__ mask,
        float      * __restrict__ dst,
        float2     * __restrict__ dst_meta,
        const float scale,
        const float max_bias,
        const float m0,
        const float m1,
        const uint32_t n_head_log2,
        const int ne00,
        const int ne01,
        const int ne02,
        const int ne03,
        const int ne10,
        const int ne11,
        const int ne12,
        const int ne13,
        const int ne31,
        const int nb31,
        const int nb01,
        const int nb02,
        const int nb03,
        const int nb11,
        const int nb12,
        const int nb13,
        const int ne0,
        const int ne1,
        const int ne2,
        const int ne3) {
    //In this kernel Q, K, V are matrices while i, j, k are matrix indices.

    const int ic0 = (blockIdx.x / parallel_blocks) * ncols; // Index of the Q/QKV column to work on.
    const int ip  =  blockIdx.x % parallel_blocks; // Index in group of blocks running for the same column in parallel.

    const int gqa_ratio = ne02 / ne12; // With grouped query attention there are > 1 Q matrices per K, V matrix.
    const float2 * Q_f2  = (const float2 *) (Q    + nb02* blockIdx.y              + nb01*ic0);
    const half2  * K_h2  = (const half2  *) (K    + nb12*(blockIdx.y / gqa_ratio));
    const half   * V_h   = (const half   *) (V    + nb12*(blockIdx.y / gqa_ratio)); // K and V have same shape
    const half   * maskh = (const half   *)  mask + ne11*ic0;

    const int stride_KV  = nb11 / sizeof(half);
    const int stride_KV2 = nb11 / sizeof(half2);

    float slope = 1.0f;

    // ALiBi
    if (max_bias > 0.0f) {
        const uint32_t h = blockIdx.y;

        const float base = h < n_head_log2 ? m0 : m1;
        const int   exph = h < n_head_log2 ? h + 1 : 2*(h - n_head_log2) + 1;

        slope = powf(base, exph);
    }

    static_assert(D % (2*WARP_SIZE) == 0, "D not divisible by 2*WARP_SIZE == 64.");
    constexpr int nwarps = D / WARP_SIZE;
    const int tid = WARP_SIZE*threadIdx.y + threadIdx.x;
    __builtin_assume(tid < D);

    __shared__ float KQ[ncols*D];
#pragma unroll
    for (int j = 0; j < ncols; ++j) {
        KQ[j*D + tid] = -FLT_MAX/2.0f;
    }

    float kqmax[ncols];
#pragma unroll
    for (int j = 0; j < ncols; ++j) {
        kqmax[j] = -FLT_MAX/2.0f;
    }
    float kqsum[ncols] = {0.0f};

    __shared__ float kqmax_shared[ncols][WARP_SIZE];
    __shared__ float kqsum_shared[ncols][WARP_SIZE];
#pragma unroll
    for (int j = 0; j < ncols; ++j) {
        if (threadIdx.y == 0) {
            kqmax_shared[j][threadIdx.x] = -FLT_MAX/2.0f;
            kqsum_shared[j][threadIdx.x] = 0.0f;
        }
    }
    __syncthreads();

    // Convert Q to half2 and store in registers:
    float2 Q_h2[ncols][D/(2*WARP_SIZE)];
#pragma unroll
    for (int j = 0; j < ncols; ++j) {
#pragma unroll
        for (int i0 = 0; i0 < D/2; i0 += WARP_SIZE) {
            const int i = i0 + threadIdx.x;

            Q_h2[j][i0/WARP_SIZE]    = Q_f2[j*(nb01/sizeof(float2)) + i];
            Q_h2[j][i0/WARP_SIZE].x *= scale;
            Q_h2[j][i0/WARP_SIZE].y *= scale;
        }
    }

    float VKQ[ncols] = {0.0f};

    const int k_start = parallel_blocks == 1 ? 0 : ip*D;
    for (int k_VKQ_0 = k_start; k_VKQ_0 < ne11; k_VKQ_0 += parallel_blocks*D) {
        // Calculate KQ tile and keep track of new maximum KQ values:

        float kqmax_new_arr[ncols];
#pragma unroll
        for (int j = 0; j < ncols; ++j) {
            kqmax_new_arr[j] = kqmax[j];
        }

#pragma unroll
        for (int i_KQ_0 = 0; i_KQ_0 < D; i_KQ_0 += nwarps) {
            const int i_KQ = i_KQ_0 + threadIdx.y;

            if ((i_KQ_0 + nwarps > D && i_KQ >= D) || (FATTN_KQ_STRIDE % D != 0 && k_VKQ_0 + i_KQ >= ne11)) {
                break;
            }

            float sum[ncols] = {0.0f};
#pragma unroll
            for (int k_KQ_0 = 0; k_KQ_0 < D/2; k_KQ_0 += WARP_SIZE) {
                const int k_KQ = k_KQ_0 + threadIdx.x;

                const half2 K_ik = K_h2[(k_VKQ_0 + i_KQ)*stride_KV2 + k_KQ];
#pragma unroll
                for (int j = 0; j < ncols; ++j) {
                    sum[j] +=  __low2float(K_ik) * Q_h2[j][k_KQ_0/WARP_SIZE].x;
                    sum[j] += __high2float(K_ik) * Q_h2[j][k_KQ_0/WARP_SIZE].y;
                }
            }

#pragma unroll
            for (int j = 0; j < ncols; ++j) {
                sum[j] = warp_reduce_sum(sum[j]);
                sum[j] += mask ? slope*__half2float(maskh[j*ne11 + k_VKQ_0 + i_KQ]) : 0.0f;

                kqmax_new_arr[j] = fmaxf(kqmax_new_arr[j], sum[j]);

                if (threadIdx.x == 0) {
                    KQ[j*D + i_KQ] = sum[j];
                }
            }
        }

#pragma unroll
        for (int j = 0; j < ncols; ++j) {
            float kqmax_new_j = kqmax_new_arr[j];

            kqmax_new_j = warp_reduce_max(kqmax_new_j);
            if (threadIdx.x == 0) {
                kqmax_shared[j][threadIdx.y] = kqmax_new_j;
            }
        }

        __syncthreads();

#pragma unroll
        for (int j = 0; j < ncols; ++j) {
            float kqmax_new_j = kqmax_shared[j][threadIdx.x];
            kqmax_new_j = warp_reduce_max(kqmax_new_j);

            const float KQ_max_scale = expf(kqmax[j] - kqmax_new_j);
            kqmax[j] = kqmax_new_j;

            const float val = expf(KQ[j*D + tid] - kqmax[j]);
            kqsum[j] = kqsum[j]*KQ_max_scale + val;
            KQ[j*D + tid] = val;

            VKQ[j] *= KQ_max_scale;
        }

        __syncthreads();

#pragma unroll
        for (int k = 0; k < D; ++k) {
            if (FATTN_KQ_STRIDE % D != 0 && k_VKQ_0 + k >= ne11) {
                break;
            }

            const float V_ki = __half2float(V_h[(k_VKQ_0 + k)*stride_KV + tid]);
#pragma unroll
            for (int j = 0; j < ncols; ++j) {
                VKQ[j] += V_ki*KQ[j*D + k];
            }
        }

        __syncthreads();
    }

#pragma unroll
    for (int j = 0; j < ncols; ++j) {
        kqsum[j] = warp_reduce_sum(kqsum[j]);
        if (threadIdx.x == 0) {
            kqsum_shared[j][threadIdx.y] = kqsum[j];
        }
    }

    __syncthreads();

#pragma unroll
    for (int j_VKQ = 0; j_VKQ < ncols; ++j_VKQ) {
        kqsum[j_VKQ] = kqsum_shared[j_VKQ][threadIdx.x];
        kqsum[j_VKQ] = warp_reduce_sum(kqsum[j_VKQ]);

        float dst_val = VKQ[j_VKQ];
        if (parallel_blocks == 1) {
            dst_val /= kqsum[j_VKQ];
        }
        const int j_dst = (ic0 + j_VKQ)*parallel_blocks + ip;
        dst[j_dst*D*gridDim.y + D*blockIdx.y + tid] = dst_val;
    }

    if (parallel_blocks != 1 && threadIdx.x < ncols) {
        dst_meta[(ic0 + threadIdx.x)*gridDim.y*parallel_blocks + blockIdx.y*parallel_blocks + ip] = make_float2(kqmax[threadIdx.x], kqsum[threadIdx.x]);
    }
}

template <int D, int cols_per_block, int parallel_blocks> void launch_fattn_vec_f32(
        const ggml_tensor * Q, const ggml_tensor * K, const ggml_tensor * V, ggml_tensor * KQV, const ggml_tensor * mask,
        ggml_cuda_pool & pool, hipStream_t main_stream
) {
    ggml_cuda_pool_alloc<float>  dst_tmp(pool);
    ggml_cuda_pool_alloc<float2> dst_tmp_meta(pool);

    if (parallel_blocks > 1) {
        dst_tmp.alloc(parallel_blocks*ggml_nelements(KQV));
        dst_tmp_meta.alloc(parallel_blocks*ggml_nrows(KQV));
    }

    constexpr int  nwarps = (D + WARP_SIZE - 1) / WARP_SIZE;
    const     dim3 block_dim(WARP_SIZE, nwarps, 1);
    const     dim3 blocks_num(parallel_blocks*((Q->ne[1] + cols_per_block - 1) / cols_per_block), Q->ne[2], Q->ne[3]);
    const     int  shmem = 0;

    float scale    = 1.0f;
    float max_bias = 0.0f;

    memcpy(&scale,    (float *) KQV->op_params + 0, sizeof(float));
    memcpy(&max_bias, (float *) KQV->op_params + 1, sizeof(float));

    const uint32_t n_head      = Q->ne[2];
    const uint32_t n_head_log2 = 1u << (uint32_t) floorf(log2f((float) n_head));

    const float m0 = powf(2.0f, -(max_bias       ) / n_head_log2);
    const float m1 = powf(2.0f, -(max_bias / 2.0f) / n_head_log2);

    flash_attn_vec_ext_f32<D, cols_per_block, parallel_blocks>
        <<<blocks_num, block_dim, shmem, main_stream>>> (
                (const char *) Q->data,
                (const char *) K->data,
                (const char *) V->data,
                mask ? ((const char *) mask->data) : nullptr,
                parallel_blocks == 1 ? (float *) KQV->data : dst_tmp.ptr, dst_tmp_meta.ptr,
                scale, max_bias, m0, m1, n_head_log2,
                Q->ne[0], Q->ne[1], Q->ne[2], Q->ne[3],
                K->ne[0], K->ne[1], K->ne[2], K->ne[3],
                mask ? mask->ne[1] : 0, mask ?  mask->nb[1] : 0,
                Q->nb[1], Q->nb[2], Q->nb[3],
                K->nb[1], K->nb[2], K->nb[3],
                KQV->ne[0], KQV->ne[1], KQV->ne[2], KQV->ne[3]
                );
    CUDA_CHECK(hipGetLastError());

    if (parallel_blocks == 1) {
        return;
    }

    const dim3 block_dim_combine(D, 1, 1);
    const dim3 blocks_num_combine(Q->ne[1], blocks_num.y, blocks_num.z);
    const int  shmem_combine = 0;

    flash_attn_combine_results<D, parallel_blocks>
        <<<blocks_num_combine, block_dim_combine, shmem_combine, main_stream>>>
        (dst_tmp.ptr, dst_tmp_meta.ptr, (float *) KQV->data);
    CUDA_CHECK(hipGetLastError());
}

void ggml_cuda_flash_attn_ext_vec_f32(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * Q = dst->src[0];
    const ggml_tensor * K = dst->src[1];
    const ggml_tensor * V = dst->src[2];

    const ggml_tensor * mask = dst->src[3];

    ggml_tensor * KQV = dst;

    GGML_ASSERT(Q->ne[0] == 64 || Q->ne[0] == 128 && "FlashAttention without tensor cores only supports head sizes 64 and 128.");

    if (Q->ne[1] == 1) {
        constexpr int cols_per_block = 1;
        constexpr int parallel_blocks = 4;
        switch (Q->ne[0]) {
            case 64:
                launch_fattn_vec_f32< 64, cols_per_block, parallel_blocks>(Q, K, V, KQV, mask, ctx.pool(), ctx.stream());
                break;
            case 128:
                launch_fattn_vec_f32<128, cols_per_block, parallel_blocks>(Q, K, V, KQV, mask, ctx.pool(), ctx.stream());
                break;
            default:
                GGML_ASSERT(false);
                break;
        }
        return;
    }

    if (Q->ne[1] == 2) {
        constexpr int cols_per_block = 2;
        constexpr int parallel_blocks = 4;
        switch (Q->ne[0]) {
            case 64:
                launch_fattn_vec_f32< 64, cols_per_block, parallel_blocks>(Q, K, V, KQV, mask, ctx.pool(), ctx.stream());
                break;
            case 128:
                launch_fattn_vec_f32<128, cols_per_block, parallel_blocks>(Q, K, V, KQV, mask, ctx.pool(), ctx.stream());
                break;
            default:
                GGML_ASSERT(false);
                break;
        }
        return;
    }

    if (Q->ne[1] <= 4) {
        constexpr int cols_per_block = 4;
        constexpr int parallel_blocks = 4;
        switch (Q->ne[0]) {
            case 64:
                launch_fattn_vec_f32< 64, cols_per_block, parallel_blocks>(Q, K, V, KQV, mask, ctx.pool(), ctx.stream());
                break;
            case 128:
                launch_fattn_vec_f32<128, cols_per_block, parallel_blocks>(Q, K, V, KQV, mask, ctx.pool(), ctx.stream());
                break;
            default:
                GGML_ASSERT(false);
                break;
        }
        return;
    }

    if (Q->ne[1] <= 8) {
        constexpr int cols_per_block = 8;
        constexpr int parallel_blocks = 4;
        switch (Q->ne[0]) {
            case 64:
                launch_fattn_vec_f32< 64, cols_per_block, parallel_blocks>(Q, K, V, KQV, mask, ctx.pool(), ctx.stream());
                break;
            case 128:
                launch_fattn_vec_f32<128, cols_per_block, parallel_blocks>(Q, K, V, KQV, mask, ctx.pool(), ctx.stream());
                break;
            default:
                GGML_ASSERT(false);
                break;
        }
        return;
    }

    constexpr int cols_per_block = 8;
    constexpr int parallel_blocks = 1;
    switch (Q->ne[0]) {
        case 64:
            launch_fattn_vec_f32< 64, cols_per_block, parallel_blocks>(Q, K, V, KQV, mask, ctx.pool(), ctx.stream());
            break;
        case 128:
            launch_fattn_vec_f32<128, cols_per_block, parallel_blocks>(Q, K, V, KQV, mask, ctx.pool(), ctx.stream());
            break;
        default:
            GGML_ASSERT(false);
            break;
    }
}

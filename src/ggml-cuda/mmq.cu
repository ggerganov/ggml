#include "hip/hip_runtime.h"
#include "mmq.cuh"
#include "vecdotq.cuh"

typedef void (*allocate_tiles_cuda_t)(int ** x_ql, half2 ** x_dm, int ** x_qh, int ** x_sc);
typedef void (*load_tiles_cuda_t)(
    const void * __restrict__ vx, int * __restrict__ x_ql, half2 * __restrict__ x_dm, int * __restrict__ x_qh,
    int * __restrict__ x_sc, const int & i_offset, const int & i_max, const int & k, const int & blocks_per_row);
typedef float (*vec_dot_q_mul_mat_cuda_t)(
    const int * __restrict__ x_ql, const half2 * __restrict__ x_dm, const int * __restrict__ x_qh, const int * __restrict__ x_sc,
    const int * __restrict__ y_qs, const half2 * __restrict__ y_ms, const int & i, const int & j, const int & k);
typedef void (*dot_kernel_k_t)(const void * __restrict__ vx, const int ib, const int iqs, const float * __restrict__ y, float & v);
typedef void (mul_mat_q_t)(
    const void * __restrict__ vx, const void * __restrict__ vy, float * __restrict__ dst,
    const int ncols_x, const int nrows_x, const int ncols_y, const int nrows_y, const int nrows_dst);

struct mmq_arch_config_t {
    int x;
    int y;
    int nwarps;
};

struct mmq_config_t {
    mmq_arch_config_t rdna2;
    mmq_arch_config_t rdna1;
    mmq_arch_config_t ampere;
    mmq_arch_config_t pascal;
};

constexpr mmq_config_t MMQ_CONFIG_Q4_0 = {
//        x    y  nwarps
        { 64, 128, 8},
        { 64,  64, 8},
#ifdef CUDA_USE_TENSOR_CORES
        {  4,  32, 4},
#else
        { 64, 128, 4},
#endif // CUDA_USE_TENSOR_CORES
        { 64,  64, 8},
};
constexpr mmq_config_t MMQ_CONFIG_Q4_1 = {
//        x    y  nwarps
        { 64, 128, 8},
        { 64,  64, 8},
#ifdef CUDA_USE_TENSOR_CORES
        {  4,  32, 4},
#else
        { 64, 128, 4},
#endif // CUDA_USE_TENSOR_CORES
        { 64,  64, 8},
};
constexpr mmq_config_t MMQ_CONFIG_Q5_0 = {
//        x    y  nwarps
        { 64, 128, 8},
        { 64,  64, 8},
#ifdef CUDA_USE_TENSOR_CORES
        {  4,  32, 4},
#else
        {128,  64, 4},
#endif // CUDA_USE_TENSOR_CORES
        { 64,  64, 8},
};
constexpr mmq_config_t MMQ_CONFIG_Q5_1 = {
//        x    y  nwarps
        { 64, 128, 8},
        { 64,  64, 8},
#ifdef CUDA_USE_TENSOR_CORES
        {  4,  32, 4},
#else
        {128,  64, 4},
#endif // CUDA_USE_TENSOR_CORES
        { 64,  64, 8},
};
constexpr mmq_config_t MMQ_CONFIG_Q8_0 = {
//        x    y  nwarps
        { 64, 128, 8},
        { 64,  64, 8},
#ifdef CUDA_USE_TENSOR_CORES
        {  4,  32, 4},
#else
        {128,  64, 4},
#endif // CUDA_USE_TENSOR_CORES
        { 64,  64, 8},
};
constexpr mmq_config_t MMQ_CONFIG_Q2_K = {
//        x    y  nwarps
        { 64, 128, 8},
        {128,  32, 8},
#ifdef CUDA_USE_TENSOR_CORES
        {  4,  32, 4},
#else
        { 64, 128, 4},
#endif // CUDA_USE_TENSOR_CORES
        { 64,  64, 8},
};
constexpr mmq_config_t MMQ_CONFIG_Q3_K = {
//        x    y  nwarps
        {128,  64, 8},
        { 32, 128, 8},
#ifdef CUDA_USE_TENSOR_CORES
        {  4,  32, 4},
#else
        {128, 128, 4},
#endif // CUDA_USE_TENSOR_CORES
        { 64,  64, 8},
};
constexpr mmq_config_t MMQ_CONFIG_Q4_K = {
//        x    y  nwarps
        { 64, 128, 8},
        { 32,  64, 8},
#ifdef CUDA_USE_TENSOR_CORES
        {  4,  32, 4},
#else
        { 64, 128, 4},
#endif // CUDA_USE_TENSOR_CORES
        { 64,  64, 8},
};
constexpr mmq_config_t MMQ_CONFIG_Q5_K = {
//        x    y  nwarps
        { 64, 128, 8},
        { 32,  64, 8},
#ifdef CUDA_USE_TENSOR_CORES
        {  4,  32, 4},
#else
        { 64, 128, 4},
#endif // CUDA_USE_TENSOR_CORES
        { 64,  64, 8},
};
constexpr mmq_config_t MMQ_CONFIG_Q6_K = {
//        x    y  nwarps
        { 64, 128, 8},
        { 32,  64, 8},
#ifdef CUDA_USE_TENSOR_CORES
        {  4,  32, 4},
#else
        { 64,  64, 4},
#endif // CUDA_USE_TENSOR_CORES
        { 64,  64, 8},
};

// ------------------------------------------------------------

template <int mmq_y> static __device__ __forceinline__ void allocate_tiles_q4_0(int ** x_ql, half2 ** x_dm, int ** x_qh, int ** x_sc) {
    GGML_UNUSED(x_qh);
    GGML_UNUSED(x_sc);

    __shared__ int  tile_x_qs[mmq_y * (WARP_SIZE)       + mmq_y];
    __shared__ float tile_x_d[mmq_y * (WARP_SIZE/QI4_0) + mmq_y/QI4_0];

    *x_ql = tile_x_qs;
    *x_dm = (half2 *) tile_x_d;
}

template <int mmq_y, int nwarps, bool need_check> static __device__ __forceinline__ void load_tiles_q4_0(
    const void * __restrict__ vx, int * __restrict__ x_ql, half2 * __restrict__ x_dm, int * __restrict__ x_qh,
    int * __restrict__ x_sc, const int & i_offset, const int & i_max, const int & k, const int & blocks_per_row) {
    GGML_UNUSED(x_qh); GGML_UNUSED(x_sc);
    GGML_CUDA_ASSUME(i_offset >= 0);
    GGML_CUDA_ASSUME(i_offset <  nwarps);
    GGML_CUDA_ASSUME(k >= 0);
    GGML_CUDA_ASSUME(k <  WARP_SIZE);

    const int kbx  = k / QI4_0;
    const int kqsx = k % QI4_0;

    const block_q4_0 * bx0 = (const block_q4_0 *) vx;

    float * x_dmf = (float *) x_dm;

#pragma unroll
    for (int i0 = 0; i0 < mmq_y; i0 += nwarps) {
        int i = i0 + i_offset;

        if (need_check) {
            i = min(i, i_max);
        }

        const block_q4_0 * bxi = bx0 + i*blocks_per_row + kbx;

        x_ql[i * (WARP_SIZE + 1) + k] = get_int_from_uint8(bxi->qs, kqsx);
        // x_dmf[i * (WARP_SIZE/QI4_0) + i / QI4_0 + kbx] = bxi->d;
    }

    const int blocks_per_tile_x_row = WARP_SIZE / QI4_0;
    const int kbxd = k % blocks_per_tile_x_row;

#pragma unroll
    for (int i0 = 0; i0 < mmq_y; i0 += nwarps * QI4_0) {
        int i = i0 + i_offset * QI4_0 + k / blocks_per_tile_x_row;

        if (need_check) {
            i = min(i, i_max);
        }

        const block_q4_0 * bxi = bx0 + i*blocks_per_row + kbxd;

        x_dmf[i * (WARP_SIZE/QI4_0) + i / QI4_0 + kbxd] = bxi->d;
    }
}

static __device__ __forceinline__ float vec_dot_q4_0_q8_1_mul_mat(
    const int * __restrict__ x_ql, const half2 * __restrict__ x_dm, const int * __restrict__ x_qh, const int * __restrict__ x_sc,
    const int * __restrict__ y_qs, const half2 * __restrict__ y_ds, const int & i, const int & j, const int & k) {
    GGML_UNUSED(x_qh); GGML_UNUSED(x_sc);

    const int kyqs = k % (QI8_1/2) + QI8_1 * (k / (QI8_1/2));
    const float * x_dmf = (const float *) x_dm;

    int u[2*VDR_Q4_0_Q8_1_MMQ];

#pragma unroll
    for (int l = 0; l < VDR_Q4_0_Q8_1_MMQ; ++l) {
        u[2*l+0] = y_qs[j * WARP_SIZE + (kyqs + l)         % WARP_SIZE];
        u[2*l+1] = y_qs[j * WARP_SIZE + (kyqs + l + QI4_0) % WARP_SIZE];
    }

    return vec_dot_q4_0_q8_1_impl<VDR_Q4_0_Q8_1_MMQ>
        (&x_ql[i * (WARP_SIZE + 1) + k], u, x_dmf[i * (WARP_SIZE/QI4_0) + i/QI4_0 + k/QI4_0],
         y_ds[j * (WARP_SIZE/QI8_1) + (2*k/QI8_1) % (WARP_SIZE/QI8_1)]);
}

template <int mmq_y> static __device__ __forceinline__ void allocate_tiles_q4_1(int ** x_ql, half2 ** x_dm, int ** x_qh, int ** x_sc) {
    GGML_UNUSED(x_qh); GGML_UNUSED(x_sc);

    __shared__ int   tile_x_qs[mmq_y * (WARP_SIZE) +     + mmq_y];
    __shared__ half2 tile_x_dm[mmq_y * (WARP_SIZE/QI4_1) + mmq_y/QI4_1];

    *x_ql = tile_x_qs;
    *x_dm = tile_x_dm;
}

template <int mmq_y, int nwarps, bool need_check> static __device__ __forceinline__ void load_tiles_q4_1(
    const void * __restrict__ vx, int * __restrict__ x_ql, half2 * __restrict__ x_dm, int * __restrict__ x_qh,
    int * __restrict__ x_sc, const int & i_offset, const int & i_max, const int & k, const int & blocks_per_row) {
    GGML_UNUSED(x_qh); GGML_UNUSED(x_sc);

    GGML_CUDA_ASSUME(i_offset >= 0);
    GGML_CUDA_ASSUME(i_offset <  nwarps);
    GGML_CUDA_ASSUME(k >= 0);
    GGML_CUDA_ASSUME(k <  WARP_SIZE);

    const int kbx  = k / QI4_1;
    const int kqsx = k % QI4_1;

    const block_q4_1 * bx0 = (const block_q4_1 *) vx;

#pragma unroll
    for (int i0 = 0; i0 < mmq_y; i0 += nwarps) {
        int i = i0 + i_offset;

        if (need_check) {
            i = min(i, i_max);
        }

        const block_q4_1 * bxi = bx0 + i*blocks_per_row + kbx;

        x_ql[i * (WARP_SIZE + 1) + k] = get_int_from_uint8_aligned(bxi->qs, kqsx);
    }

    const int blocks_per_tile_x_row = WARP_SIZE / QI4_1;
    const int kbxd = k % blocks_per_tile_x_row;

#pragma unroll
    for (int i0 = 0; i0 < mmq_y; i0 += nwarps * QI4_1) {
        int i = i0 + i_offset * QI4_1 + k / blocks_per_tile_x_row;

        if (need_check) {
            i = min(i, i_max);
        }

        const block_q4_1 * bxi = bx0 + i*blocks_per_row + kbxd;

        x_dm[i * (WARP_SIZE/QI4_1) + i / QI4_1 + kbxd] = bxi->dm;
    }
}

static __device__ __forceinline__ float vec_dot_q4_1_q8_1_mul_mat(
    const int * __restrict__ x_ql, const half2 * __restrict__ x_dm, const int * __restrict__ x_qh, const int * __restrict__ x_sc,
    const int * __restrict__ y_qs, const half2 * __restrict__ y_ds, const int & i, const int & j, const int & k) {
    GGML_UNUSED(x_qh); GGML_UNUSED(x_sc);

    const int kyqs = k % (QI8_1/2) + QI8_1 * (k / (QI8_1/2));

    int u[2*VDR_Q4_1_Q8_1_MMQ];

#pragma unroll
    for (int l = 0; l < VDR_Q4_1_Q8_1_MMQ; ++l) {
        u[2*l+0] = y_qs[j * WARP_SIZE + (kyqs + l)         % WARP_SIZE];
        u[2*l+1] = y_qs[j * WARP_SIZE + (kyqs + l + QI4_1) % WARP_SIZE];
    }

    return vec_dot_q4_1_q8_1_impl<VDR_Q4_1_Q8_1_MMQ>
        (&x_ql[i * (WARP_SIZE + 1) + k], u, x_dm[i * (WARP_SIZE/QI4_1) + i/QI4_1 + k/QI4_1],
         y_ds[j * (WARP_SIZE/QI8_1) + (2*k/QI8_1) % (WARP_SIZE/QI8_1)]);
}

template <int mmq_y> static __device__ __forceinline__ void allocate_tiles_q5_0(int ** x_ql, half2 ** x_dm, int ** x_qh, int ** x_sc) {
    GGML_UNUSED(x_qh); GGML_UNUSED(x_sc);

    __shared__ int  tile_x_ql[mmq_y * (2*WARP_SIZE)     + mmq_y];
    __shared__ float tile_x_d[mmq_y * (WARP_SIZE/QI5_0) + mmq_y/QI5_0];

    *x_ql = tile_x_ql;
    *x_dm = (half2 *) tile_x_d;
}

template <int mmq_y, int nwarps, bool need_check> static __device__ __forceinline__ void load_tiles_q5_0(
    const void * __restrict__ vx, int * __restrict__ x_ql, half2 * __restrict__ x_dm, int * __restrict__ x_qh,
    int * __restrict__ x_sc, const int & i_offset, const int & i_max, const int & k, const int & blocks_per_row) {
    GGML_UNUSED(x_qh); GGML_UNUSED(x_sc);

    GGML_CUDA_ASSUME(i_offset >= 0);
    GGML_CUDA_ASSUME(i_offset <  nwarps);
    GGML_CUDA_ASSUME(k >= 0);
    GGML_CUDA_ASSUME(k <  WARP_SIZE);

    const int kbx  = k / QI5_0;
    const int kqsx = k % QI5_0;

    const block_q5_0 * bx0 = (const block_q5_0 *) vx;

#pragma unroll
    for (int i0 = 0; i0 < mmq_y; i0 += nwarps) {
        int i = i0 + i_offset;

        if (need_check) {
            i = min(i, i_max);
        }

        const block_q5_0 * bxi = bx0 + i*blocks_per_row + kbx;

        const int ql = get_int_from_uint8(bxi->qs, kqsx);
        const int qh = get_int_from_uint8(bxi->qh, 0) >> (4 * (k % QI5_0));

        int qs0 = (ql >>  0)   & 0x0F0F0F0F;
        qs0    |= (qh <<  4)   & 0x00000010;  // 0 ->  4
        qs0    |= (qh << 11)   & 0x00001000;  // 1 -> 12
        qs0    |= (qh << 18)   & 0x00100000;  // 2 -> 20
        qs0    |= (qh << 25)   & 0x10000000;  // 3 -> 28
        qs0     = __vsubss4(qs0, 0x10101010); // subtract 16

        x_ql[i * (2*WARP_SIZE + 1) + 2*k+0] = qs0;

        int qs1 = (ql >>  4)   & 0x0F0F0F0F;
        qs1    |= (qh >> 12)   & 0x00000010;  // 16 ->  4
        qs1    |= (qh >>  5)   & 0x00001000;  // 17 -> 12
        qs1    |= (qh <<  2)   & 0x00100000;  // 18 -> 20
        qs1    |= (qh <<  9)   & 0x10000000;  // 19 -> 28
        qs1     = __vsubss4(qs1, 0x10101010); // subtract 16

        x_ql[i * (2*WARP_SIZE + 1) + 2*k+1] = qs1;
    }

    const int blocks_per_tile_x_row = WARP_SIZE / QI5_0;
    const int kbxd = k % blocks_per_tile_x_row;
    float * x_dmf = (float *) x_dm;

#pragma unroll
    for (int i0 = 0; i0 < mmq_y; i0 += nwarps * QI5_0) {
        int i = i0 + i_offset * QI5_0 + k / blocks_per_tile_x_row;

        if (need_check) {
            i = min(i, i_max);
        }

        const block_q5_0 * bxi = bx0 + i*blocks_per_row + kbxd;

        x_dmf[i * (WARP_SIZE/QI5_0) + i / QI5_0 + kbxd] = bxi->d;
    }
}

static __device__ __forceinline__ float vec_dot_q5_0_q8_1_mul_mat(
    const int * __restrict__ x_ql, const half2 * __restrict__ x_dm, const int * __restrict__ x_qh, const int * __restrict__ x_sc,
    const int * __restrict__ y_qs, const half2 * __restrict__ y_ds, const int & i, const int & j, const int & k) {
    GGML_UNUSED(x_qh); GGML_UNUSED(x_sc);

    const int kyqs = k % (QI8_1/2) + QI8_1 * (k / (QI8_1/2));
    const int index_bx = i * (WARP_SIZE/QI5_0) + i/QI5_0 + k/QI5_0;
    const float * x_dmf = (const float *) x_dm;
    const float * y_df  = (const float *) y_ds;

    int u[2*VDR_Q5_0_Q8_1_MMQ];

#pragma unroll
    for (int l = 0; l < VDR_Q5_0_Q8_1_MMQ; ++l) {
        u[2*l+0] = y_qs[j * WARP_SIZE + (kyqs + l)         % WARP_SIZE];
        u[2*l+1] = y_qs[j * WARP_SIZE + (kyqs + l + QI5_0) % WARP_SIZE];
    }

    return vec_dot_q8_0_q8_1_impl<QR5_0*VDR_Q5_0_Q8_1_MMQ>
        (&x_ql[i * (2*WARP_SIZE + 1) + 2 * k], u, x_dmf[index_bx], y_df[j * (WARP_SIZE/QI8_1) + (2*k/QI8_1) % (WARP_SIZE/QI8_1)]);
}


template <int mmq_y> static __device__ __forceinline__ void allocate_tiles_q5_1(int ** x_ql, half2 ** x_dm, int ** x_qh, int ** x_sc) {
    GGML_UNUSED(x_qh); GGML_UNUSED(x_sc);

    __shared__ int   tile_x_ql[mmq_y * (2*WARP_SIZE)     + mmq_y];
    __shared__ half2 tile_x_dm[mmq_y * (WARP_SIZE/QI5_1) + mmq_y/QI5_1];

    *x_ql = tile_x_ql;
    *x_dm = tile_x_dm;
}

template <int mmq_y, int nwarps, bool need_check> static __device__ __forceinline__ void load_tiles_q5_1(
    const void * __restrict__ vx, int * __restrict__ x_ql, half2 * __restrict__ x_dm, int * __restrict__ x_qh,
    int * __restrict__ x_sc, const int & i_offset, const int & i_max, const int & k, const int & blocks_per_row) {
    GGML_UNUSED(x_qh); GGML_UNUSED(x_sc);

    GGML_CUDA_ASSUME(i_offset >= 0);
    GGML_CUDA_ASSUME(i_offset < nwarps);
    GGML_CUDA_ASSUME(k >= 0);
    GGML_CUDA_ASSUME(k <  WARP_SIZE);

    const int kbx  = k / QI5_1;
    const int kqsx = k % QI5_1;

    const block_q5_1 * bx0 = (const block_q5_1 *) vx;

#pragma unroll
    for (int i0 = 0; i0 < mmq_y; i0 += nwarps) {
        int i = i0 + i_offset;

        if (need_check) {
            i = min(i, i_max);
        }

        const block_q5_1 * bxi = bx0 + i*blocks_per_row + kbx;

        const int ql = get_int_from_uint8_aligned(bxi->qs, kqsx);
        const int qh = get_int_from_uint8_aligned(bxi->qh, 0) >> (4 * (k % QI5_1));

        int qs0 = (ql >>  0) & 0x0F0F0F0F;
        qs0    |= (qh <<  4) & 0x00000010; // 0 ->  4
        qs0    |= (qh << 11) & 0x00001000; // 1 -> 12
        qs0    |= (qh << 18) & 0x00100000; // 2 -> 20
        qs0    |= (qh << 25) & 0x10000000; // 3 -> 28

        x_ql[i * (2*WARP_SIZE + 1) + 2*k+0] = qs0;

        int qs1 = (ql >>  4) & 0x0F0F0F0F;
        qs1    |= (qh >> 12) & 0x00000010; // 16 ->  4
        qs1    |= (qh >>  5) & 0x00001000; // 17 -> 12
        qs1    |= (qh <<  2) & 0x00100000; // 18 -> 20
        qs1    |= (qh <<  9) & 0x10000000; // 19 -> 28

        x_ql[i * (2*WARP_SIZE + 1) + 2*k+1] = qs1;
    }

    const int blocks_per_tile_x_row = WARP_SIZE / QI5_1;
    const int kbxd = k % blocks_per_tile_x_row;

#pragma unroll
    for (int i0 = 0; i0 < mmq_y; i0 += nwarps * QI5_1) {
        int i = i0 + i_offset * QI5_1 + k / blocks_per_tile_x_row;

        if (need_check) {
            i = min(i, i_max);
        }

        const block_q5_1 * bxi = bx0 + i*blocks_per_row + kbxd;

        x_dm[i * (WARP_SIZE/QI5_1) + i / QI5_1 + kbxd] = bxi->dm;
    }
}

static __device__ __forceinline__ float vec_dot_q5_1_q8_1_mul_mat(
    const int * __restrict__ x_ql, const half2 * __restrict__ x_dm, const int * __restrict__ x_qh, const int * __restrict__ x_sc,
    const int * __restrict__ y_qs, const half2 * __restrict__ y_ds, const int & i, const int & j, const int & k) {
    GGML_UNUSED(x_qh); GGML_UNUSED(x_sc);

    const int kyqs = k % (QI8_1/2) + QI8_1 * (k / (QI8_1/2));
    const int index_bx = i * (WARP_SIZE/QI5_1) + + i/QI5_1 + k/QI5_1;

    int u[2*VDR_Q5_1_Q8_1_MMQ];

#pragma unroll
    for (int l = 0; l < VDR_Q5_1_Q8_1_MMQ; ++l) {
        u[2*l+0] = y_qs[j * WARP_SIZE + (kyqs + l)         % WARP_SIZE];
        u[2*l+1] = y_qs[j * WARP_SIZE + (kyqs + l + QI5_1) % WARP_SIZE];
    }

    return vec_dot_q8_1_q8_1_impl<QR5_1*VDR_Q5_1_Q8_1_MMQ>
        (&x_ql[i * (2*WARP_SIZE + 1) + 2 * k], u, x_dm[index_bx], y_ds[j * (WARP_SIZE/QI8_1) + (2*k/QI8_1) % (WARP_SIZE/QI8_1)]);
}

template <int mmq_y> static __device__ __forceinline__ void allocate_tiles_q8_0(int ** x_ql, half2 ** x_dm, int ** x_qh, int ** x_sc) {
    GGML_UNUSED(x_qh); GGML_UNUSED(x_sc);

    __shared__ int  tile_x_qs[mmq_y * (WARP_SIZE)       + mmq_y];
    __shared__ float tile_x_d[mmq_y * (WARP_SIZE/QI8_0) + mmq_y/QI8_0];

    *x_ql = tile_x_qs;
    *x_dm = (half2 *) tile_x_d;
}

template <int mmq_y, int nwarps, bool need_check> static __device__ __forceinline__ void load_tiles_q8_0(
    const void * __restrict__ vx, int * __restrict__ x_ql, half2 * __restrict__ x_dm, int * __restrict__ x_qh,
    int * __restrict__ x_sc, const int & i_offset, const int & i_max, const int & k, const int & blocks_per_row) {
    GGML_UNUSED(x_qh); GGML_UNUSED(x_sc);

    GGML_CUDA_ASSUME(i_offset >= 0);
    GGML_CUDA_ASSUME(i_offset <  nwarps);
    GGML_CUDA_ASSUME(k >= 0);
    GGML_CUDA_ASSUME(k <  WARP_SIZE);

    const int kbx  = k / QI8_0;
    const int kqsx = k % QI8_0;
    float * x_dmf = (float *) x_dm;

    const block_q8_0 * bx0 = (const block_q8_0 *) vx;

#pragma unroll
    for (int i0 = 0; i0 < mmq_y; i0 += nwarps) {
        int i = i0 + i_offset;

        if (need_check) {
            i = min(i, i_max);
        }

        const block_q8_0 * bxi = bx0 + i*blocks_per_row + kbx;

        x_ql[i * (WARP_SIZE + 1) + k] = get_int_from_int8(bxi->qs, kqsx);
    }

    const int blocks_per_tile_x_row = WARP_SIZE / QI8_0;
    const int kbxd = k % blocks_per_tile_x_row;

#pragma unroll
    for (int i0 = 0; i0 < mmq_y; i0 += nwarps * QI8_0) {
        int i = i0 + i_offset * QI8_0 + k / blocks_per_tile_x_row;

        if (need_check) {
            i = min(i, i_max);
        }

        const block_q8_0 * bxi = bx0 + i*blocks_per_row + kbxd;

        x_dmf[i * (WARP_SIZE/QI8_0) + i / QI8_0 + kbxd] = bxi->d;
    }
}

static __device__ __forceinline__ float vec_dot_q8_0_q8_1_mul_mat(
    const int * __restrict__ x_ql, const half2 * __restrict__ x_dm, const int * __restrict__ x_qh, const int * __restrict__ x_sc,
    const int * __restrict__ y_qs, const half2 * __restrict__ y_ds, const int & i, const int & j, const int & k) {
    GGML_UNUSED(x_qh); GGML_UNUSED(x_sc);

    const float * x_dmf = (const float *) x_dm;
    const float * y_df  = (const float *) y_ds;

    return vec_dot_q8_0_q8_1_impl<VDR_Q8_0_Q8_1_MMQ>
        (&x_ql[i * (WARP_SIZE + 1) + k], &y_qs[j * WARP_SIZE + k], x_dmf[i * (WARP_SIZE/QI8_0) + i/QI8_0 + k/QI8_0],
         y_df[j * (WARP_SIZE/QI8_1) + k/QI8_1]);
}

template <int mmq_y> static __device__ __forceinline__ void allocate_tiles_q2_K(int ** x_ql, half2 ** x_dm, int ** x_qh, int ** x_sc) {
    GGML_UNUSED(x_qh);

    __shared__ int   tile_x_ql[mmq_y * (WARP_SIZE)       + mmq_y];
    __shared__ half2 tile_x_dm[mmq_y * (WARP_SIZE/QI2_K) + mmq_y/QI2_K];
    __shared__ int   tile_x_sc[mmq_y * (WARP_SIZE/4)     + mmq_y/4];

    *x_ql = tile_x_ql;
    *x_dm = tile_x_dm;
    *x_sc = tile_x_sc;
}

template <int mmq_y, int nwarps, bool need_check> static __device__ __forceinline__ void load_tiles_q2_K(
    const void * __restrict__ vx, int * __restrict__ x_ql, half2 * __restrict__ x_dm, int * __restrict__ x_qh,
    int * __restrict__ x_sc, const int & i_offset, const int & i_max, const int & k, const int & blocks_per_row) {
    GGML_UNUSED(x_qh);

    GGML_CUDA_ASSUME(i_offset >= 0);
    GGML_CUDA_ASSUME(i_offset <  nwarps);
    GGML_CUDA_ASSUME(k >= 0);
    GGML_CUDA_ASSUME(k <  WARP_SIZE);

    const int kbx  = k / QI2_K;
    const int kqsx = k % QI2_K;

    const block_q2_K * bx0 = (const block_q2_K *) vx;

#pragma unroll
    for (int i0 = 0; i0 < mmq_y; i0 += nwarps) {
        int i = i0 + i_offset;

        if (need_check) {
            i = min(i, i_max);
        }

        const block_q2_K * bxi = bx0 + i*blocks_per_row + kbx;

        x_ql[i * (WARP_SIZE + 1) + k] = get_int_from_uint8_aligned(bxi->qs, kqsx);
    }

    const int blocks_per_tile_x_row = WARP_SIZE / QI2_K;
    const int kbxd = k % blocks_per_tile_x_row;

#pragma unroll
    for (int i0 = 0; i0 < mmq_y; i0 += nwarps * QI2_K) {
        int i = (i0 + i_offset * QI2_K + k / blocks_per_tile_x_row) % mmq_y;

        if (need_check) {
            i = min(i, i_max);
        }

        const block_q2_K * bxi = bx0 + i*blocks_per_row + kbxd;

        x_dm[i * (WARP_SIZE/QI2_K) + i / QI2_K + kbxd] = bxi->dm;
    }

#pragma unroll
    for (int i0 = 0; i0 < mmq_y; i0 += nwarps * 4) {
        int i = i0 + i_offset * 4 + k / (WARP_SIZE/4);

        if (need_check) {
            i = min(i, i_max);
        }

        const block_q2_K * bxi = bx0 + i*blocks_per_row + (k % (WARP_SIZE/4)) / (QI2_K/4);

        x_sc[i * (WARP_SIZE/4) + i / 4 + k % (WARP_SIZE/4)] = get_int_from_uint8_aligned(bxi->scales, k % (QI2_K/4));
    }
}

static __device__ __forceinline__ float vec_dot_q2_K_q8_1_mul_mat(
    const int * __restrict__ x_ql, const half2 * __restrict__ x_dm, const int * __restrict__ x_qh, const int * __restrict__ x_sc,
    const int * __restrict__ y_qs, const half2 * __restrict__ y_ds, const int & i, const int & j, const int & k) {
    GGML_UNUSED(x_qh);

    const int kbx = k / QI2_K;
    const int ky  = (k % QI2_K) * QR2_K;
    const float * y_df = (const float *) y_ds;

    int v[QR2_K*VDR_Q2_K_Q8_1_MMQ];

    const int kqsx = i * (WARP_SIZE + 1) + kbx*QI2_K + (QI2_K/2) * (ky/(2*QI2_K)) + ky % (QI2_K/2);
    const int shift = 2 * ((ky % (2*QI2_K)) / (QI2_K/2));

#pragma unroll
    for (int l = 0; l < QR2_K*VDR_Q2_K_Q8_1_MMQ; ++l) {
        v[l] = (x_ql[kqsx + l] >> shift) & 0x03030303;
    }

    const uint8_t * scales = ((const uint8_t *) &x_sc[i * (WARP_SIZE/4) + i/4 + kbx*4]) + ky/4;

    const int index_y = j * WARP_SIZE + (QR2_K*k) % WARP_SIZE;
    return vec_dot_q2_K_q8_1_impl_mmq(v, &y_qs[index_y], scales, x_dm[i * (WARP_SIZE/QI2_K) + i/QI2_K + kbx], y_df[index_y/QI8_1]);
}

template <int mmq_y> static __device__ __forceinline__ void allocate_tiles_q3_K(int ** x_ql, half2 ** x_dm, int ** x_qh, int ** x_sc) {

    __shared__ int   tile_x_ql[mmq_y * (WARP_SIZE)       + mmq_y];
    __shared__ half2 tile_x_dm[mmq_y * (WARP_SIZE/QI3_K) + mmq_y/QI3_K];
    __shared__ int   tile_x_qh[mmq_y * (WARP_SIZE/2)     + mmq_y/2];
    __shared__ int   tile_x_sc[mmq_y * (WARP_SIZE/4)     + mmq_y/4];

    *x_ql = tile_x_ql;
    *x_dm = tile_x_dm;
    *x_qh = tile_x_qh;
    *x_sc = tile_x_sc;
}

template <int mmq_y, int nwarps, bool need_check> static __device__ __forceinline__ void load_tiles_q3_K(
    const void * __restrict__ vx, int * __restrict__ x_ql, half2 * __restrict__ x_dm, int * __restrict__ x_qh,
    int * __restrict__ x_sc, const int & i_offset, const int & i_max, const int & k, const int & blocks_per_row) {

    GGML_CUDA_ASSUME(i_offset >= 0);
    GGML_CUDA_ASSUME(i_offset <  nwarps);
    GGML_CUDA_ASSUME(k >= 0);
    GGML_CUDA_ASSUME(k <  WARP_SIZE);

    const int kbx  = k / QI3_K;
    const int kqsx = k % QI3_K;

    const block_q3_K * bx0 = (const block_q3_K *) vx;

#pragma unroll
    for (int i0 = 0; i0 < mmq_y; i0 += nwarps) {
        int i = i0 + i_offset;

        if (need_check) {
            i = min(i, i_max);
        }

        const block_q3_K * bxi = bx0 + i*blocks_per_row + kbx;

        x_ql[i * (WARP_SIZE + 1) + k] = get_int_from_uint8(bxi->qs, kqsx);
    }

    const int blocks_per_tile_x_row = WARP_SIZE / QI3_K;
    const int kbxd = k % blocks_per_tile_x_row;
    float * x_dmf = (float *) x_dm;

#pragma unroll
    for (int i0 = 0; i0 < mmq_y; i0 += nwarps * QI3_K) {
        int i = (i0 + i_offset * QI3_K + k / blocks_per_tile_x_row) % mmq_y;

        if (need_check) {
            i = min(i, i_max);
        }

        const block_q3_K * bxi = bx0 + i*blocks_per_row + kbxd;

        x_dmf[i * (WARP_SIZE/QI3_K) + i / QI3_K + kbxd] = bxi->d;
    }

#pragma unroll
    for (int i0 = 0; i0 < mmq_y; i0 += nwarps * 2) {
        int i = i0 + i_offset * 2 + k / (WARP_SIZE/2);

        if (need_check) {
            i = min(i, i_max);
        }

        const block_q3_K * bxi = bx0 + i*blocks_per_row + (k % (WARP_SIZE/2)) / (QI3_K/2);

        // invert the mask with ~ so that a 0/1 results in 4/0 being subtracted
        x_qh[i * (WARP_SIZE/2) + i / 2 + k % (WARP_SIZE/2)] = ~get_int_from_uint8(bxi->hmask, k % (QI3_K/2));
    }

#pragma unroll
    for (int i0 = 0; i0 < mmq_y; i0 += nwarps * 4) {
        int i = i0 + i_offset * 4 + k / (WARP_SIZE/4);

        if (need_check) {
            i = min(i, i_max);
        }

        const block_q3_K * bxi = bx0 + i*blocks_per_row + (k % (WARP_SIZE/4)) / (QI3_K/4);

        const int ksc = k % (QI3_K/4);

        const int ksc_low = ksc % (QI3_K/8);
        const int shift_low = 4 * (ksc / (QI3_K/8));
        const int sc_low = (get_int_from_uint8(bxi->scales, ksc_low) >> shift_low) & 0x0F0F0F0F;

        const int ksc_high = QI3_K/8;
        const int shift_high = 2 * ksc;
        const int sc_high = ((get_int_from_uint8(bxi->scales, ksc_high) >> shift_high) << 4) & 0x30303030;

        const int sc = __vsubss4(sc_low | sc_high, 0x20202020);

        x_sc[i * (WARP_SIZE/4) + i / 4 + k % (WARP_SIZE/4)] = sc;
    }
}

static __device__ __forceinline__ float vec_dot_q3_K_q8_1_mul_mat(
    const int * __restrict__ x_ql, const half2 * __restrict__ x_dm, const int * __restrict__ x_qh, const int * __restrict__ x_sc,
    const int * __restrict__ y_qs, const half2 * __restrict__ y_ds, const int & i, const int & j, const int & k) {

    const int kbx  = k / QI3_K;
    const int ky  = (k % QI3_K) * QR3_K;
    const float * x_dmf = (const float *) x_dm;
    const float * y_df  = (const float *) y_ds;

    const int8_t * scales = ((const int8_t *) (x_sc + i * (WARP_SIZE/4) + i/4 + kbx*4)) + ky/4;

    int v[QR3_K*VDR_Q3_K_Q8_1_MMQ];

#pragma unroll
    for (int l = 0; l < QR3_K*VDR_Q3_K_Q8_1_MMQ; ++l) {
        const int kqsx = i * (WARP_SIZE + 1) + kbx*QI3_K + (QI3_K/2) * (ky/(2*QI3_K)) + ky % (QI3_K/2);
        const int shift = 2 * ((ky % 32) / 8);
        const int vll = (x_ql[kqsx + l] >> shift) & 0x03030303;

        const int vh = x_qh[i * (WARP_SIZE/2) + i/2 + kbx * (QI3_K/2) + (ky+l)%8] >> ((ky+l) / 8);
        const int vlh = (vh << 2) & 0x04040404;

        v[l] = __vsubss4(vll, vlh);
    }

    const int index_y = j * WARP_SIZE + (k*QR3_K) % WARP_SIZE;
    return vec_dot_q3_K_q8_1_impl_mmq(v, &y_qs[index_y], scales, x_dmf[i * (WARP_SIZE/QI3_K) + i/QI3_K + kbx], y_df[index_y/QI8_1]);
}

template <int mmq_y> static __device__ __forceinline__ void allocate_tiles_q4_K(int ** x_ql, half2 ** x_dm, int ** x_qh, int ** x_sc) {
    GGML_UNUSED(x_qh);

    __shared__ int   tile_x_ql[mmq_y * (WARP_SIZE)       + mmq_y];
    __shared__ half2 tile_x_dm[mmq_y * (WARP_SIZE/QI4_K) + mmq_y/QI4_K];
    __shared__ int   tile_x_sc[mmq_y * (WARP_SIZE/8)     + mmq_y/8];

    *x_ql = tile_x_ql;
    *x_dm = tile_x_dm;
    *x_sc = tile_x_sc;
}

template <int mmq_y, int nwarps, bool need_check> static __device__ __forceinline__ void load_tiles_q4_K(
    const void * __restrict__ vx, int * __restrict__ x_ql, half2 * __restrict__ x_dm, int * __restrict__ x_qh,
    int * __restrict__ x_sc, const int & i_offset, const int & i_max, const int & k, const int & blocks_per_row) {
    GGML_UNUSED(x_qh);

    GGML_CUDA_ASSUME(i_offset >= 0);
    GGML_CUDA_ASSUME(i_offset <  nwarps);
    GGML_CUDA_ASSUME(k >= 0);
    GGML_CUDA_ASSUME(k <  WARP_SIZE);

    const int kbx  = k / QI4_K; // == 0 if QK_K == 256
    const int kqsx = k % QI4_K; // == k if QK_K == 256

    const block_q4_K * bx0 = (const block_q4_K *) vx;

#pragma unroll
    for (int i0 = 0; i0 < mmq_y; i0 += nwarps) {
        int i = i0 + i_offset;

        if (need_check) {
            i = min(i, i_max);
        }

        const block_q4_K * bxi = bx0 + i*blocks_per_row + kbx;

        x_ql[i * (WARP_SIZE + 1) + k] = get_int_from_uint8_aligned(bxi->qs, kqsx);
    }

    const int blocks_per_tile_x_row = WARP_SIZE / QI4_K; // == 1 if QK_K == 256
    const int kbxd = k % blocks_per_tile_x_row;          // == 0 if QK_K == 256

#pragma unroll
    for (int i0 = 0; i0 < mmq_y; i0 += nwarps * QI4_K) {
        int i = (i0 + i_offset * QI4_K + k / blocks_per_tile_x_row) % mmq_y;

        if (need_check) {
            i = min(i, i_max);
        }

        const block_q4_K * bxi = bx0 + i*blocks_per_row + kbxd;

        x_dm[i * (WARP_SIZE/QI4_K) + i / QI4_K + kbxd] = bxi->dm;
    }

#pragma unroll
    for (int i0 = 0; i0 < mmq_y; i0 += nwarps * 8) {
        int i = (i0 + i_offset * 8 + k / (WARP_SIZE/8)) % mmq_y;

        if (need_check) {
            i = min(i, i_max);
        }

        const block_q4_K * bxi = bx0 + i*blocks_per_row + (k % (WARP_SIZE/8)) / (QI4_K/8);

        const int * scales = (const int *) bxi->scales;

        const int ksc = k % (WARP_SIZE/8);

        // scale arrangement after the following two lines: sc0,...,sc3, sc4,...,sc7, m0,...,m3, m4,...,m8
        int scales8 = (scales[(ksc%2) + (ksc!=0)] >> (4 * (ksc & (ksc/2)))) & 0x0F0F0F0F; // lower 4 bits
        scales8    |= (scales[ksc/2]              >> (2 * (ksc % 2)))       & 0x30303030; // upper 2 bits

        x_sc[i * (WARP_SIZE/8) + i / 8 + ksc] = scales8;
    }
}

static __device__ __forceinline__ float vec_dot_q4_K_q8_1_mul_mat(
    const int * __restrict__ x_ql, const half2 * __restrict__ x_dm, const int * __restrict__ x_qh, const int * __restrict__ x_sc,
    const int * __restrict__ y_qs, const half2 * __restrict__ y_ds, const int & i, const int & j, const int & k) {
    GGML_UNUSED(x_qh);

    const uint8_t * sc = ((const uint8_t *) &x_sc[i * (WARP_SIZE/8) + i/8 + k/16]) + 2*((k % 16) / 8);

    const int index_y = j * WARP_SIZE + (QR4_K*k) % WARP_SIZE;
    return vec_dot_q4_K_q8_1_impl_mmq(&x_ql[i * (WARP_SIZE + 1) + k], &y_qs[index_y], sc, sc+8,
                                      x_dm[i * (WARP_SIZE/QI4_K) + i/QI4_K], &y_ds[index_y/QI8_1]);
}

template <int mmq_y> static __device__ __forceinline__ void allocate_tiles_q5_K(int ** x_ql, half2 ** x_dm, int ** x_qh, int ** x_sc) {
    GGML_UNUSED(x_qh);

    __shared__ int   tile_x_ql[mmq_y * (2*WARP_SIZE)     + mmq_y];
    __shared__ half2 tile_x_dm[mmq_y * (WARP_SIZE/QI5_K) + mmq_y/QI5_K];
    __shared__ int   tile_x_sc[mmq_y * (WARP_SIZE/8)     + mmq_y/8];

    *x_ql = tile_x_ql;
    *x_dm = tile_x_dm;
    *x_sc = tile_x_sc;
}

template <int mmq_y, int nwarps, bool need_check> static __device__ __forceinline__ void load_tiles_q5_K(
    const void * __restrict__ vx, int * __restrict__ x_ql, half2 * __restrict__ x_dm, int * __restrict__ x_qh,
    int * __restrict__ x_sc, const int & i_offset, const int & i_max, const int & k, const int & blocks_per_row) {
    GGML_UNUSED(x_qh);

    GGML_CUDA_ASSUME(i_offset >= 0);
    GGML_CUDA_ASSUME(i_offset <  nwarps);
    GGML_CUDA_ASSUME(k >= 0);
    GGML_CUDA_ASSUME(k <  WARP_SIZE);

    const int kbx  = k / QI5_K; // == 0 if QK_K == 256
    const int kqsx = k % QI5_K; // == k if QK_K == 256

    const block_q5_K * bx0 = (const block_q5_K *) vx;

#pragma unroll
    for (int i0 = 0; i0 < mmq_y; i0 += nwarps) {
        int i = i0 + i_offset;

        if (need_check) {
            i = min(i, i_max);
        }

        const block_q5_K * bxi = bx0 + i*blocks_per_row + kbx;
        const int ky = QR5_K*kqsx;

        const int ql = get_int_from_uint8_aligned(bxi->qs, kqsx);
        const int ql0 = (ql >> 0) & 0x0F0F0F0F;
        const int ql1 = (ql >> 4) & 0x0F0F0F0F;

        const int qh = get_int_from_uint8_aligned(bxi->qh, kqsx % (QI5_K/4));
        const int qh0 = ((qh >> (2 * (kqsx / (QI5_K/4)) + 0)) << 4) & 0x10101010;
        const int qh1 = ((qh >> (2 * (kqsx / (QI5_K/4)) + 1)) << 4) & 0x10101010;

        const int kq0 = ky - ky % (QI5_K/2) + k % (QI5_K/4) + 0;
        const int kq1 = ky - ky % (QI5_K/2) + k % (QI5_K/4) + (QI5_K/4);

        x_ql[i * (2*WARP_SIZE + 1) + kq0] = ql0 | qh0;
        x_ql[i * (2*WARP_SIZE + 1) + kq1] = ql1 | qh1;
    }

    const int blocks_per_tile_x_row = WARP_SIZE / QI5_K; // == 1 if QK_K == 256
    const int kbxd = k % blocks_per_tile_x_row;          // == 0 if QK_K == 256

#pragma unroll
    for (int i0 = 0; i0 < mmq_y; i0 += nwarps * QI5_K) {
        int i = (i0 + i_offset * QI5_K + k / blocks_per_tile_x_row) % mmq_y;

        if (need_check) {
            i = min(i, i_max);
        }

        const block_q5_K * bxi = bx0 + i*blocks_per_row + kbxd;

        x_dm[i * (WARP_SIZE/QI5_K) + i / QI5_K + kbxd] = bxi->dm;
    }

#pragma unroll
    for (int i0 = 0; i0 < mmq_y; i0 += nwarps * 8) {
        int i = (i0 + i_offset * 8 + k / (WARP_SIZE/8)) % mmq_y;

        if (need_check) {
            i = min(i, i_max);
        }

        const block_q5_K * bxi = bx0 + i*blocks_per_row + (k % (WARP_SIZE/8)) / (QI5_K/8);

        const int * scales = (const int *) bxi->scales;

        const int ksc = k % (WARP_SIZE/8);

        // scale arrangement after the following two lines: sc0,...,sc3, sc4,...,sc7, m0,...,m3, m4,...,m8
        int scales8 = (scales[(ksc%2) + (ksc!=0)] >> (4 * (ksc & (ksc/2)))) & 0x0F0F0F0F; // lower 4 bits
        scales8    |= (scales[ksc/2]              >> (2 * (ksc % 2)))       & 0x30303030; // upper 2 bits

        x_sc[i * (WARP_SIZE/8) + i / 8 + ksc] = scales8;
    }
}

static __device__ __forceinline__ float vec_dot_q5_K_q8_1_mul_mat(
    const int * __restrict__ x_ql, const half2 * __restrict__ x_dm, const int * __restrict__ x_qh, const int * __restrict__ x_sc,
    const int * __restrict__ y_qs, const half2 * __restrict__ y_ds, const int & i, const int & j, const int & k) {
    GGML_UNUSED(x_qh);

    const uint8_t * sc = ((const uint8_t *) &x_sc[i * (WARP_SIZE/8) + i/8 + k/16]) + 2 * ((k % 16) / 8);

    const int index_x = i * (QR5_K*WARP_SIZE + 1) +  QR5_K*k;
    const int index_y = j * WARP_SIZE             + (QR5_K*k) % WARP_SIZE;
    return vec_dot_q5_K_q8_1_impl_mmq(&x_ql[index_x], &y_qs[index_y], sc, sc+8,
                                      x_dm[i * (WARP_SIZE/QI5_K) + i/QI5_K], &y_ds[index_y/QI8_1]);
}

template <int mmq_y> static __device__ __forceinline__ void allocate_tiles_q6_K(int ** x_ql, half2 ** x_dm, int ** x_qh, int ** x_sc) {
    GGML_UNUSED(x_qh);

    __shared__ int   tile_x_ql[mmq_y * (2*WARP_SIZE)     + mmq_y];
    __shared__ half2 tile_x_dm[mmq_y * (WARP_SIZE/QI6_K) + mmq_y/QI6_K];
    __shared__ int   tile_x_sc[mmq_y * (WARP_SIZE/8)     + mmq_y/8];

    *x_ql = tile_x_ql;
    *x_dm = tile_x_dm;
    *x_sc = tile_x_sc;
}

template <int mmq_y, int nwarps, bool need_check> static __device__ __forceinline__ void load_tiles_q6_K(
    const void * __restrict__ vx, int * __restrict__ x_ql, half2 * __restrict__ x_dm, int * __restrict__ x_qh,
    int * __restrict__ x_sc, const int & i_offset, const int & i_max, const int & k, const int & blocks_per_row) {
    GGML_UNUSED(x_qh);

    GGML_CUDA_ASSUME(i_offset >= 0);
    GGML_CUDA_ASSUME(i_offset <  nwarps);
    GGML_CUDA_ASSUME(k >= 0);
    GGML_CUDA_ASSUME(k <  WARP_SIZE);

    const int kbx  = k / QI6_K; // == 0 if QK_K == 256
    const int kqsx = k % QI6_K; // == k if QK_K == 256

    const block_q6_K * bx0 = (const block_q6_K *) vx;

#pragma unroll
    for (int i0 = 0; i0 < mmq_y; i0 += nwarps) {
        int i = i0 + i_offset;

        if (need_check) {
            i = min(i, i_max);
        }

        const block_q6_K * bxi = bx0 + i*blocks_per_row + kbx;
        const int ky = QR6_K*kqsx;

        const int ql = get_int_from_uint8(bxi->ql, kqsx);
        const int ql0 = (ql >> 0) & 0x0F0F0F0F;
        const int ql1 = (ql >> 4) & 0x0F0F0F0F;

        const int qh = get_int_from_uint8(bxi->qh, (QI6_K/4) * (kqsx / (QI6_K/2)) + kqsx % (QI6_K/4));
        const int qh0 = ((qh >> (2 * ((kqsx % (QI6_K/2)) / (QI6_K/4)))) << 4) & 0x30303030;
        const int qh1 =  (qh >> (2 * ((kqsx % (QI6_K/2)) / (QI6_K/4))))       & 0x30303030;

        const int kq0 = ky - ky % QI6_K + k % (QI6_K/2) + 0;
        const int kq1 = ky - ky % QI6_K + k % (QI6_K/2) + (QI6_K/2);

        x_ql[i * (2*WARP_SIZE + 1) + kq0] = __vsubss4(ql0 | qh0, 0x20202020);
        x_ql[i * (2*WARP_SIZE + 1) + kq1] = __vsubss4(ql1 | qh1, 0x20202020);
    }

    const int blocks_per_tile_x_row = WARP_SIZE / QI6_K; // == 1 if QK_K == 256
    const int kbxd = k % blocks_per_tile_x_row;          // == 0 if QK_K == 256
    float * x_dmf = (float *) x_dm;

#pragma unroll
    for (int i0 = 0; i0 < mmq_y; i0 += nwarps * QI6_K) {
        int i = (i0 + i_offset * QI6_K + k / blocks_per_tile_x_row) % mmq_y;

        if (need_check) {
            i = min(i, i_max);
        }

        const block_q6_K * bxi = bx0 + i*blocks_per_row + kbxd;

        x_dmf[i * (WARP_SIZE/QI6_K) + i / QI6_K + kbxd] = bxi->d;
    }

#pragma unroll
    for (int i0 = 0; i0 < mmq_y; i0 += nwarps * 8) {
        int i = (i0 + i_offset * 8 + k / (WARP_SIZE/8)) % mmq_y;

        if (need_check) {
            i = min(i, i_max);
        }

        const block_q6_K * bxi = bx0 + i*blocks_per_row + (k % (WARP_SIZE/8)) / 4;

        x_sc[i * (WARP_SIZE/8) + i / 8 + k % (WARP_SIZE/8)] = get_int_from_int8(bxi->scales, k % (QI6_K/8));
    }
}

static __device__ __forceinline__ float vec_dot_q6_K_q8_1_mul_mat(
    const int * __restrict__ x_ql, const half2 * __restrict__ x_dm, const int * __restrict__ x_qh, const int * __restrict__ x_sc,
    const int * __restrict__ y_qs, const half2 * __restrict__ y_ds, const int & i, const int & j, const int & k) {
    GGML_UNUSED(x_qh);

    const float * x_dmf = (const float *) x_dm;
    const float * y_df  = (const float *) y_ds;

    const int8_t * sc = ((const int8_t *) &x_sc[i * (WARP_SIZE/8) + i/8 + k/8]);

    const int index_x = i * (QR6_K*WARP_SIZE + 1) +  QR6_K*k;
    const int index_y = j * WARP_SIZE             + (QR6_K*k) % WARP_SIZE;
    return vec_dot_q6_K_q8_1_impl_mmq(&x_ql[index_x], &y_qs[index_y], sc, x_dmf[i * (WARP_SIZE/QI6_K) + i/QI6_K], &y_df[index_y/QI8_1]);
}

template <int qk, int qr, int qi, bool need_sum, typename block_q_t, int mmq_x, int mmq_y, int nwarps,
              allocate_tiles_cuda_t allocate_tiles, load_tiles_cuda_t load_tiles, int vdr, vec_dot_q_mul_mat_cuda_t vec_dot>
static __device__ __forceinline__ void mul_mat_q(
    const void * __restrict__ vx, const void * __restrict__ vy, float * __restrict__ dst,
    const int ncols_x, const int nrows_x, const int ncols_y, const int nrows_y, const int nrows_dst) {

    const block_q_t  * x = (const block_q_t  *) vx;
    const block_q8_1 * y = (const block_q8_1 *) vy;

    const int blocks_per_row_x = ncols_x / qk;
    const int blocks_per_col_y = nrows_y / QK8_1;
    const int blocks_per_warp = WARP_SIZE / qi;

    const int & ncols_dst = ncols_y;

    const int row_dst_0 = blockIdx.x*mmq_y;
    const int & row_x_0 = row_dst_0;

    const int col_dst_0 = blockIdx.y*mmq_x;
    const int & col_y_0 = col_dst_0;

    int   * tile_x_ql = nullptr;
    half2 * tile_x_dm = nullptr;
    int   * tile_x_qh = nullptr;
    int   * tile_x_sc = nullptr;

    allocate_tiles(&tile_x_ql, &tile_x_dm, &tile_x_qh, &tile_x_sc);

    __shared__ int    tile_y_qs[mmq_x * WARP_SIZE];
    __shared__ half2  tile_y_ds[mmq_x * WARP_SIZE/QI8_1];

    float sum[mmq_y/WARP_SIZE][mmq_x/nwarps] = {{0.0f}};

    for (int ib0 = 0; ib0 < blocks_per_row_x; ib0 += blocks_per_warp) {

        load_tiles(x + row_x_0*blocks_per_row_x + ib0, tile_x_ql, tile_x_dm, tile_x_qh, tile_x_sc,
                   threadIdx.y, nrows_x-row_x_0-1, threadIdx.x, blocks_per_row_x);

#pragma unroll
        for (int ir = 0; ir < qr; ++ir) {
            const int kqs = ir*WARP_SIZE + threadIdx.x;
            const int kbxd = kqs / QI8_1;

#pragma unroll
            for (int i = 0; i < mmq_x; i += nwarps) {
                const int col_y_eff = min(col_y_0 + threadIdx.y + i, ncols_y-1); // to prevent out-of-bounds memory accesses

                const block_q8_1 * by0 = &y[col_y_eff*blocks_per_col_y + ib0 * (qk/QK8_1) + kbxd];

                const int index_y = (threadIdx.y + i) * WARP_SIZE + kqs % WARP_SIZE;
                tile_y_qs[index_y] = get_int_from_int8_aligned(by0->qs, threadIdx.x % QI8_1);
            }

#pragma unroll
            for (int ids0 = 0; ids0 < mmq_x; ids0 += nwarps * QI8_1) {
                const int ids = (ids0 + threadIdx.y * QI8_1 + threadIdx.x / (WARP_SIZE/QI8_1)) % mmq_x;
                const int kby = threadIdx.x % (WARP_SIZE/QI8_1);
                const int col_y_eff = min(col_y_0 + ids, ncols_y-1);

                // if the sum is not needed it's faster to transform the scale to f32 ahead of time
                const half2 * dsi_src = &y[col_y_eff*blocks_per_col_y + ib0 * (qk/QK8_1) + ir*(WARP_SIZE/QI8_1) + kby].ds;
                half2       * dsi_dst = &tile_y_ds[ids * (WARP_SIZE/QI8_1) + kby];
                if (need_sum) {
                    *dsi_dst = *dsi_src;
                } else {
                    float * dfi_dst = (float *) dsi_dst;
                    *dfi_dst = __low2float(*dsi_src);
                }
            }

            __syncthreads();

// #pragma unroll // unrolling this loop causes too much register pressure
            for (int k = ir*WARP_SIZE/qr; k < (ir+1)*WARP_SIZE/qr; k += vdr) {
#pragma unroll
                for (int j = 0; j < mmq_x; j += nwarps) {
#pragma unroll
                    for (int i = 0; i < mmq_y; i += WARP_SIZE) {
                        sum[i/WARP_SIZE][j/nwarps] += vec_dot(
                            tile_x_ql, tile_x_dm, tile_x_qh, tile_x_sc, tile_y_qs, tile_y_ds,
                            threadIdx.x + i, threadIdx.y + j, k);
                    }
                }
            }

            __syncthreads();
        }
    }

#pragma unroll
    for (int j = 0; j < mmq_x; j += nwarps) {
        const int col_dst = col_dst_0 + j + threadIdx.y;

        if (col_dst >= ncols_dst) {
            return;
        }

#pragma unroll
        for (int i = 0; i < mmq_y; i += WARP_SIZE) {
            const int row_dst = row_dst_0 + threadIdx.x + i;

            if (row_dst >= nrows_dst) {
                continue;
            }

            dst[col_dst*nrows_dst + row_dst] = sum[i/WARP_SIZE][j/nwarps];
        }
    }
}

static constexpr __device__ mmq_arch_config_t get_arch_config_device(mmq_config_t mmq_config) {

#if defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__)

#if defined(RDNA3) || defined(RDNA2)
    return mmq_config.rdna2;
#else
    return mmq_config.rdna1;
#endif // defined(RDNA3) || defined(RDNA2)

#else

#if __CUDA_ARCH__ >= CC_VOLTA
    return mmq_config.ampere;
#else
    return mmq_config.pascal;
#endif // __CUDA_ARCH__ >= CC_VOLTA

#endif // defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__)
}

template <bool need_check> static __global__ void
#if defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__)
#if defined(RDNA3) || defined(RDNA2)
    __launch_bounds__(WARP_SIZE*MMQ_CONFIG_Q4_0.rdna2.nwarps, 2)
#endif // defined(RDNA3) || defined(RDNA2)
#endif // defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__)
    mul_mat_q4_0(
    const void * __restrict__ vx, const void * __restrict__ vy, float * __restrict__ dst,
    const int ncols_x, const int nrows_x, const int ncols_y, const int nrows_y, const int nrows_dst) {

#if __CUDA_ARCH__ >= MIN_CC_DP4A
    constexpr mmq_arch_config_t arch_config = get_arch_config_device(MMQ_CONFIG_Q4_0);

    mul_mat_q<QK4_0, QR4_0, QI4_0, true, block_q4_0, arch_config.x, arch_config.y, arch_config.nwarps, allocate_tiles_q4_0<arch_config.y>,
        load_tiles_q4_0<arch_config.y, arch_config.nwarps, need_check>, VDR_Q4_0_Q8_1_MMQ, vec_dot_q4_0_q8_1_mul_mat>
        (vx, vy, dst, ncols_x, nrows_x, ncols_y, nrows_y, nrows_dst);
#else
    GGML_UNUSED(get_arch_config_device);
    GGML_UNUSED(vec_dot_q4_0_q8_1_mul_mat);
    NO_DEVICE_CODE;
#endif // __CUDA_ARCH__ >= MIN_CC_DP4A
}

template <bool need_check> static __global__ void
#if defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__)
#if defined(RDNA3) || defined(RDNA2)
    __launch_bounds__(WARP_SIZE*MMQ_CONFIG_Q4_1.rdna2.nwarps, 2)
#endif // defined(RDNA3) || defined(RDNA2)
#elif __CUDA_ARCH__ < CC_VOLTA
    __launch_bounds__(WARP_SIZE*MMQ_CONFIG_Q4_1.pascal.nwarps, 2)
#endif // __CUDA_ARCH__ < CC_VOLTA
    mul_mat_q4_1(
    const void * __restrict__ vx, const void * __restrict__ vy, float * __restrict__ dst,
    const int ncols_x, const int nrows_x, const int ncols_y, const int nrows_y, const int nrows_dst) {

#if __CUDA_ARCH__ >= MIN_CC_DP4A
    constexpr mmq_arch_config_t arch_config = get_arch_config_device(MMQ_CONFIG_Q4_1);

    mul_mat_q<QK4_1, QR4_1, QI4_1, true, block_q4_1, arch_config.x, arch_config.y, arch_config.nwarps, allocate_tiles_q4_1<arch_config.y>,
        load_tiles_q4_1<arch_config.y, arch_config.nwarps, need_check>, VDR_Q4_1_Q8_1_MMQ, vec_dot_q4_1_q8_1_mul_mat>
        (vx, vy, dst, ncols_x, nrows_x, ncols_y, nrows_y, nrows_dst);
#else
    GGML_UNUSED(get_arch_config_device);
    GGML_UNUSED(vec_dot_q4_1_q8_1_mul_mat);
    NO_DEVICE_CODE;
#endif // __CUDA_ARCH__ >= MIN_CC_DP4A
}

template <bool need_check> static __global__ void
#if defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__)
#if defined(RDNA3) || defined(RDNA2)
    __launch_bounds__(WARP_SIZE*MMQ_CONFIG_Q5_0.rdna2.nwarps, 2)
#endif // defined(RDNA3) || defined(RDNA2)
#endif // defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__)
    mul_mat_q5_0(
    const void * __restrict__ vx, const void * __restrict__ vy, float * __restrict__ dst,
    const int ncols_x, const int nrows_x, const int ncols_y, const int nrows_y, const int nrows_dst) {

#if __CUDA_ARCH__ >= MIN_CC_DP4A
    constexpr mmq_arch_config_t arch_config = get_arch_config_device(MMQ_CONFIG_Q5_0);

    mul_mat_q<QK5_0, QR5_0, QI5_0, false, block_q5_0, arch_config.x, arch_config.y, arch_config.nwarps, allocate_tiles_q5_0<arch_config.y>,
        load_tiles_q5_0<arch_config.y, arch_config.nwarps, need_check>, VDR_Q5_0_Q8_1_MMQ, vec_dot_q5_0_q8_1_mul_mat>
        (vx, vy, dst, ncols_x, nrows_x, ncols_y, nrows_y, nrows_dst);
#else
    GGML_UNUSED(get_arch_config_device);
    GGML_UNUSED(vec_dot_q5_0_q8_1_mul_mat);
    NO_DEVICE_CODE;
#endif // __CUDA_ARCH__ >= MIN_CC_DP4A
}

template <bool need_check> static __global__ void
#if defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__)
#if defined(RDNA3) || defined(RDNA2)
    __launch_bounds__(WARP_SIZE*MMQ_CONFIG_Q5_1.rdna2.nwarps, 2)
#endif // defined(RDNA3) || defined(RDNA2)
#endif // defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__)
mul_mat_q5_1(
    const void * __restrict__ vx, const void * __restrict__ vy, float * __restrict__ dst,
    const int ncols_x, const int nrows_x, const int ncols_y, const int nrows_y, const int nrows_dst) {

#if __CUDA_ARCH__ >= MIN_CC_DP4A
    constexpr mmq_arch_config_t arch_config = get_arch_config_device(MMQ_CONFIG_Q5_1);

    mul_mat_q<QK5_1, QR5_1, QI5_1, true, block_q5_1, arch_config.x, arch_config.y, arch_config.nwarps, allocate_tiles_q5_1<arch_config.y>,
        load_tiles_q5_1<arch_config.y, arch_config.nwarps, need_check>, VDR_Q5_1_Q8_1_MMQ, vec_dot_q5_1_q8_1_mul_mat>
        (vx, vy, dst, ncols_x, nrows_x, ncols_y, nrows_y, nrows_dst);
#else
    GGML_UNUSED(get_arch_config_device);
    GGML_UNUSED(vec_dot_q5_1_q8_1_mul_mat);
    NO_DEVICE_CODE;
#endif // __CUDA_ARCH__ >= MIN_CC_DP4A
}

template <bool need_check> static __global__ void
#if defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__)
#if defined(RDNA3) || defined(RDNA2)
    __launch_bounds__(WARP_SIZE*MMQ_CONFIG_Q8_0.rdna2.nwarps, 2)
#endif // defined(RDNA3) || defined(RDNA2)
#endif // defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__)
    mul_mat_q8_0(
    const void * __restrict__ vx, const void * __restrict__ vy, float * __restrict__ dst,
    const int ncols_x, const int nrows_x, const int ncols_y, const int nrows_y, const int nrows_dst) {

#if __CUDA_ARCH__ >= MIN_CC_DP4A
    constexpr mmq_arch_config_t arch_config = get_arch_config_device(MMQ_CONFIG_Q8_0);

    mul_mat_q<QK8_0, QR8_0, QI8_0, false, block_q8_0, arch_config.x, arch_config.y, arch_config.nwarps, allocate_tiles_q8_0<arch_config.y>,
        load_tiles_q8_0<arch_config.y, arch_config.nwarps, need_check>, VDR_Q8_0_Q8_1_MMQ, vec_dot_q8_0_q8_1_mul_mat>
        (vx, vy, dst, ncols_x, nrows_x, ncols_y, nrows_y, nrows_dst);
#else
    GGML_UNUSED(get_arch_config_device);
    GGML_UNUSED(vec_dot_q8_0_q8_1_mul_mat);
    NO_DEVICE_CODE;
#endif // __CUDA_ARCH__ >= MIN_CC_DP4A
}

template <bool need_check> static __global__ void
#if defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__)
#if defined(RDNA3) || defined(RDNA2)
    __launch_bounds__(WARP_SIZE*MMQ_CONFIG_Q2_K.rdna2.nwarps, 2)
#endif // defined(RDNA3) || defined(RDNA2)
#endif // defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__)
mul_mat_q2_K(
    const void * __restrict__ vx, const void * __restrict__ vy, float * __restrict__ dst,
    const int ncols_x, const int nrows_x, const int ncols_y, const int nrows_y, const int nrows_dst) {

#if __CUDA_ARCH__ >= MIN_CC_DP4A
    constexpr mmq_arch_config_t arch_config = get_arch_config_device(MMQ_CONFIG_Q2_K);

    mul_mat_q<QK_K, QR2_K, QI2_K, false, block_q2_K, arch_config.x, arch_config.y, arch_config.nwarps, allocate_tiles_q2_K<arch_config.y>,
        load_tiles_q2_K<arch_config.y, arch_config.nwarps, need_check>, VDR_Q2_K_Q8_1_MMQ, vec_dot_q2_K_q8_1_mul_mat>
        (vx, vy, dst, ncols_x, nrows_x, ncols_y, nrows_y, nrows_dst);
#else
    GGML_UNUSED(get_arch_config_device);
    GGML_UNUSED(vec_dot_q2_K_q8_1_mul_mat);
    NO_DEVICE_CODE;
#endif // __CUDA_ARCH__ >= MIN_CC_DP4A
}

template <bool need_check> static __global__ void
#if defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__)
#if defined(RDNA3) || defined(RDNA2)
    __launch_bounds__(WARP_SIZE*MMQ_CONFIG_Q3_K.rdna2.nwarps, 2)
#endif // defined(RDNA3) || defined(RDNA2)
#elif __CUDA_ARCH__ < CC_VOLTA
    __launch_bounds__(WARP_SIZE*MMQ_CONFIG_Q3_K.pascal.nwarps, 2)
#endif // __CUDA_ARCH__ < CC_VOLTA
    mul_mat_q3_K(
    const void * __restrict__ vx, const void * __restrict__ vy, float * __restrict__ dst,
    const int ncols_x, const int nrows_x, const int ncols_y, const int nrows_y, const int nrows_dst) {

#if __CUDA_ARCH__ >= MIN_CC_DP4A
    constexpr mmq_arch_config_t arch_config = get_arch_config_device(MMQ_CONFIG_Q3_K);

    mul_mat_q<QK_K, QR3_K, QI3_K, false, block_q3_K, arch_config.x, arch_config.y, arch_config.nwarps, allocate_tiles_q3_K<arch_config.y>,
        load_tiles_q3_K<arch_config.y, arch_config.nwarps, need_check>, VDR_Q3_K_Q8_1_MMQ, vec_dot_q3_K_q8_1_mul_mat>
        (vx, vy, dst, ncols_x, nrows_x, ncols_y, nrows_y, nrows_dst);
#else
    GGML_UNUSED(get_arch_config_device);
    GGML_UNUSED(vec_dot_q3_K_q8_1_mul_mat);
    NO_DEVICE_CODE;
#endif // __CUDA_ARCH__ >= MIN_CC_DP4A
}

template <bool need_check> static __global__ void
#if defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__)
#if defined(RDNA3) || defined(RDNA2)
    __launch_bounds__(WARP_SIZE*MMQ_CONFIG_Q4_K.rdna2.nwarps, 2)
#endif // defined(RDNA3) || defined(RDNA2)
#elif __CUDA_ARCH__ < CC_VOLTA
    __launch_bounds__(WARP_SIZE*MMQ_CONFIG_Q4_K.pascal.nwarps, 2)
#endif // __CUDA_ARCH__ < CC_VOLTA
    mul_mat_q4_K(
    const void * __restrict__ vx, const void * __restrict__ vy, float * __restrict__ dst,
    const int ncols_x, const int nrows_x, const int ncols_y, const int nrows_y, const int nrows_dst) {

#if __CUDA_ARCH__ >= MIN_CC_DP4A
    constexpr mmq_arch_config_t arch_config = get_arch_config_device(MMQ_CONFIG_Q4_K);

    mul_mat_q<QK_K, QR4_K, QI4_K, true, block_q4_K, arch_config.x, arch_config.y, arch_config.nwarps, allocate_tiles_q4_K<arch_config.y>,
        load_tiles_q4_K<arch_config.y, arch_config.nwarps, need_check>, VDR_Q4_K_Q8_1_MMQ, vec_dot_q4_K_q8_1_mul_mat>
        (vx, vy, dst, ncols_x, nrows_x, ncols_y, nrows_y, nrows_dst);
#else
    GGML_UNUSED(get_arch_config_device);
    GGML_UNUSED(vec_dot_q4_K_q8_1_mul_mat);
    NO_DEVICE_CODE;
#endif // __CUDA_ARCH__ >= MIN_CC_DP4A
}

template <bool need_check> static __global__ void
#if defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__)
#if defined(RDNA3) || defined(RDNA2)
    __launch_bounds__(WARP_SIZE*MMQ_CONFIG_Q5_K.rdna2.nwarps, 2)
#endif // defined(RDNA3) || defined(RDNA2)
#endif // defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__)
mul_mat_q5_K(
    const void * __restrict__ vx, const void * __restrict__ vy, float * __restrict__ dst,
    const int ncols_x, const int nrows_x, const int ncols_y, const int nrows_y, const int nrows_dst) {

#if __CUDA_ARCH__ >= MIN_CC_DP4A
    constexpr mmq_arch_config_t arch_config = get_arch_config_device(MMQ_CONFIG_Q5_K);

    mul_mat_q<QK_K, QR5_K, QI5_K, true, block_q5_K, arch_config.x, arch_config.y, arch_config.nwarps, allocate_tiles_q5_K<arch_config.y>,
        load_tiles_q5_K<arch_config.y, arch_config.nwarps, need_check>, VDR_Q5_K_Q8_1_MMQ, vec_dot_q5_K_q8_1_mul_mat>
        (vx, vy, dst, ncols_x, nrows_x, ncols_y, nrows_y, nrows_dst);
#else
    GGML_UNUSED(get_arch_config_device);
    GGML_UNUSED(vec_dot_q5_K_q8_1_mul_mat);
    NO_DEVICE_CODE;
#endif // __CUDA_ARCH__ >= MIN_CC_DP4A
}

template <bool need_check> static __global__ void
#if defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__)
#if defined(RDNA3) || defined(RDNA2)
    __launch_bounds__(WARP_SIZE*MMQ_CONFIG_Q6_K.rdna2.nwarps, 2)
#endif // defined(RDNA3) || defined(RDNA2)
#elif __CUDA_ARCH__ < CC_VOLTA
    __launch_bounds__(WARP_SIZE*MMQ_CONFIG_Q4_K.pascal.nwarps, 2)
#endif // __CUDA_ARCH__ < CC_VOLTA
    mul_mat_q6_K(
    const void * __restrict__ vx, const void * __restrict__ vy, float * __restrict__ dst,
    const int ncols_x, const int nrows_x, const int ncols_y, const int nrows_y, const int nrows_dst) {

#if __CUDA_ARCH__ >= MIN_CC_DP4A
    constexpr mmq_arch_config_t arch_config = get_arch_config_device(MMQ_CONFIG_Q6_K);

    mul_mat_q<QK_K, QR6_K, QI6_K, false, block_q6_K, arch_config.x, arch_config.y, arch_config.nwarps, allocate_tiles_q6_K<arch_config.y>,
        load_tiles_q6_K<arch_config.y, arch_config.nwarps, need_check>, VDR_Q6_K_Q8_1_MMQ, vec_dot_q6_K_q8_1_mul_mat>
        (vx, vy, dst, ncols_x, nrows_x, ncols_y, nrows_y, nrows_dst);
#else
    GGML_UNUSED(get_arch_config_device);
    GGML_UNUSED(vec_dot_q6_K_q8_1_mul_mat);
    NO_DEVICE_CODE;
#endif // __CUDA_ARCH__ >= MIN_CC_DP4A
}

#define MMQ_SWITCH_CASE(type_suffix)                                                                        \
    case GGML_TYPE_Q##type_suffix: if (row_diff % arch_config.y == 0) {                                     \
        const bool need_check = false;                                                                      \
        mul_mat_q##type_suffix<need_check><<<block_nums, block_dims, 0, stream>>>                           \
            (src0_dd_i, src1_ddq_i, dst_dd_i, ne00, row_diff, src1_ncols, src1_padded_row_size, nrows_dst); \
    } else {                                                                                                \
        const bool need_check = true;                                                                       \
        mul_mat_q##type_suffix<need_check><<<block_nums, block_dims, 0, stream>>>                           \
            (src0_dd_i, src1_ddq_i, dst_dd_i, ne00, row_diff, src1_ncols, src1_padded_row_size, nrows_dst); \
    } break;                                                                                                \

void ggml_cuda_op_mul_mat_q(
    ggml_backend_cuda_context & ctx,
    const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst, const char * src0_dd_i, const float * src1_ddf_i,
    const char * src1_ddq_i, float * dst_dd_i, const int64_t row_low, const int64_t row_high, const int64_t src1_ncols,
    const int64_t src1_padded_row_size, hipStream_t stream) {

    const int64_t ne00 = src0->ne[0];

    const int64_t ne10 = src1->ne[0];
    GGML_ASSERT(ne10 % QK8_1 == 0);

    const int64_t ne0 = dst->ne[0];

    const int64_t row_diff = row_high - row_low;

    int id = ggml_cuda_get_device();
    const int compute_capability = ggml_cuda_info().devices[id].cc;

    // the main device has a larger memory buffer to hold the results from all GPUs
    // nrows_dst == nrows of the matrix that the kernel writes into
    const int64_t nrows_dst = id == ctx.device ? ne0 : row_diff;

    mmq_config_t mmq_config;

    switch (src0->type) {
        case GGML_TYPE_Q4_0:
            mmq_config = MMQ_CONFIG_Q4_0;
            break;
        case GGML_TYPE_Q4_1:
            mmq_config = MMQ_CONFIG_Q4_1;
            break;
        case GGML_TYPE_Q5_0:
            mmq_config = MMQ_CONFIG_Q5_0;
            break;
        case GGML_TYPE_Q5_1:
            mmq_config = MMQ_CONFIG_Q5_1;
            break;
        case GGML_TYPE_Q8_0:
            mmq_config = MMQ_CONFIG_Q8_0;
            break;
        case GGML_TYPE_Q2_K:
            mmq_config = MMQ_CONFIG_Q2_K;
            break;
        case GGML_TYPE_Q3_K:
            mmq_config = MMQ_CONFIG_Q3_K;
            break;
        case GGML_TYPE_Q4_K:
            mmq_config = MMQ_CONFIG_Q4_K;
            break;
        case GGML_TYPE_Q5_K:
            mmq_config = MMQ_CONFIG_Q5_K;
            break;
        case GGML_TYPE_Q6_K:
            mmq_config = MMQ_CONFIG_Q6_K;
            break;
        default:
            GGML_ASSERT(false);
            break;
    }

    mmq_arch_config_t arch_config;
    if (compute_capability >= CC_RDNA2) {
        arch_config = mmq_config.rdna2;
    } else if (compute_capability >= CC_OFFSET_AMD) {
        arch_config = mmq_config.rdna1;
    } else if (compute_capability >= CC_VOLTA) {
        arch_config = mmq_config.ampere;
    } else if (compute_capability >= MIN_CC_DP4A) {
        arch_config = mmq_config.pascal;
    } else {
        GGML_ASSERT(false);
    }

    const int block_num_x = (row_diff   + arch_config.y - 1) / arch_config.y;
    const int block_num_y = (src1_ncols + arch_config.x - 1) / arch_config.x;
    const dim3 block_nums(block_num_x, block_num_y, 1);
    const dim3 block_dims(WARP_SIZE, arch_config.nwarps, 1);

    switch (src0->type) {
        MMQ_SWITCH_CASE(4_0)
        MMQ_SWITCH_CASE(4_1)
        MMQ_SWITCH_CASE(5_0)
        MMQ_SWITCH_CASE(5_1)
        MMQ_SWITCH_CASE(8_0)
        MMQ_SWITCH_CASE(2_K)
        MMQ_SWITCH_CASE(3_K)
        MMQ_SWITCH_CASE(4_K)
        MMQ_SWITCH_CASE(5_K)
        MMQ_SWITCH_CASE(6_K)
        default:
            GGML_ASSERT(false);
            break;
    }

    GGML_UNUSED(src1);
    GGML_UNUSED(dst);
    GGML_UNUSED(src1_ddf_i);
}

bool ggml_cuda_supports_mmq(enum ggml_type type) {
    switch (type) {
        case GGML_TYPE_Q4_0:
        case GGML_TYPE_Q4_1:
        case GGML_TYPE_Q5_0:
        case GGML_TYPE_Q5_1:
        case GGML_TYPE_Q8_0:
        case GGML_TYPE_Q2_K:
        case GGML_TYPE_Q3_K:
        case GGML_TYPE_Q4_K:
        case GGML_TYPE_Q5_K:
        case GGML_TYPE_Q6_K:
            return true;
        default:
            return false;
    }
}

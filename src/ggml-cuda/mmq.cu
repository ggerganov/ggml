#include "mmq.cuh"

void ggml_cuda_op_mul_mat_q(
    ggml_backend_cuda_context & ctx,
    const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst, const char * src0_dd_i, const float * src1_ddf_i,
    const char * src1_ddq_i, float * dst_dd_i, const int64_t row_low, const int64_t row_high, const int64_t src1_ncols,
    const int64_t src1_padded_row_size, hipStream_t stream) {

    const int64_t ne00 = src0->ne[0];

    const int64_t nb01 = src0->nb[1];

    const int64_t ne10 = src1->ne[0];
    GGML_ASSERT(ne10 % QK8_1 == 0);

    const int64_t ne0 = dst->ne[0];

    const int64_t row_diff = row_high - row_low;
    const int64_t stride00 = nb01 / ggml_type_size(src0->type);

    int id = ggml_cuda_get_device();
    const int compute_capability = ggml_cuda_info().devices[id].cc;

    // the main device has a larger memory buffer to hold the results from all GPUs
    // nrows_dst == nrows of the matrix that the kernel writes into
    const int64_t nrows_dst = id == ctx.device ? ne0 : row_diff;

    const mmq_args args = {src0_dd_i, src1_ddq_i, dst_dd_i, ne00, row_diff, stride00, src1_padded_row_size, src1_ncols, nrows_dst};

    switch (src0->type) {
        case GGML_TYPE_Q4_0:
            mul_mat_q_case<GGML_TYPE_Q4_0>(args, stream);
            break;
        case GGML_TYPE_Q4_1:
            mul_mat_q_case<GGML_TYPE_Q4_1>(args, stream);
            break;
        case GGML_TYPE_Q5_0:
            mul_mat_q_case<GGML_TYPE_Q5_0>(args, stream);
            break;
        case GGML_TYPE_Q5_1:
            mul_mat_q_case<GGML_TYPE_Q5_1>(args, stream);
            break;
        case GGML_TYPE_Q8_0:
            mul_mat_q_case<GGML_TYPE_Q8_0>(args, stream);
            break;
        case GGML_TYPE_Q2_K:
            mul_mat_q_case<GGML_TYPE_Q2_K>(args, stream);
            break;
        case GGML_TYPE_Q3_K:
            mul_mat_q_case<GGML_TYPE_Q3_K>(args, stream);
            break;
        case GGML_TYPE_Q4_K:
            mul_mat_q_case<GGML_TYPE_Q4_K>(args, stream);
            break;
        case GGML_TYPE_Q5_K:
            mul_mat_q_case<GGML_TYPE_Q5_K>(args, stream);
            break;
        case GGML_TYPE_Q6_K:
            mul_mat_q_case<GGML_TYPE_Q6_K>(args, stream);
            break;
        default:
            GGML_ASSERT(false);
            break;
    }

    GGML_UNUSED(src1);
    GGML_UNUSED(dst);
    GGML_UNUSED(src1_ddf_i);
}

bool ggml_cuda_supports_mmq(enum ggml_type type) {
    switch (type) {
        case GGML_TYPE_Q4_0:
        case GGML_TYPE_Q4_1:
        case GGML_TYPE_Q5_0:
        case GGML_TYPE_Q5_1:
        case GGML_TYPE_Q8_0:
        case GGML_TYPE_Q2_K:
        case GGML_TYPE_Q3_K:
        case GGML_TYPE_Q4_K:
        case GGML_TYPE_Q5_K:
        case GGML_TYPE_Q6_K:
            return true;
        default:
            return false;
    }
}

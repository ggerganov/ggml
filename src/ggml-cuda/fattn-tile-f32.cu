#include "hip/hip_runtime.h"
#include "common.cuh"
#include "fattn-common.cuh"
#include "fattn-tile-f32.cuh"

#define FATTN_KQ_STRIDE_TILE_F32 32

template<int D, int ncols, int nwarps, int parallel_blocks> // D == head size
#if !(defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__))
__launch_bounds__(nwarps*WARP_SIZE, 1)
#endif // !(defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__))
static __global__ void flash_attn_tile_ext_f32(
        const char * __restrict__ Q,
        const char * __restrict__ K,
        const char * __restrict__ V,
        const char * __restrict__ mask,
        float      * __restrict__ dst,
        float2     * __restrict__ dst_meta,
        const float scale,
        const float max_bias,
        const float m0,
        const float m1,
        const uint32_t n_head_log2,
        const int ne00,
        const int ne01,
        const int ne02,
        const int ne03,
        const int ne10,
        const int ne11,
        const int ne12,
        const int ne13,
        const int ne31,
        const int nb31,
        const int nb01,
        const int nb02,
        const int nb03,
        const int nb11,
        const int nb12,
        const int nb13,
        const int ne0,
        const int ne1,
        const int ne2,
        const int ne3) {
    //In this kernel Q, K, V are matrices while i, j, k are matrix indices.

    const int ic0 = (blockIdx.x / parallel_blocks) * ncols; // Index of the Q/QKV column to work on.
    const int ip  =  blockIdx.x % parallel_blocks; // Index in group of blocks running for the same column in parallel.

    const int gqa_ratio = ne02 / ne12; // With grouped query attention there are > 1 Q matrices per K, V matrix.
    const float2 * Q_f2  = (const float2 *) (Q    + nb02* blockIdx.y              + nb01*ic0);
    const half2  * K_h2  = (const half2  *) (K    + nb12*(blockIdx.y / gqa_ratio));
    const half2  * V_h2  = (const half2  *) (V    + nb12*(blockIdx.y / gqa_ratio)); // K and V have same shape
    const half   * maskh = (const half   *)  mask + ne11*ic0;

    const int stride_KV2 = nb11 / sizeof(half2);

    float slope = 1.0f;

    // ALiBi
    if (max_bias > 0.0f) {
        const uint32_t h = blockIdx.y;

        const float base = h < n_head_log2 ? m0 : m1;
        const int   exph = h < n_head_log2 ? h + 1 : 2*(h - n_head_log2) + 1;

        slope = powf(base, exph);
    }

    static_assert(D % (2*WARP_SIZE) == 0, "D not divisible by 2*WARP_SIZE == 64.");

    __shared__ float KQ[ncols*FATTN_KQ_STRIDE_TILE_F32];

    __shared__ float KV_tmp[FATTN_KQ_STRIDE_TILE_F32][D + 1]; // Pad D to avoid memory bank conflicts.
    float2 * KV_tmp2 = (float2 *) KV_tmp;

    float kqmax[ncols/nwarps];
#pragma unroll
    for (int j0 = 0; j0 < ncols; j0 += nwarps) {
        kqmax[j0/nwarps] = -FLT_MAX/2.0f;
    }
    float kqsum[ncols/nwarps] = {0.0f};

    float2 VKQ[ncols/nwarps][(D/2)/WARP_SIZE] = {{{0.0f, 0.0f}}};

    // Convert Q to half2 and store in registers:
    __shared__ float Q_f[ncols][D];
#pragma unroll
    for (int j0 = 0; j0 < ncols; j0 += nwarps) {
        const int j = j0 + threadIdx.y;

#pragma unroll
        for (int i0 = 0; i0 < D; i0 += 2*WARP_SIZE) {
            float2 tmp = Q_f2[j*(nb01/sizeof(float2)) + i0/2 + threadIdx.x];
            Q_f[j][i0 + 0*WARP_SIZE + threadIdx.x] = tmp.x * scale;
            Q_f[j][i0 + 1*WARP_SIZE + threadIdx.x] = tmp.y * scale;
        }
    }

    __syncthreads();

    const int k_start = parallel_blocks == 1 ? 0 : ip*FATTN_KQ_STRIDE_TILE_F32;
    for (int k_VKQ_0 = k_start; k_VKQ_0 < ne11; k_VKQ_0 += parallel_blocks*FATTN_KQ_STRIDE_TILE_F32) {
        // Calculate KQ tile and keep track of new maximum KQ values:

        float kqmax_new[ncols/nwarps];
#pragma unroll
        for (int j = 0; j < ncols/nwarps; ++j) {
            kqmax_new[j] = kqmax[j];
        }

#pragma unroll
        for (int i_KQ_0 = 0; i_KQ_0 < FATTN_KQ_STRIDE_TILE_F32; i_KQ_0 += nwarps) {
            const int i_KQ = i_KQ_0 + threadIdx.y;

#pragma unroll
            for (int k_KQ_0 = 0; k_KQ_0 < D; k_KQ_0 += 2*WARP_SIZE) {
                const half2 tmp = K_h2[(k_VKQ_0 + i_KQ)*stride_KV2 + k_KQ_0/2 + threadIdx.x];
                KV_tmp[i_KQ][k_KQ_0 + 0*WARP_SIZE + threadIdx.x] =  __low2float(tmp);
                KV_tmp[i_KQ][k_KQ_0 + 1*WARP_SIZE + threadIdx.x] = __high2float(tmp);
            }
        }

        __syncthreads();

        float sum[FATTN_KQ_STRIDE_TILE_F32/WARP_SIZE][ncols/nwarps] = {{0.0f}};

#pragma unroll
        for (int k_KQ = 0; k_KQ < D; ++k_KQ) {
            float K_k[FATTN_KQ_STRIDE_TILE_F32/WARP_SIZE];
            float Q_k[ncols/nwarps];

#pragma unroll
            for (int i_KQ_0 = 0; i_KQ_0 < FATTN_KQ_STRIDE_TILE_F32; i_KQ_0 += WARP_SIZE) {
                const int i_KQ = i_KQ_0 + threadIdx.x;

                K_k[i_KQ_0/WARP_SIZE] = KV_tmp[i_KQ][k_KQ];
            }
#pragma unroll
            for (int j_KQ_0 = 0; j_KQ_0 < ncols; j_KQ_0 += nwarps) {
                const int j_KQ = j_KQ_0 + threadIdx.y;

                Q_k[j_KQ_0/nwarps] = Q_f[j_KQ][k_KQ];
            }

#pragma unroll
            for (int i_KQ_0 = 0; i_KQ_0 < FATTN_KQ_STRIDE_TILE_F32; i_KQ_0 += WARP_SIZE) {
#pragma unroll
                for (int j_KQ_0 = 0; j_KQ_0 < ncols; j_KQ_0 += nwarps) {
                    sum[i_KQ_0/WARP_SIZE][j_KQ_0/nwarps] += K_k[i_KQ_0/WARP_SIZE] * Q_k[j_KQ_0/nwarps];
                }
            }
        }

#pragma unroll
        for (int i_KQ_0 = 0; i_KQ_0 < FATTN_KQ_STRIDE_TILE_F32; i_KQ_0 += WARP_SIZE) {
            const int i_KQ = i_KQ_0 + threadIdx.x;

#pragma unroll
            for (int j_KQ_0 = 0; j_KQ_0 < ncols; j_KQ_0 += nwarps) {
                const int j_KQ = j_KQ_0 + threadIdx.y;

                sum[i_KQ_0/WARP_SIZE][j_KQ_0/nwarps] += mask ? slope*__half2float(maskh[j_KQ*ne11 + k_VKQ_0 + i_KQ]) : 0.0f;

                kqmax_new[j_KQ_0/nwarps] = fmaxf(kqmax_new[j_KQ_0/nwarps], sum[i_KQ_0/WARP_SIZE][j_KQ_0/nwarps]);

                KQ[j_KQ*FATTN_KQ_STRIDE_TILE_F32 + i_KQ] = sum[i_KQ_0/WARP_SIZE][j_KQ_0/nwarps];
            }
        }

        __syncthreads();

#pragma unroll
        for (int j0 = 0; j0 < ncols; j0 += nwarps) {
            const int j = j0 + threadIdx.y;

            kqmax_new[j0/nwarps] = warp_reduce_max(kqmax_new[j0/nwarps]);
            const float KQ_max_scale = expf(kqmax[j0/nwarps] - kqmax_new[j0/nwarps]);
            kqmax[j0/nwarps] = kqmax_new[j0/nwarps];

            float kqsum_add = 0.0f;
#pragma unroll
            for (int i0 = 0; i0 < FATTN_KQ_STRIDE_TILE_F32; i0 += WARP_SIZE) {
                const int i = i0 + threadIdx.x;

                const float diff = KQ[j*FATTN_KQ_STRIDE_TILE_F32 + i] - kqmax[j0/nwarps];
                const float val = expf(diff);
                kqsum_add += val;
                KQ[j*FATTN_KQ_STRIDE_TILE_F32 + i] = val;
            }
            kqsum[j0/nwarps] = kqsum[j0/nwarps]*KQ_max_scale + kqsum_add;

#pragma unroll
            for (int i0 = 0; i0 < D/2; i0 += WARP_SIZE) {
                VKQ[j0/nwarps][i0/WARP_SIZE].x *= KQ_max_scale;
                VKQ[j0/nwarps][i0/WARP_SIZE].y *= KQ_max_scale;
            }
        }

        __syncthreads();

#pragma unroll
        for (int k0 = 0; k0 < FATTN_KQ_STRIDE_TILE_F32; k0 += nwarps) {
            const int k = k0 + threadIdx.y;

#pragma unroll
            for (int i0 = 0; i0 < D/2; i0 += WARP_SIZE) {
                const int i = i0 + threadIdx.x;

                KV_tmp2[k*(D/2) + i].x =  __low2float(V_h2[(k_VKQ_0 + k)*stride_KV2 + i]);
                KV_tmp2[k*(D/2) + i].y = __high2float(V_h2[(k_VKQ_0 + k)*stride_KV2 + i]);
            }
        }

        __syncthreads();

#pragma unroll
        for (int k = 0; k < FATTN_KQ_STRIDE_TILE_F32; ++k) {
            float2 V_k[(D/2)/WARP_SIZE];
            float  KQ_k[ncols/nwarps];

#pragma unroll
            for (int i0 = 0; i0 < D/2; i0 += WARP_SIZE) {
                const int i = i0 + threadIdx.x;

                V_k[i0/WARP_SIZE] = KV_tmp2[k*(D/2) + i];
            }
#pragma unroll
            for (int j0 = 0; j0 < ncols; j0 += nwarps) {
                const int j = j0 + threadIdx.y;

                KQ_k[j0/nwarps] = KQ[j*FATTN_KQ_STRIDE_TILE_F32 + k];
            }

#pragma unroll
            for (int i0 = 0; i0 < D/2; i0 += WARP_SIZE) {
#pragma unroll
                for (int j0 = 0; j0 < ncols; j0 += nwarps) {
                    VKQ[j0/nwarps][i0/WARP_SIZE].x += V_k[i0/WARP_SIZE].x*KQ_k[j0/nwarps];
                    VKQ[j0/nwarps][i0/WARP_SIZE].y += V_k[i0/WARP_SIZE].y*KQ_k[j0/nwarps];
                }
            }
        }

        __syncthreads();
    }

#pragma unroll
    for (int j_VKQ_0 = 0; j_VKQ_0 < ncols; j_VKQ_0 += nwarps) {
        const int j_VKQ = j_VKQ_0 + threadIdx.y;

        float kqsum_j = kqsum[j_VKQ_0/nwarps];
        kqsum_j = warp_reduce_sum(kqsum_j);

#pragma unroll
        for (int i00 = 0; i00 < D; i00 += 2*WARP_SIZE) {
            const int i0 = i00 + 2*threadIdx.x;

            float2 dst_val = VKQ[j_VKQ_0/nwarps][i0/(2*WARP_SIZE)];
            if (parallel_blocks == 1) {
                dst_val.x /= kqsum_j;
                dst_val.y /= kqsum_j;
            }
            const int j_dst = (ic0 + j_VKQ)*parallel_blocks + ip;
            dst[j_dst*D*gridDim.y + D*blockIdx.y + i0 + 0] = dst_val.x;
            dst[j_dst*D*gridDim.y + D*blockIdx.y + i0 + 1] = dst_val.y;
        }

        if (parallel_blocks != 1 && threadIdx.x == 0) {
            dst_meta[(ic0 + j_VKQ)*gridDim.y*parallel_blocks + blockIdx.y*parallel_blocks + ip] = make_float2(kqmax[j_VKQ_0/nwarps], kqsum_j);
        }
    }
}

template <int D, int cols_per_block, int parallel_blocks> void launch_fattn_tile_f32(
        const ggml_tensor * Q, const ggml_tensor * K, const ggml_tensor * V, ggml_tensor * KQV, const ggml_tensor * mask,
        ggml_cuda_pool & pool, hipStream_t main_stream
) {
    ggml_cuda_pool_alloc<float>  dst_tmp(pool);
    ggml_cuda_pool_alloc<float2> dst_tmp_meta(pool);

    if (parallel_blocks > 1) {
        dst_tmp.alloc(parallel_blocks*ggml_nelements(KQV));
        dst_tmp_meta.alloc(parallel_blocks*ggml_nrows(KQV));
    }

    constexpr int  nwarps = 8;
    const     dim3 block_dim(WARP_SIZE, nwarps, 1);
    const     dim3 blocks_num(parallel_blocks*((Q->ne[1] + cols_per_block - 1) / cols_per_block), Q->ne[2], Q->ne[3]);
    const     int  shmem = 0;

    float scale    = 1.0f;
    float max_bias = 0.0f;

    memcpy(&scale,    (float *) KQV->op_params + 0, sizeof(float));
    memcpy(&max_bias, (float *) KQV->op_params + 1, sizeof(float));

    const uint32_t n_head      = Q->ne[2];
    const uint32_t n_head_log2 = 1u << (uint32_t) floorf(log2f((float) n_head));

    const float m0 = powf(2.0f, -(max_bias       ) / n_head_log2);
    const float m1 = powf(2.0f, -(max_bias / 2.0f) / n_head_log2);

    flash_attn_tile_ext_f32<D, cols_per_block, nwarps, parallel_blocks>
        <<<blocks_num, block_dim, shmem, main_stream>>> (
                (const char *) Q->data,
                (const char *) K->data,
                (const char *) V->data,
                mask ? ((const char *) mask->data) : nullptr,
                parallel_blocks == 1 ? (float *) KQV->data : dst_tmp.ptr, dst_tmp_meta.ptr,
                scale, max_bias, m0, m1, n_head_log2,
                Q->ne[0], Q->ne[1], Q->ne[2], Q->ne[3],
                K->ne[0], K->ne[1], K->ne[2], K->ne[3],
                mask ? mask->ne[1] : 0, mask ?  mask->nb[1] : 0,
                Q->nb[1], Q->nb[2], Q->nb[3],
                K->nb[1], K->nb[2], K->nb[3],
                KQV->ne[0], KQV->ne[1], KQV->ne[2], KQV->ne[3]
                );
    CUDA_CHECK(hipGetLastError());

    if (parallel_blocks == 1) {
        return;
    }

    const dim3 block_dim_combine(D, 1, 1);
    const dim3 blocks_num_combine(Q->ne[1], blocks_num.y, blocks_num.z);
    const int  shmem_combine = 0;

    flash_attn_combine_results<D, parallel_blocks>
        <<<blocks_num_combine, block_dim_combine, shmem_combine, main_stream>>>
        (dst_tmp.ptr, dst_tmp_meta.ptr, (float *) KQV->data);
    CUDA_CHECK(hipGetLastError());
}

void ggml_cuda_flash_attn_ext_tile_f32(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * Q = dst->src[0];
    const ggml_tensor * K = dst->src[1];
    const ggml_tensor * V = dst->src[2];

    const ggml_tensor * mask = dst->src[3];

    ggml_tensor * KQV = dst;

    const int32_t precision = KQV->op_params[2];
    GGML_ASSERT(precision == GGML_PREC_DEFAULT);
    GGML_ASSERT(Q->ne[0] == 64 || Q->ne[0] == 128 && "FlashAttention without tensor cores only supports head sizes 64 and 128.");

    if (Q->ne[1] <= 16) {
        constexpr int cols_per_block = 16;
        constexpr int parallel_blocks = 4;
        switch (Q->ne[0]) {
            case 64:
                launch_fattn_tile_f32< 64, cols_per_block, parallel_blocks>(Q, K, V, KQV, mask, ctx.pool(), ctx.stream());
                break;
            case 128:
                launch_fattn_tile_f32<128, cols_per_block, parallel_blocks>(Q, K, V, KQV, mask, ctx.pool(), ctx.stream());
                break;
            default:
                GGML_ASSERT(false);
                break;
        }
        return;
    }

    if (Q->ne[1] <= 32) {
        constexpr int cols_per_block = 32;
        constexpr int parallel_blocks = 4;
        switch (Q->ne[0]) {
            case 64:
                launch_fattn_tile_f32< 64, cols_per_block, parallel_blocks>(Q, K, V, KQV, mask, ctx.pool(), ctx.stream());
                break;
            case 128:
                launch_fattn_tile_f32<128, cols_per_block, parallel_blocks>(Q, K, V, KQV, mask, ctx.pool(), ctx.stream());
                break;
            default:
                GGML_ASSERT(false);
                break;
        }
        return;
    }

    constexpr int cols_per_block = 32;
    constexpr int parallel_blocks = 1;
    switch (Q->ne[0]) {
        case 64:
            launch_fattn_tile_f32< 64, cols_per_block, parallel_blocks>(Q, K, V, KQV, mask, ctx.pool(), ctx.stream());
            break;
        case 128:
            launch_fattn_tile_f32<128, cols_per_block, parallel_blocks>(Q, K, V, KQV, mask, ctx.pool(), ctx.stream());
            break;
        default:
            GGML_ASSERT(false);
            break;
    }
}

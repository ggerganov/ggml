#include "hip/hip_runtime.h"
#include "common.cuh"
#include "fattn.cuh"

#include <cstdint>

#if FP16_MMA_AVAILABLE
#include <mma.h>
#endif

#define FATTN_KQ_STRIDE       256
#define HALF_MAX_HALF         __float2half(65504.0f/2) // Use neg. of this instead of -INFINITY to initialize KQ max vals to avoid NaN upon subtraction.
#define SOFTMAX_FTZ_THRESHOLD -20.0f                   // Softmax exp. of values smaller than this are flushed to zero to avoid NaNs.

template<int D, int ncols, int parallel_blocks> // D == head size
#if !(defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__))
__launch_bounds__(D, 1)
#endif // !(defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__))
static __global__ void flash_attn_vec_ext_f16(
        const char * __restrict__ Q,
        const char * __restrict__ K,
        const char * __restrict__ V,
        const char * __restrict__ mask,
        float      * __restrict__ dst,
        float2     * __restrict__ dst_meta,
        const float scale,
        const int ne00,
        const int ne01,
        const int ne02,
        const int ne03,
        const int ne10,
        const int ne11,
        const int ne12,
        const int ne13,
        const int ne31,
        const int nb31,
        const int nb01,
        const int nb02,
        const int nb03,
        const int nb11,
        const int nb12,
        const int nb13,
        const int ne0,
        const int ne1,
        const int ne2,
        const int ne3) {
#if FP16_AVAILABLE
    //In this kernel Q, K, V are matrices while i, j, k are matrix indices.

    const int ic0 = (blockIdx.x / parallel_blocks) * ncols; // Index of the Q/QKV column to work on.
    const int ip  =  blockIdx.x % parallel_blocks; // Index in group of blocks running for the same column in parallel.

    const int gqa_ratio = ne02 / ne12; // With grouped query attention there are > 1 Q matrices per K, V matrix.
    const float2 * Q_f2  = (const float2 *) (Q    + nb02* blockIdx.y              + nb01*ic0);
    const half2  * K_h2  = (const half2  *) (K    + nb12*(blockIdx.y / gqa_ratio));
    const half   * V_h   = (const half   *) (V    + nb12*(blockIdx.y / gqa_ratio)); // K and V have same shape
    const half   * maskh = (const half   *)  mask + ne11*ic0;

    const int stride_KV  = nb11 / sizeof(half);
    const int stride_KV2 = nb11 / sizeof(half2);

    static_assert(D % (2*WARP_SIZE) == 0, "D not divisible by 2*WARP_SIZE == 64.");
    constexpr int nwarps = D / WARP_SIZE;
    const int tid = WARP_SIZE*threadIdx.y + threadIdx.x;
    __builtin_assume(tid < D);

    __shared__ half KQ[ncols*D];
#pragma unroll
    for (int j = 0; j < ncols; ++j) {
        KQ[j*D + tid] = -HALF_MAX_HALF;
    }
    half2 * KQ2 = (half2 *) KQ;

    half kqmax[ncols];
#pragma unroll
    for (int j = 0; j < ncols; ++j) {
        kqmax[j] = -HALF_MAX_HALF;
    }
    half kqsum[ncols] = {0.0f};

    __shared__ half kqmax_shared[ncols][WARP_SIZE];
    __shared__ half kqsum_shared[ncols][WARP_SIZE];
#pragma unroll
    for (int j = 0; j < ncols; ++j) {
        if (threadIdx.y == 0) {
            kqmax_shared[j][threadIdx.x] = -HALF_MAX_HALF;
            kqsum_shared[j][threadIdx.x] = 0.0f;
        }
    }
    __syncthreads();

    // Convert Q to half2 and store in registers:
    half2 Q_h2[ncols][D/(2*WARP_SIZE)];
#pragma unroll
    for (int j = 0; j < ncols; ++j) {
#pragma unroll
        for (int i0 = 0; i0 < D/2; i0 += WARP_SIZE) {
            const int i = i0 + threadIdx.x;

            const float2 tmp = Q_f2[j*(nb01/sizeof(float2)) + i];
            Q_h2[j][i0/WARP_SIZE] = make_half2(scale, scale) * make_half2(tmp.x, tmp.y);
        }
    }

    half2 VKQ[ncols] = {{0.0f, 0.0f}};

    const int k_start = parallel_blocks == 1 ? 0 : ip*D;
    for (int k_VKQ_0 = k_start; k_VKQ_0 < ne11; k_VKQ_0 += parallel_blocks*D) {
        // Calculate KQ tile and keep track of new maximum KQ values:

        // For unknown reasons using a half array of size 1 for kqmax_new causes a performance regression,
        // see https://github.com/ggerganov/llama.cpp/pull/7061 .
        // Therefore this variable is defined twice but only used once (so that the compiler can optimize out the unused variable).
        half kqmax_new = kqmax[0];
        half kqmax_new_arr[ncols];
#pragma unroll
        for (int j = 0; j < ncols; ++j) {
            kqmax_new_arr[j] = kqmax[j];
        }

#pragma unroll
        for (int i_KQ_0 = 0; i_KQ_0 < D; i_KQ_0 += nwarps) {
            const int i_KQ = i_KQ_0 + threadIdx.y;

            if ((i_KQ_0 + nwarps > D && i_KQ >= D) || (FATTN_KQ_STRIDE % D != 0 && k_VKQ_0 + i_KQ >= ne11)) {
                break;
            }

            half2 sum2[ncols] = {{0.0f, 0.0f}};
#pragma unroll
            for (int k_KQ_0 = 0; k_KQ_0 < D/2; k_KQ_0 += WARP_SIZE) {
                const int k_KQ = k_KQ_0 + threadIdx.x;

                const half2 K_ik = K_h2[(k_VKQ_0 + i_KQ)*stride_KV2 + k_KQ];
#pragma unroll
                for (int j = 0; j < ncols; ++j) {
                    sum2[j] += K_ik * Q_h2[j][k_KQ_0/WARP_SIZE];
                }
            }

#pragma unroll
            for (int j = 0; j < ncols; ++j) {
                sum2[j] = warp_reduce_sum(sum2[j]);
                half sum = __low2half(sum2[j]) + __high2half(sum2[j]);
                sum += mask ? maskh[j*ne11 + k_VKQ_0 + i_KQ] : __float2half(0.0f);

                if (ncols == 1) {
                    kqmax_new        = ggml_cuda_hmax(kqmax_new,        sum);
                } else {
                    kqmax_new_arr[j] = ggml_cuda_hmax(kqmax_new_arr[j], sum);
                }

                if (threadIdx.x == 0) {
                    KQ[j*D + i_KQ] = sum;
                }
            }
        }

#pragma unroll
        for (int j = 0; j < ncols; ++j) {
            half kqmax_new_j = ncols == 1 ? kqmax_new : kqmax_new_arr[j];

            kqmax_new_j = warp_reduce_max(kqmax_new_j);
            if (threadIdx.x == 0) {
                kqmax_shared[j][threadIdx.y] = kqmax_new_j;
            }
        }

        __syncthreads();

#pragma unroll
        for (int j = 0; j < ncols; ++j) {
            half kqmax_new_j = kqmax_shared[j][threadIdx.x];
            kqmax_new_j = warp_reduce_max(kqmax_new_j);

            const half KQ_max_scale = hexp(kqmax[j] - kqmax_new_j);
            kqmax[j] = kqmax_new_j;

            const half val = hexp(KQ[j*D + tid] - kqmax[j]);
            kqsum[j] = kqsum[j]*KQ_max_scale + val;
            KQ[j*D + tid] = val;

            VKQ[j] *= __half2half2(KQ_max_scale);
        }

        __syncthreads();

#pragma unroll
        for (int k0 = 0; k0 < D; k0 += 2) {
            if (FATTN_KQ_STRIDE % D != 0 && k_VKQ_0 + k0 >= ne11) {
                break;
            }

            half2 V_k;
            reinterpret_cast<half&>(V_k.x) = V_h[(k_VKQ_0 + k0 + 0)*stride_KV + tid];
            reinterpret_cast<half&>(V_k.y) = V_h[(k_VKQ_0 + k0 + 1)*stride_KV + tid];
#pragma unroll
            for (int j = 0; j < ncols; ++j) {
                VKQ[j] += V_k*KQ2[j*(D/2) + k0/2];
            }
        }

        __syncthreads();
    }

#pragma unroll
    for (int j = 0; j < ncols; ++j) {
        kqsum[j] = warp_reduce_sum(kqsum[j]);
        if (threadIdx.x == 0) {
            kqsum_shared[j][threadIdx.y] = kqsum[j];
        }
    }

    __syncthreads();

#pragma unroll
    for (int j_VKQ = 0; j_VKQ < ncols; ++j_VKQ) {
        kqsum[j_VKQ] = kqsum_shared[j_VKQ][threadIdx.x];
        kqsum[j_VKQ] = warp_reduce_sum(kqsum[j_VKQ]);

        half dst_val = (__low2half(VKQ[j_VKQ]) + __high2half(VKQ[j_VKQ]));
        if (parallel_blocks == 1) {
            dst_val /= kqsum[j_VKQ];
        }
        const int j_dst = (ic0 + j_VKQ)*parallel_blocks + ip;
        dst[j_dst*D*gridDim.y + D*blockIdx.y + tid] = dst_val;
    }

    if (parallel_blocks != 1 && tid != 0) {
#pragma unroll
        for (int j = 0; j < ncols; ++j) {
            dst_meta[(ic0 + j)*gridDim.y*parallel_blocks + blockIdx.y*parallel_blocks + ip] = make_float2(kqmax[j], kqsum[j]);
        }
    }
#else
   NO_DEVICE_CODE;
#endif // FP16_AVAILABLE
}

// D == head size, VKQ_stride == num VKQ rows calculated in parallel:
template<int D, int ncols, int nwarps, int VKQ_stride, int parallel_blocks, typename KQ_acc_t>
#if !(defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__))
__launch_bounds__(nwarps*WARP_SIZE, 1)
#endif // !(defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__))
static __global__ void flash_attn_ext_f16(
        const char * __restrict__ Q,
        const char * __restrict__ K,
        const char * __restrict__ V,
        const char * __restrict__ mask,
        float      * __restrict__ dst,
        float2     * __restrict__ dst_meta,
        const float scale,
        const int ne00,
        const int ne01,
        const int ne02,
        const int ne03,
        const int ne10,
        const int ne11,
        const int ne12,
        const int ne13,
        const int ne31,
        const int nb31,
        const int nb01,
        const int nb02,
        const int nb03,
        const int nb11,
        const int nb12,
        const int nb13,
        const int ne0,
        const int ne1,
        const int ne2,
        const int ne3) {
#if FP16_MMA_AVAILABLE
    //In this kernel Q, K, V are matrices while i, j, k are matrix indices.

    const int ic0 = ncols*(blockIdx.x / parallel_blocks); // Index of the first Q/QKV column to work on.
    const int ip  =        blockIdx.x % parallel_blocks;  // Index in group of blocks running for the same column in parallel.

    static_assert(D <= FATTN_KQ_STRIDE, "D must be <= FATTN_KQ_STRIDE.");
    static_assert(ncols == 8 || ncols % 16 == 0, "ncols must be 8 or a multiple of 16.");
    constexpr int frag_m = ncols == 8 ? 32 : 16;
    constexpr int frag_n = ncols == 8 ?  8 : 16;
    static_assert(D % frag_m == 0, "If ncols == 8 then D % frag_m must be 0.");
    typedef nvcuda::wmma::fragment<nvcuda::wmma::matrix_a,    frag_m, frag_n, 16, half, nvcuda::wmma::row_major> frag_a_K;
    typedef nvcuda::wmma::fragment<nvcuda::wmma::matrix_a,    frag_m, frag_n, 16, half, nvcuda::wmma::col_major> frag_a_V;
    typedef nvcuda::wmma::fragment<nvcuda::wmma::matrix_b,    frag_m, frag_n, 16, half, nvcuda::wmma::col_major> frag_b;
    typedef nvcuda::wmma::fragment<nvcuda::wmma::accumulator, frag_m, frag_n, 16, KQ_acc_t>                      frag_c_KQ;
    typedef nvcuda::wmma::fragment<nvcuda::wmma::accumulator, frag_m, frag_n, 16, half>                          frag_c_VKQ;

    constexpr int KQ_stride_tc  = nwarps*frag_m; // Number of KQ rows calculated in parallel.
    constexpr int VKQ_ratio = KQ_stride_tc/VKQ_stride; // Number of parallel VKQ accumulators needed to keep all warps busy.
    static_assert(VKQ_ratio <= nwarps, "VKQ_ratio must be <= nwarps.");

    // Pad internal representation of KQ, KQV to reduce shared memory bank conflicts:
    constexpr int D_padded = D + 8;
    constexpr int kqs_padded = FATTN_KQ_STRIDE + 8;
    constexpr int kqar = sizeof(KQ_acc_t)/sizeof(half);

    const int gqa_ratio = ne02 / ne12; // With grouped query attention there are > 1 Q matrices per K, V matrix.
    const float * Q_f   = (const float *) (Q + nb02* blockIdx.y              + nb01*ic0);
    const half  * K_h   = (const half  *) (K + nb12*(blockIdx.y / gqa_ratio));
    const half  * V_h   = (const half  *) (V + nb12*(blockIdx.y / gqa_ratio)); // K and V have same shape
    const half  * maskh = (const half  *)  mask + (nb31/sizeof(half))* ic0;
    const half2 * mask2 = (const half2 *)  mask + (nb31/sizeof(half))*(ic0/2);

    const int stride_Q  = nb01 / sizeof(float);
    const int stride_KV = nb11 / sizeof(half);

    frag_b Q_b[D/16][ncols/frag_n];

    // A single buffer for temporarily holding tiles of KQ and VKQ parts:
    constexpr int mem_KQ = ncols*kqs_padded*kqar;
    constexpr int mem_VKQ_parts = VKQ_ratio*ncols*D_padded;
    __shared__ half KQ[mem_KQ >= mem_VKQ_parts ? mem_KQ : mem_VKQ_parts];
    float * KQ_f = (float *) KQ;
    half2 * KQ2 = (half2 *) KQ;

    float    KQ_rowsum_f[ncols/nwarps] = {0.0f};
    float       KQ_max_f[ncols/nwarps];
    float KQ_max_scale_f[ncols/nwarps] = {0.0f};

#pragma unroll
    for (int j = 0; j < ncols/nwarps; ++j) {
        KQ_max_f[j] = -FLT_MAX/2.0f;
    }

    half2    KQ_rowsum_h2[ncols/nwarps] = {{0.0f, 0.0f}};
    half2       KQ_max_h2[ncols/nwarps];
    half2 KQ_max_scale_h2[ncols/nwarps] = {{0.0f, 0.0f}};

#pragma unroll
    for (int j = 0; j < ncols/nwarps; ++j) {
        KQ_max_h2[j] = make_half2(-HALF_MAX_HALF, -HALF_MAX_HALF);
    }

    __shared__ half VKQ[ncols*D_padded]; // Accumulator for final VKQ slice.
    half2 * VKQ2 = (half2 *) VKQ;
#pragma unroll
    for (int j0 = 0; j0 < ncols; j0 += nwarps) {
        const int j = j0 + threadIdx.y;
#pragma unroll
        for (int i0 = 0; i0 < D/2; i0 += WARP_SIZE) {
            const int i = i0 + threadIdx.x;
            if (i0 + WARP_SIZE > D/2 && i >= D/2) {
                break;
            }
            VKQ2[j*(D_padded/2) + i] = make_half2(0.0f, 0.0f);
        }
    }

    // Convert Q to half and apply scale, temporarily store in KQ:
#pragma unroll
    for (int j0 = 0; j0 < ncols; j0 += nwarps) {
        const int j = j0 + threadIdx.y;
#pragma unroll
        for (int i0 = 0; i0 < D; i0 += WARP_SIZE) {
            const int i = i0 + threadIdx.x;
            if (i0 + WARP_SIZE > D && i >= D) {
                break;
            }
            KQ[j*D_padded + i] = ic0 + j < ne01 ? Q_f[j*stride_Q + i] * scale : 0.0f;
        }
    }

    __syncthreads();

    // Load Q into tensor core fragments/registers since it will be used frequently:
#pragma unroll
    for (int i0 = 0; i0 < D; i0 += 16) {
#pragma unroll
        for (int j0 = 0; j0 < ncols; j0 += frag_n) {
            nvcuda::wmma::load_matrix_sync(Q_b[i0/16][j0/frag_n], KQ + j0*D_padded + i0, D_padded);
        }
    }

    __syncthreads();

    // Iterate over ne11 == previous tokens:
    for (int k_VKQ_0 = ip*FATTN_KQ_STRIDE; k_VKQ_0 < ne11; k_VKQ_0 += parallel_blocks*FATTN_KQ_STRIDE) {
        // Calculate tile of KQ:
#pragma unroll
        for (int i_KQ_0 = 0; i_KQ_0 < FATTN_KQ_STRIDE; i_KQ_0 += KQ_stride_tc) {
            frag_c_KQ KQ_c[ncols/frag_n];
#pragma unroll
            for (int j = 0; j < ncols/frag_n; ++j) {
                nvcuda::wmma::fill_fragment(KQ_c[j], 0.0f);
            }
#pragma unroll
            for (int k_KQ_0 = 0; k_KQ_0 < D; k_KQ_0 += 16) {
                frag_a_K K_a;
                nvcuda::wmma::load_matrix_sync(K_a, K_h + (k_VKQ_0 + i_KQ_0 + frag_m*threadIdx.y)*stride_KV + k_KQ_0, stride_KV);
#pragma unroll
                for (int j = 0; j < ncols/frag_n; ++j) {
                    nvcuda::wmma::mma_sync(KQ_c[j], K_a, Q_b[k_KQ_0/16][j], KQ_c[j]);
                }
            }
#pragma unroll
            for (int j0 = 0; j0 < ncols; j0 += frag_n) {
                nvcuda::wmma::store_matrix_sync((KQ_acc_t *) KQ + j0*kqs_padded + i_KQ_0 + frag_m*threadIdx.y, KQ_c[j0/frag_n], kqs_padded, nvcuda::wmma::mem_col_major);
            }
        }

        __syncthreads();

        // Calculate softmax for each KQ column using the current max. value.
        // The divisor is stored in KQ_rowsum and will be applied at the end.
#pragma unroll
        for (int j0 = 0; j0 < ncols; j0 += nwarps) {
            const int j = j0 + threadIdx.y;

            if (std::is_same<KQ_acc_t, float>::value) {
                float KQ_f_tmp[FATTN_KQ_STRIDE / WARP_SIZE];
#pragma unroll
                for (int k0 = 0; k0 < FATTN_KQ_STRIDE; k0 += WARP_SIZE) {
                    const int k = k0 + threadIdx.x;

                    KQ_f_tmp[k0/WARP_SIZE] = KQ_f[j*kqs_padded + k];
                }

                float KQ_max_new = KQ_max_f[j0/nwarps];
#pragma unroll
                for (int k0 = 0; k0 < FATTN_KQ_STRIDE; k0 += WARP_SIZE) {
                    const int k = k0 + threadIdx.x;

                    KQ_f_tmp[k0/WARP_SIZE] += mask ? __half2float(maskh[j*(nb31/sizeof(half)) + k_VKQ_0 + k]) : 0.0f;
                    KQ_max_new = max(KQ_max_new, KQ_f_tmp[k0/WARP_SIZE]);
                }
                KQ_max_new = warp_reduce_max(KQ_max_new);

                const float diff = KQ_max_f[j0/nwarps] - KQ_max_new;
                KQ_max_scale_f[j0/nwarps] = expf(diff);
                if (diff <= SOFTMAX_FTZ_THRESHOLD) {
                    KQ_max_scale_f[j0/nwarps] = 0.0f;
                }
                KQ_max_f[j0/nwarps] = KQ_max_new;

                float KQ_rowsum_add = 0.0f;
#pragma unroll
                for (int k0 = 0; k0 < FATTN_KQ_STRIDE; k0 += WARP_SIZE) {
                    const int k = k0 + threadIdx.x;

                    const float diff = KQ_f_tmp[k0/WARP_SIZE] - KQ_max_f[j0/nwarps];
                    KQ_f_tmp[k0/WARP_SIZE] = expf(diff);
                    if (diff <= SOFTMAX_FTZ_THRESHOLD) {
                        KQ_f_tmp[k0/WARP_SIZE] = 0.0f;
                    }
                    KQ_rowsum_add += KQ_f_tmp[k0/WARP_SIZE];
                    KQ[j*(kqar*kqs_padded) + k] = KQ_f_tmp[k0/WARP_SIZE];
                }
                KQ_rowsum_add = warp_reduce_sum(KQ_rowsum_add);

                // Scale previous KQ_rowsum to account for a potential increase in KQ_max:
                KQ_rowsum_f[j0/nwarps] = KQ_max_scale_f[j0/nwarps]*KQ_rowsum_f[j0/nwarps] + KQ_rowsum_add;
            } else {
                half2 KQ2_tmp[FATTN_KQ_STRIDE/(2*WARP_SIZE)];
#pragma unroll
                for (int k0 = 0; k0 < FATTN_KQ_STRIDE/2; k0 += WARP_SIZE) {
                    const int k = k0 + threadIdx.x;

                    KQ2_tmp[k0/WARP_SIZE] = KQ2[j*(kqs_padded/2) + k];
                }

                half2 KQ_max_new = KQ_max_h2[j0/nwarps];
#pragma unroll
                for (int k0 = 0; k0 < FATTN_KQ_STRIDE/2; k0 += WARP_SIZE) {
                    const int k = k0 + threadIdx.x;

                    KQ2_tmp[k0/WARP_SIZE] += mask ? mask2[(j*ne11 + k_VKQ_0)/2 + k] : make_half2(0.0f, 0.0f);
                    KQ_max_new = ggml_cuda_hmax2(KQ_max_new, KQ2_tmp[k0/WARP_SIZE]);
                }
                KQ_max_new = __half2half2(warp_reduce_max(ggml_cuda_hmax(__low2half(KQ_max_new), __high2half(KQ_max_new))));
                const half2 diff = KQ_max_h2[j0/nwarps] - KQ_max_new;
                KQ_max_scale_h2[j0/nwarps] = h2exp(diff);
                const uint32_t ftz_mask = __hgt2_mask(diff, make_half2(SOFTMAX_FTZ_THRESHOLD, SOFTMAX_FTZ_THRESHOLD));
                *((uint32_t *) &KQ_max_scale_h2[j0/nwarps]) &= ftz_mask;
                KQ_max_h2[j0/nwarps] = KQ_max_new;

                half2 KQ_rowsum_add = make_half2(0.0f, 0.0f);
#pragma unroll
                for (int k0 = 0; k0 < FATTN_KQ_STRIDE/2; k0 += WARP_SIZE) {
                    const int k = k0 + threadIdx.x;

                    const half2 diff = KQ2_tmp[k0/WARP_SIZE] - KQ_max_h2[j0/nwarps];
                    KQ2_tmp[k0/WARP_SIZE] = h2exp(diff);
                    const uint32_t ftz_mask = __hgt2_mask(diff, make_half2(SOFTMAX_FTZ_THRESHOLD, SOFTMAX_FTZ_THRESHOLD));
                    *((uint32_t *) &KQ2_tmp[k0/WARP_SIZE]) &= ftz_mask;
                    KQ_rowsum_add += KQ2_tmp[k0/WARP_SIZE];
                    KQ2[j*(kqs_padded/2) + k] = KQ2_tmp[k0/WARP_SIZE];
                }
                KQ_rowsum_add = warp_reduce_sum(KQ_rowsum_add);

                // Scale previous KQ_rowsum to account for a potential increase in KQ_max:
                KQ_rowsum_h2[j0/nwarps] = KQ_max_scale_h2[j0/nwarps]*KQ_rowsum_h2[j0/nwarps] + KQ_rowsum_add;
            }
        }

        __syncthreads();

        frag_b KQ_b[FATTN_KQ_STRIDE/(VKQ_ratio*16)][ncols/frag_n];
#pragma unroll
        for (int j0 = 0; j0 < ncols; j0 += frag_n) {
#pragma unroll
            for (int k0 = 0; k0 < FATTN_KQ_STRIDE; k0 += VKQ_ratio*16) {
                const int k = k0 + (threadIdx.y % VKQ_ratio)*16;
                nvcuda::wmma::load_matrix_sync(
                    KQ_b[k0/(VKQ_ratio*16)][j0/frag_n],
                    KQ + j0*(kqar*kqs_padded) + k,
                    kqar*kqs_padded);
            }
        }

        frag_c_VKQ VKQ_c[D/VKQ_stride][ncols/frag_n];
#pragma unroll
        for (int i_VKQ_0 = 0; i_VKQ_0 < D; i_VKQ_0 += VKQ_stride) {
#pragma unroll
            for (int j = 0; j < ncols/frag_n; ++j) {
                nvcuda::wmma::fill_fragment(VKQ_c[i_VKQ_0/VKQ_stride][j], 0.0f);
            }

#pragma unroll
            for (int k0 = 0; k0 < FATTN_KQ_STRIDE; k0 += VKQ_ratio*16) {
                const int k = k0 + (threadIdx.y % VKQ_ratio)*16;

                frag_a_V v_a;
                nvcuda::wmma::load_matrix_sync(v_a, V_h + (k_VKQ_0 + k)*stride_KV + i_VKQ_0 + frag_m*(threadIdx.y/VKQ_ratio), stride_KV);
#pragma unroll
                for (int j = 0; j < ncols/frag_n; ++j) {
                    nvcuda::wmma::mma_sync(VKQ_c[i_VKQ_0/VKQ_stride][j], v_a, KQ_b[k0/(VKQ_ratio*16)][j], VKQ_c[i_VKQ_0/VKQ_stride][j]);
                }
            }
        }

        __syncthreads();

        const int offset_k = (threadIdx.y % VKQ_ratio) * (ncols*D_padded);
#pragma unroll
        for (int i_KQ_0 = 0; i_KQ_0 < D; i_KQ_0 += VKQ_stride) {
#pragma unroll
            for (int j0 = 0; j0 < ncols; j0 += frag_n) {
                nvcuda::wmma::store_matrix_sync(
                    KQ + offset_k + j0*D_padded + i_KQ_0 + frag_m*(threadIdx.y/VKQ_ratio),
                    VKQ_c[i_KQ_0/VKQ_stride][j0/frag_n],
                    D_padded, nvcuda::wmma::mem_col_major);
            }
        }

        __syncthreads();

#pragma unroll
        for (int j0 = 0; j0 < ncols; j0 += nwarps) {
            const int j = j0 + threadIdx.y;

            half2 VKQ_scale;
            if (std::is_same<KQ_acc_t, float>::value) {
                VKQ_scale = make_half2(KQ_max_scale_f[j0/nwarps], KQ_max_scale_f[j0/nwarps]);
            } else {
                VKQ_scale = KQ_max_scale_h2[j0/nwarps];
            }

#pragma unroll
            for (int i0 = 0; i0 < D/2; i0 += WARP_SIZE) {
                const int i = i0 + threadIdx.x;
                if (i0 + WARP_SIZE > D/2 && i >= D/2) {
                    break;
                }

                half2 VKQ_add = make_half2(0.0f, 0.0f);
#pragma unroll
                for (int l = 0; l < VKQ_ratio; ++l) {
                    VKQ_add += KQ2[l*(ncols*D_padded/2) + j*(D_padded/2) + i];
                }
                VKQ2[j*(D_padded/2) + i] = VKQ_scale*VKQ2[j*(D_padded/2) + i] + VKQ_add;
            }
        }

        __syncthreads();
    }

#pragma unroll
    for (int j0 = 0; j0 < ncols; j0 += nwarps) {
        const int j_VKQ = j0 + threadIdx.y;
        if (ic0 + j_VKQ >= ne01) {
            return;
        }
        const int j_dst = (ic0 + j_VKQ)*parallel_blocks + ip;

        float KQ_rowsum_j;
        if (std::is_same<KQ_acc_t, float>::value) {
            KQ_rowsum_j = KQ_rowsum_f[j0/nwarps];
        } else {
            KQ_rowsum_j = __low2float(KQ_rowsum_h2[j0/nwarps]) + __high2float(KQ_rowsum_h2[j0/nwarps]);
        }

#pragma unroll
        for (int i0 = 0; i0 < D; i0 += WARP_SIZE) {
            const int i = i0 + threadIdx.x;
            if (i0 + WARP_SIZE > D && i >= D) {
                break;
            }
            float dst_val = VKQ[j_VKQ*D_padded + i];
            if (parallel_blocks == 1) {
                dst_val /= KQ_rowsum_j;
            }
            dst[j_dst*gridDim.y*D + blockIdx.y*D + i] = dst_val;
        }

        if (parallel_blocks == 1 || threadIdx.x != 0) {
            continue;
        }

        float2 dst_meta_val;
        if (std::is_same<KQ_acc_t, float>::value) {
            dst_meta_val.x = KQ_max_f[j0/nwarps];
        } else {
            dst_meta_val.x = __low2float(KQ_max_h2[j0/nwarps]);
        }
        dst_meta_val.y = KQ_rowsum_j;
        dst_meta[(ic0 + j_VKQ)*gridDim.y*parallel_blocks + blockIdx.y*parallel_blocks + ip] = dst_meta_val;
    }
#else
   NO_DEVICE_CODE;
#endif // FP16_MMA_AVAILABLE
}

template<int D, int parallel_blocks> // D == head size
#if !(defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__))
__launch_bounds__(D, 1)
#endif // !(defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__))
static __global__ void flash_attn_combine_results(
        const float  * __restrict__ VKQ_parts,
        const float2 * __restrict__ VKQ_meta,
        float * __restrict__ dst) {
#if FP16_AVAILABLE
    VKQ_parts += parallel_blocks*D * gridDim.y*blockIdx.x;
    VKQ_meta  += parallel_blocks   * gridDim.y*blockIdx.x;
    dst       +=                 D * gridDim.y*blockIdx.x;

    const int tid = threadIdx.x;
    __builtin_assume(tid < D);

    __shared__ float2 meta[parallel_blocks];
    if (tid < 2*parallel_blocks) {
        ((float *) meta)[threadIdx.x] = ((const float *)VKQ_meta) [blockIdx.y*(2*parallel_blocks) + tid];
    }

    __syncthreads();

    float kqmax = meta[0].x;
#pragma unroll
    for (int l = 1; l < parallel_blocks; ++l) {
        kqmax = max(kqmax, meta[l].x);
    }

    float VKQ_numerator   = 0.0f;
    float VKQ_denominator = 0.0f;
#pragma unroll
    for (int l = 0; l < parallel_blocks; ++l) {
        const float diff = meta[l].x - kqmax;
        const float KQ_max_scale = expf(diff);
        const uint32_t ftz_mask = 0xFFFFFFFF * (diff > SOFTMAX_FTZ_THRESHOLD);
        *((uint32_t *) &KQ_max_scale) &= ftz_mask;

        VKQ_numerator   += KQ_max_scale * VKQ_parts[l*gridDim.y*D + blockIdx.y*D + tid];
        VKQ_denominator += KQ_max_scale * meta[l].y;
    }

    dst[blockIdx.y*D + tid] = VKQ_numerator / VKQ_denominator;
#else
   NO_DEVICE_CODE;
#endif // FP16_AVAILABLE
}

constexpr int get_max_power_of_2(int x) {
    return x % 2 == 0 ? 2*get_max_power_of_2(x/2) : 1;
}

static_assert(get_max_power_of_2(1) == 1, "Test failed.");
static_assert(get_max_power_of_2(2) == 2, "Test failed.");
static_assert(get_max_power_of_2(4) == 4, "Test failed.");
static_assert(get_max_power_of_2(6) == 2, "Test failed.");

// Number of VKQ rows calculated in parallel:
constexpr int get_VKQ_stride(int D, int nwarps, int frag_m) {
    return (get_max_power_of_2(D/frag_m) < nwarps ? get_max_power_of_2(D/frag_m) : nwarps)*frag_m;
}

static_assert(get_VKQ_stride(128, 1, 32) ==  32, "Test failed.");
static_assert(get_VKQ_stride(128, 2, 32) ==  64, "Test failed.");
static_assert(get_VKQ_stride(128, 4, 32) == 128, "Test failed.");
static_assert(get_VKQ_stride( 64, 1, 32) ==  32, "Test failed.");
static_assert(get_VKQ_stride( 64, 2, 32) ==  64, "Test failed.");
static_assert(get_VKQ_stride( 64, 4, 32) ==  64, "Test failed.");
static_assert(get_VKQ_stride( 80, 1, 16) ==  16, "Test failed.");
static_assert(get_VKQ_stride( 80, 2, 16) ==  16, "Test failed.");
static_assert(get_VKQ_stride( 80, 4, 16) ==  16, "Test failed.");

template <int D, int cols_per_block, int parallel_blocks> void launch_fattn_vec_f16(
        const ggml_tensor * Q, const ggml_tensor * K, const ggml_tensor * V, ggml_tensor * KQV, const ggml_tensor * mask,
        ggml_cuda_pool & pool, hipStream_t main_stream
) {
    ggml_cuda_pool_alloc<float>  dst_tmp(pool);
    ggml_cuda_pool_alloc<float2> dst_tmp_meta(pool);

    if (parallel_blocks > 1) {
        dst_tmp.alloc(parallel_blocks*ggml_nelements(KQV));
        dst_tmp_meta.alloc(parallel_blocks*ggml_nrows(KQV));
    }

    constexpr int  nwarps = (D + WARP_SIZE - 1) / WARP_SIZE;
    const     dim3 block_dim(WARP_SIZE, nwarps, 1);
    const     dim3 blocks_num(parallel_blocks*((Q->ne[1] + cols_per_block - 1) / cols_per_block), Q->ne[2], Q->ne[3]);
    const     int  shmem = 0;

    float scale;
    memcpy(&scale, KQV->op_params, sizeof(float));

    flash_attn_vec_ext_f16<D, cols_per_block, parallel_blocks>
        <<<blocks_num, block_dim, shmem, main_stream>>> (
                (const char *) Q->data,
                (const char *) K->data,
                (const char *) V->data,
                mask ? ((const char *) mask->data) : nullptr,
                parallel_blocks == 1 ? (float *) KQV->data : dst_tmp.ptr, dst_tmp_meta.ptr,
                scale,
                Q->ne[0], Q->ne[1], Q->ne[2], Q->ne[3],
                K->ne[0], K->ne[1], K->ne[2], K->ne[3],
                mask ? mask->ne[1] : 0, mask ?  mask->nb[1] : 0,
                Q->nb[1], Q->nb[2], Q->nb[3],
                K->nb[1], K->nb[2], K->nb[3],
                KQV->ne[0], KQV->ne[1], KQV->ne[2], KQV->ne[3]
                );
    CUDA_CHECK(hipGetLastError());

    if (parallel_blocks == 1) {
        return;
    }

    const dim3 block_dim_combine(D, 1, 1);
    const dim3 blocks_num_combine(Q->ne[1], blocks_num.y, blocks_num.z);
    const int  shmem_combine = 0;

    flash_attn_combine_results<D, parallel_blocks>
        <<<blocks_num_combine, block_dim_combine, shmem_combine, main_stream>>>
        (dst_tmp.ptr, dst_tmp_meta.ptr, (float *) KQV->data);
    CUDA_CHECK(hipGetLastError());
}

template <int D, int cols_per_block, int nwarps, int parallel_blocks, typename KQ_acc_t> void launch_fattn_f16_impl(
        const ggml_tensor * Q, const ggml_tensor * K, const ggml_tensor * V, ggml_tensor * KQV, const ggml_tensor * mask,
        ggml_cuda_pool & pool, hipStream_t main_stream
) {
    ggml_cuda_pool_alloc<float>  dst_tmp(pool);
    ggml_cuda_pool_alloc<float2> dst_tmp_meta(pool);

    if (parallel_blocks > 1) {
        dst_tmp.alloc(parallel_blocks*ggml_nelements(KQV));
        dst_tmp_meta.alloc(parallel_blocks*ggml_nrows(KQV));
    }

    constexpr int  frag_m = (cols_per_block) == 8 && (D) % 32 == 0 ? 32 : 16;
    const     dim3 block_dim(WARP_SIZE, nwarps, 1);
    const     dim3 blocks_num(parallel_blocks*(Q->ne[1] + cols_per_block - 1) / cols_per_block, Q->ne[2], Q->ne[3]);
    const     int  shmem = 0;

    float scale;
    memcpy(&scale, KQV->op_params, sizeof(float));

    flash_attn_ext_f16<D, cols_per_block, nwarps, get_VKQ_stride(D, nwarps, frag_m), parallel_blocks, KQ_acc_t>
        <<<blocks_num, block_dim, shmem, main_stream>>> (
                (const char *) Q->data,
                (const char *) K->data,
                (const char *) V->data,
                mask ? ((const char *) mask->data) : nullptr,
                (parallel_blocks) == 1 ? (float *) KQV->data : dst_tmp.ptr, dst_tmp_meta.ptr,
                scale,
                Q->ne[0], Q->ne[1], Q->ne[2], Q->ne[3],
                K->ne[0], K->ne[1], K->ne[2], K->ne[3],
                mask ? mask->ne[1] : 0, mask ?  mask->nb[1] : 0,
                Q->nb[1], Q->nb[2], Q->nb[3],
                K->nb[1], K->nb[2], K->nb[3],
                KQV->ne[0], KQV->ne[1], KQV->ne[2], KQV->ne[3]
                );
    CUDA_CHECK(hipGetLastError());

    if ((parallel_blocks) == 1) {
        return;
    }

    const dim3 block_dim_combine(D, 1, 1);
    const dim3 blocks_num_combine(Q->ne[1], blocks_num.y, blocks_num.z);
    const int  shmem_combine = 0;

    flash_attn_combine_results<D, parallel_blocks>
        <<<blocks_num_combine, block_dim_combine, shmem_combine, main_stream>>>
        (dst_tmp.ptr, dst_tmp_meta.ptr, (float *) KQV->data);
    CUDA_CHECK(hipGetLastError());
}

template <int D, int cols_per_block, int nwarps, typename KQ_acc_t> void launch_fattn_f16(
        const ggml_tensor * Q, const ggml_tensor * K, const ggml_tensor * V, ggml_tensor * KQV, const ggml_tensor * mask,
        const int nsm, ggml_cuda_pool & pool, hipStream_t main_stream
) {
    const int blocks_num_pb1 = ((Q->ne[1] + cols_per_block - 1) / cols_per_block)*Q->ne[2]*Q->ne[3];

    if (4*blocks_num_pb1 < 2*nsm) {
        launch_fattn_f16_impl<D, cols_per_block, nwarps, 4, KQ_acc_t>(Q, K, V, KQV, mask, pool, main_stream);
        return;
    }
    if (2*blocks_num_pb1 < 2*nsm) {
        launch_fattn_f16_impl<D, cols_per_block, nwarps, 2, KQ_acc_t>(Q, K, V, KQV, mask, pool, main_stream);
        return;
    }
    launch_fattn_f16_impl<D, cols_per_block, nwarps, 1, KQ_acc_t>(Q, K, V, KQV, mask, pool, main_stream);
}

void ggml_cuda_flash_attn_ext(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * Q = dst->src[0];
    const ggml_tensor * K = dst->src[1];
    const ggml_tensor * V = dst->src[2];

    const ggml_tensor * mask = dst->src[3];

    ggml_tensor * KQV = dst;

    GGML_ASSERT(Q->type == GGML_TYPE_F32);
    GGML_ASSERT(K->type == GGML_TYPE_F16);
    GGML_ASSERT(V->type == GGML_TYPE_F16);
    GGML_ASSERT(KQV->type == GGML_TYPE_F32);

    GGML_ASSERT(!mask || mask->type == GGML_TYPE_F16);
    GGML_ASSERT(!mask || mask->ne[1] >= GGML_PAD(Q->ne[1], 16) &&
                                "the Flash-Attention CUDA kernel requires the mask to be padded to 16 and at least n_queries big");

    GGML_ASSERT(K->ne[1] % FATTN_KQ_STRIDE == 0 && "Incorrect KV cache padding.");

    ggml_cuda_set_device(ctx.device);

    const int cc  = ggml_cuda_info().devices[ggml_cuda_get_device()].cc;
    const int nsm = ggml_cuda_info().devices[ggml_cuda_get_device()].nsm;

    const int32_t precision = KQV->op_params[1];

    if (!fp16_mma_available(cc)) {
        GGML_ASSERT(precision == GGML_PREC_DEFAULT);
        GGML_ASSERT(Q->ne[0] == 64 || Q->ne[0] == 128 && "FlashAttention without tensor cores only supports head sizes 64 and 128.");

        if (Q->ne[1] == 1) {
            constexpr int cols_per_block = 1;
            constexpr int parallel_blocks = 4;
            switch (Q->ne[0]) {
                case 64:
                    launch_fattn_vec_f16< 64, cols_per_block, parallel_blocks>(Q, K, V, KQV, mask, ctx.pool(), ctx.stream());
                    break;
                case 128:
                    launch_fattn_vec_f16<128, cols_per_block, parallel_blocks>(Q, K, V, KQV, mask, ctx.pool(), ctx.stream());
                    break;
                default:
                    GGML_ASSERT(false);
                    break;
            }
            return;
        }

        if (Q->ne[1] == 2) {
            constexpr int cols_per_block = 2;
            constexpr int parallel_blocks = 4;
            switch (Q->ne[0]) {
                case 64:
                    launch_fattn_vec_f16< 64, cols_per_block, parallel_blocks>(Q, K, V, KQV, mask, ctx.pool(), ctx.stream());
                    break;
                case 128:
                    launch_fattn_vec_f16<128, cols_per_block, parallel_blocks>(Q, K, V, KQV, mask, ctx.pool(), ctx.stream());
                    break;
                default:
                    GGML_ASSERT(false);
                    break;
            }
            return;
        }

        if (Q->ne[1] <= 4) {
            constexpr int cols_per_block = 4;
            constexpr int parallel_blocks = 4;
            switch (Q->ne[0]) {
                case 64:
                    launch_fattn_vec_f16< 64, cols_per_block, parallel_blocks>(Q, K, V, KQV, mask, ctx.pool(), ctx.stream());
                    break;
                case 128:
                    launch_fattn_vec_f16<128, cols_per_block, parallel_blocks>(Q, K, V, KQV, mask, ctx.pool(), ctx.stream());
                    break;
                default:
                    GGML_ASSERT(false);
                    break;
            }
            return;
        }

        if (Q->ne[1] <= 8) {
            constexpr int cols_per_block = 8;
            constexpr int parallel_blocks = 4;
            switch (Q->ne[0]) {
                case 64:
                    launch_fattn_vec_f16< 64, cols_per_block, parallel_blocks>(Q, K, V, KQV, mask, ctx.pool(), ctx.stream());
                    break;
                case 128:
                    launch_fattn_vec_f16<128, cols_per_block, parallel_blocks>(Q, K, V, KQV, mask, ctx.pool(), ctx.stream());
                    break;
                default:
                    GGML_ASSERT(false);
                    break;
            }
            return;
        }

        constexpr int cols_per_block = 8;
        constexpr int parallel_blocks = 1;
        switch (Q->ne[0]) {
            case 64:
                launch_fattn_vec_f16< 64, cols_per_block, parallel_blocks>(Q, K, V, KQV, mask, ctx.pool(), ctx.stream());
                break;
            case 128:
                launch_fattn_vec_f16<128, cols_per_block, parallel_blocks>(Q, K, V, KQV, mask, ctx.pool(), ctx.stream());
                break;
            default:
                GGML_ASSERT(false);
                break;
        }
        return;
    }

    if (precision != GGML_PREC_DEFAULT) {
        if (Q->ne[1] <= 32 || Q->ne[0] > 128) {
            constexpr int cols_per_block = 16;
            constexpr int nwarps         =  4;
            switch (Q->ne[0]) {
                case 64:
                    launch_fattn_f16< 64, cols_per_block, nwarps, float>(Q, K, V, KQV, mask, nsm, ctx.pool(), ctx.stream());
                    break;
                case 80:
                    launch_fattn_f16< 80, cols_per_block, nwarps, float>(Q, K, V, KQV, mask, nsm, ctx.pool(), ctx.stream());
                    break;
                case 96:
                    launch_fattn_f16< 96, cols_per_block, nwarps, float>(Q, K, V, KQV, mask, nsm, ctx.pool(), ctx.stream());
                    break;
                case 112:
                    launch_fattn_f16<112, cols_per_block, nwarps, float>(Q, K, V, KQV, mask, nsm, ctx.pool(), ctx.stream());
                    break;
                case 128:
                    launch_fattn_f16<128, cols_per_block, nwarps, float>(Q, K, V, KQV, mask, nsm, ctx.pool(), ctx.stream());
                    break;
                case 256:
                    launch_fattn_f16<256, cols_per_block, nwarps, float>(Q, K, V, KQV, mask, nsm, ctx.pool(), ctx.stream());
                    break;
                default:
                    GGML_ASSERT(false);
                    break;
            }
        } else {
            constexpr int cols_per_block = 32;
            constexpr int nwarps         =  4;
            switch (Q->ne[0]) {
                case 64:
                    launch_fattn_f16< 64, cols_per_block, nwarps, float>(Q, K, V, KQV, mask, nsm, ctx.pool(), ctx.stream());
                    break;
                case 80:
                    launch_fattn_f16< 80, cols_per_block, nwarps, float>(Q, K, V, KQV, mask, nsm, ctx.pool(), ctx.stream());
                    break;
                case 96:
                    launch_fattn_f16< 96, cols_per_block, nwarps, float>(Q, K, V, KQV, mask, nsm, ctx.pool(), ctx.stream());
                    break;
                case 112:
                    launch_fattn_f16<112, cols_per_block, nwarps, float>(Q, K, V, KQV, mask, nsm, ctx.pool(), ctx.stream());
                    break;
                case 128:
                    launch_fattn_f16<128, cols_per_block, nwarps, float>(Q, K, V, KQV, mask, nsm, ctx.pool(), ctx.stream());
                    break;
                // case 256:
                //     launch_fattn_f16<256, cols_per_block, nwarps, float>(Q, K, V, KQV, mask, nsm, ctx.pool(), ctx.stream());
                //     break;
                default:
                    GGML_ASSERT(false);
                    break;
            }
        }
        return;
    }

    if (Q->ne[1] == 1 && Q->ne[0] % (2*WARP_SIZE) == 0) {
        constexpr int cols_per_block = 1;
        constexpr int parallel_blocks = 4;
        switch (Q->ne[0]) {
            case 64:
                launch_fattn_vec_f16< 64, cols_per_block, parallel_blocks>(Q, K, V, KQV, mask, ctx.pool(), ctx.stream());
                break;
            case 128:
                launch_fattn_vec_f16<128, cols_per_block, parallel_blocks>(Q, K, V, KQV, mask, ctx.pool(), ctx.stream());
                break;
            case 256:
                launch_fattn_vec_f16<256, cols_per_block, parallel_blocks>(Q, K, V, KQV, mask, ctx.pool(), ctx.stream());
                break;
            default:
                GGML_ASSERT(false);
                break;
        }
        return;
    }

    if (Q->ne[1] <= 8 && Q->ne[0] % WARP_SIZE == 0) {
        constexpr int cols_per_block = 8;
        constexpr int nwarps         = 4;
        switch (Q->ne[0]) {
            case 64:
                launch_fattn_f16< 64, cols_per_block, nwarps, half>(Q, K, V, KQV, mask, nsm, ctx.pool(), ctx.stream());
                break;
            case 96:
                launch_fattn_f16< 96, cols_per_block, nwarps, half>(Q, K, V, KQV, mask, nsm, ctx.pool(), ctx.stream());
                break;
            case 128:
                launch_fattn_f16<128, cols_per_block, nwarps, half>(Q, K, V, KQV, mask, nsm, ctx.pool(), ctx.stream());
                break;
            case 256:
                launch_fattn_f16<256, cols_per_block, nwarps, half>(Q, K, V, KQV, mask, nsm, ctx.pool(), ctx.stream());
                break;
            default:
                GGML_ASSERT(false);
                break;
        }
        return;
    }

    if (Q->ne[1] <= 32) {
        constexpr int cols_per_block = 16;
        constexpr int nwarps         =  4;
        switch (Q->ne[0]) {
            case 64:
                launch_fattn_f16< 64, cols_per_block, nwarps, half>(Q, K, V, KQV, mask, nsm, ctx.pool(), ctx.stream());
                break;
            case 80:
                launch_fattn_f16< 80, cols_per_block, nwarps, half>(Q, K, V, KQV, mask, nsm, ctx.pool(), ctx.stream());
                break;
            case 96:
                launch_fattn_f16< 96, cols_per_block, nwarps, half>(Q, K, V, KQV, mask, nsm, ctx.pool(), ctx.stream());
                break;
            case 112:
                launch_fattn_f16<112, cols_per_block, nwarps, half>(Q, K, V, KQV, mask, nsm, ctx.pool(), ctx.stream());
                break;
            case 128:
                launch_fattn_f16<128, cols_per_block, nwarps, half>(Q, K, V, KQV, mask, nsm, ctx.pool(), ctx.stream());
                break;
            case 256:
                launch_fattn_f16<256, cols_per_block, nwarps, half>(Q, K, V, KQV, mask, nsm, ctx.pool(), ctx.stream());
                break;
            default:
                GGML_ASSERT(false);
                break;
        }
        return;
    }

    constexpr int cols_per_block = 32;
    constexpr int nwarps         =  4;
    switch (Q->ne[0]) {
        case 64:
            launch_fattn_f16< 64, cols_per_block, nwarps, half>(Q, K, V, KQV, mask, nsm, ctx.pool(), ctx.stream());
            break;
        case 80:
            launch_fattn_f16< 80, cols_per_block, nwarps, half>(Q, K, V, KQV, mask, nsm, ctx.pool(), ctx.stream());
            break;
        case 96:
            launch_fattn_f16< 96, cols_per_block, nwarps, half>(Q, K, V, KQV, mask, nsm, ctx.pool(), ctx.stream());
            break;
        case 112:
            launch_fattn_f16<112, cols_per_block, nwarps, half>(Q, K, V, KQV, mask, nsm, ctx.pool(), ctx.stream());
            break;
        case 128:
            launch_fattn_f16<128, cols_per_block, nwarps, half>(Q, K, V, KQV, mask, nsm, ctx.pool(), ctx.stream());
            break;
        case 256:
            launch_fattn_f16<256, cols_per_block, nwarps, half>(Q, K, V, KQV, mask, nsm, ctx.pool(), ctx.stream());
            break;
        default:
            GGML_ASSERT(false);
            break;
    }
    return;
}

#include "hip/hip_runtime.h"
#include "concat.cuh"

static __global__ void concat_f32_dim0(const float * x, const float * y, float * dst, const int ne0, const int ne00) {
    int nidx = threadIdx.x + blockIdx.x * blockDim.x;
    if (nidx >= ne0) {
        return;
    }

    int offset_dst =
        nidx +
        blockIdx.y * ne0 +
        blockIdx.z * ne0 * gridDim.y;

    if (nidx < ne00) { // src0
        int offset_src =
            nidx +
            blockIdx.y * ne00 +
            blockIdx.z * ne00 * gridDim.y;
        dst[offset_dst] = x[offset_src];
    } else {
        int offset_src =
            (nidx - ne00) +
            blockIdx.y * (ne0 - ne00) +
            blockIdx.z * (ne0 - ne00) * gridDim.y;
        dst[offset_dst] = y[offset_src];
    }
}

static __global__ void concat_f32_dim1(const float * x, const float * y, float * dst, const int ne0, const int ne01) {
    int nidx = threadIdx.x + blockIdx.x * blockDim.x;
    if (nidx >= ne0) {
        return;
    }

    int offset_dst =
        nidx +
        blockIdx.y * ne0 +
        blockIdx.z * ne0 * gridDim.y;

    if (blockIdx.y < ne01) { // src0
        int offset_src =
            nidx +
            blockIdx.y * ne0 +
            blockIdx.z * ne0 * ne01;
        dst[offset_dst] = x[offset_src];
    } else {
        int offset_src =
            nidx +
            (blockIdx.y - ne01) * ne0 +
            blockIdx.z * ne0 * (gridDim.y - ne01);
        dst[offset_dst] = y[offset_src];
    }
}

static __global__ void concat_f32_dim2(const float * x, const float * y, float * dst, const int ne0, const int ne02) {
    int nidx = threadIdx.x + blockIdx.x * blockDim.x;
    if (nidx >= ne0) {
        return;
    }

    int offset_dst =
        nidx +
        blockIdx.y * ne0 +
        blockIdx.z * ne0 * gridDim.y;

    if (blockIdx.z < ne02) { // src0
        int offset_src =
            nidx +
            blockIdx.y * ne0 +
            blockIdx.z * ne0 * gridDim.y;
        dst[offset_dst] = x[offset_src];
    } else {
        int offset_src =
            nidx +
            blockIdx.y * ne0 +
            (blockIdx.z - ne02) * ne0 *  gridDim.y;
        dst[offset_dst] = y[offset_src];
    }
}

static void concat_f32_cuda(const float * x, const float * y, float * dst, int ne00, int ne01, int ne02, int ne0, int ne1, int ne2, int dim, hipStream_t stream) {
    int num_blocks = (ne0 + CUDA_CONCAT_BLOCK_SIZE - 1) / CUDA_CONCAT_BLOCK_SIZE;
    dim3 gridDim(num_blocks, ne1, ne2);
    if (dim == 0) {
        concat_f32_dim0<<<gridDim, CUDA_CONCAT_BLOCK_SIZE, 0, stream>>>(x, y, dst, ne0, ne00);
        return;
    }
    if (dim == 1) {
        concat_f32_dim1<<<gridDim, CUDA_CONCAT_BLOCK_SIZE, 0, stream>>>(x, y, dst, ne0, ne01);
        return;
    }
    concat_f32_dim2<<<gridDim, CUDA_CONCAT_BLOCK_SIZE, 0, stream>>>(x, y, dst, ne0, ne02);
}

void ggml_cuda_op_concat(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const ggml_tensor * src1 = dst->src[1];

    const float * src0_d = (const float *)src0->data;
    const float * src1_d = (const float *)src1->data;

    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    const int32_t dim = ((int32_t *) dst->op_params)[0];

    GGML_ASSERT(ggml_is_contiguous(src0));
    GGML_ASSERT(ggml_is_contiguous(src1));

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT(src1->type == GGML_TYPE_F32);
    GGML_ASSERT(dst->type == GGML_TYPE_F32);

    if (dim != 3) {
        for (int i3 = 0; i3 < dst->ne[3]; i3++) {
            concat_f32_cuda(
                    src0_d + i3 * (src0->nb[3] / 4),
                    src1_d + i3 * (src1->nb[3] / 4),
                     dst_d + i3 * ( dst->nb[3] / 4),
                    src0->ne[0], src0->ne[1], src0->ne[2],
                     dst->ne[0],  dst->ne[1],  dst->ne[2], dim, stream);
        }
    } else {
        const size_t size0 = ggml_nbytes(src0);
        const size_t size1 = ggml_nbytes(src1);

        CUDA_CHECK(hipMemcpyAsync(dst_d,           src0_d, size0, hipMemcpyDeviceToDevice, stream));
        CUDA_CHECK(hipMemcpyAsync(dst_d + size0/4, src1_d, size1, hipMemcpyDeviceToDevice, stream));
    }
}

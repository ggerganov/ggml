#include "hip/hip_runtime.h"
#include <cstddef>
#include <cstdint>
#include <limits>
#include <stdint.h>
#include <stdio.h>
#include <atomic>
#include <assert.h>

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_fp16.h>

#include "ggml-cuda.h"
#include "ggml.h"

#if defined(_MSC_VER)
#pragma warning(disable: 4244 4267) // possible loss of data
#endif

static_assert(sizeof(half) == sizeof(ggml_fp16_t), "wrong fp16 size");

#define CUDA_CHECK(err)                                                                 \
    do {                                                                                \
        hipError_t err_ = (err);                                                       \
        if (err_ != hipSuccess) {                                                      \
            fprintf(stderr, "CUDA error %d at %s:%d: %s\n", err_, __FILE__, __LINE__,   \
                hipGetErrorString(err_));                                              \
            exit(1);                                                                    \
        }                                                                               \
    } while (0)

#if CUDART_VERSION >= 12000
#define CUBLAS_CHECK(err)                                                               \
    do {                                                                                \
        hipblasStatus_t err_ = (err);                                                    \
        if (err_ != HIPBLAS_STATUS_SUCCESS) {                                            \
            fprintf(stderr, "\ncuBLAS error %d at %s:%d: %s\n",                         \
                    err_, __FILE__, __LINE__, cublasGetStatusString(err_));             \
            exit(1);                                                                    \
        }                                                                               \
    } while (0)
#else
#define CUBLAS_CHECK(err)                                                               \
    do {                                                                                \
        hipblasStatus_t err_ = (err);                                                    \
        if (err_ != HIPBLAS_STATUS_SUCCESS) {                                            \
            fprintf(stderr, "\ncuBLAS error %d at %s:%d\n", err_, __FILE__, __LINE__);  \
            exit(1);                                                                    \
        }                                                                               \
    } while (0)
#endif // CUDART_VERSION >= 11

#ifdef GGML_CUDA_DMMV_F16
typedef half dfloat; // dequantize float
typedef half2 dfloat2;
#else
typedef float dfloat; // dequantize float
typedef float2 dfloat2;
#endif //GGML_CUDA_DMMV_F16

typedef void (*dequantize_kernel_t)(const void * vx, const int ib, const int iqs, dfloat2 & v);
typedef void (*to_fp32_cuda_t)(const void * x, float * y, int k, hipStream_t stream);
typedef void (*dot_kernel_k_t)(const void * vx, const int ib, const int iqs, const float * y, float & v);
typedef void (*cpy_kernel_t)(const char * cx, char * cdst);
typedef void (*ggml_cuda_func_t)(const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst);
typedef void (*ggml_cuda_op_t)(
    const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst, char * src0_ddq_i, float * src0_ddf_i,
    float * src1_ddf_i, float * dst_ddf_i, int64_t i02, int64_t i01_low, int64_t i01_high, int i1,
    hipStream_t & cudaStream_main);

// QK = number of values after dequantization
// QR = QK / number of values before dequantization

#define QK4_0 32
#define QR4_0 2
typedef struct {
    half    d;              // delta
    uint8_t qs[QK4_0 / 2];  // nibbles / quants
} block_q4_0;
static_assert(sizeof(block_q4_0) == sizeof(ggml_fp16_t) + QK4_0 / 2, "wrong q4_0 block size/padding");

#define QK4_1 32
#define QR4_1 2
typedef struct {
    half    d;              // delta
    half    m;              // min
    uint8_t qs[QK4_1 / 2];  // nibbles / quants
} block_q4_1;
static_assert(sizeof(block_q4_1) == sizeof(ggml_fp16_t) * 2 + QK4_1 / 2, "wrong q4_1 block size/padding");

#define QK5_0 32
#define QR5_0 2
typedef struct {
    half d;                 // delta
    uint8_t qh[4];          // 5-th bit of quants
    uint8_t qs[QK5_0 / 2];  // nibbles / quants
} block_q5_0;
static_assert(sizeof(block_q5_0) == sizeof(ggml_fp16_t) + sizeof(uint32_t) + QK5_0 / 2, "wrong q5_0 block size/padding");

#define QK5_1 32
#define QR5_1 2
typedef struct {
    half d;                 // delta
    half m;                 // min
    uint8_t qh[4];          // 5-th bit of quants
    uint8_t qs[QK5_1 / 2];  // nibbles / quants
} block_q5_1;
static_assert(sizeof(block_q5_1) == 2 * sizeof(ggml_fp16_t) + sizeof(uint32_t) + QK5_1 / 2, "wrong q5_1 block size/padding");

#define QK8_0 32
#define QR8_0 1
typedef struct {
    half    d;              // delta
    int8_t  qs[QK8_0];      // quants
} block_q8_0;
static_assert(sizeof(block_q8_0) == sizeof(ggml_fp16_t) + QK8_0, "wrong q8_0 block size/padding");

//================================= k-quants

#define QK_K 256

typedef struct {
    uint8_t scales[QK_K/16]; // scales and mins, quantized with 4 bits
    uint8_t qs[QK_K/4];      // quants
    half d;                  // super-block scale for quantized scales
    half dmin;               // super-block scale for quantized mins
} block_q2_K;
static_assert(sizeof(block_q2_K) == 2*sizeof(ggml_fp16_t) + QK_K/16 + QK_K/4, "wrong q2_K block size/padding");

typedef struct {
    uint8_t hmask[QK_K/8];
    uint8_t qs[QK_K/4]; // nibbles / quants
    uint8_t scales[3*QK_K/64];
    half d;
} block_q3_K;
static_assert(sizeof(block_q3_K) == sizeof(ggml_fp16_t) + QK_K / 4 + 11 * QK_K / 64, "wrong q3_K block size/padding");

typedef struct {
    half d;                    // super-block scale for quantized scales
    half dmin;                 // super-block scale for quantized mins
    uint8_t scales[3*QK_K/64]; // scales, quantized with 6 bits
    uint8_t qs[QK_K/2];        // 4--bit quants
} block_q4_K;
static_assert(sizeof(block_q4_K) == 2*sizeof(ggml_fp16_t) + 3*QK_K/64 + QK_K/2, "wrong q4_K block size/padding");

typedef struct {
    half    d;                   // super-block scale for quantized scales
    half    dmin;                // super-block scale for quantized mins
    uint8_t scales[3*QK_K/64];   // scales, quantized with 6 bits
    uint8_t qh[QK_K/8];          // quants, high bit
    uint8_t qs[QK_K/2];          // quants, low 4 bits
} block_q5_K;
static_assert(sizeof(block_q5_K) == 2*sizeof(ggml_fp16_t) + 3*QK_K/64 + QK_K/2 + QK_K/8, "wrong q5_K block size/padding");

typedef struct {
    uint8_t ql[QK_K/2];   // quants, lower 4 bits
    uint8_t qh[QK_K/4];   // quants, upper 2 bits
    int8_t  scales[QK_K/16]; // scales
    half    d;         // delta
} block_q6_K;
static_assert(sizeof(block_q6_K) == sizeof(ggml_fp16_t) + 13*QK_K/16, "wrong q6_K block size/padding");

#define WARP_SIZE 32

#define CUDA_ADD_BLOCK_SIZE 256
#define CUDA_MUL_BLOCK_SIZE 256
#define CUDA_SILU_BLOCK_SIZE 256
#define CUDA_CPY_BLOCK_SIZE 32
#define CUDA_SCALE_BLOCK_SIZE 256
#define CUDA_ROPE_BLOCK_SIZE 256
#define CUDA_DIAG_MASK_INF_BLOCK_SIZE 32
#define CUDA_DEQUANTIZE_BLOCK_SIZE 256

// dmmv = dequantize_mul_mat_vec
#ifndef GGML_CUDA_DMMV_X
#define GGML_CUDA_DMMV_X 32
#endif
#ifndef GGML_CUDA_DMMV_Y
#define GGML_CUDA_DMMV_Y 1
#endif

#ifndef K_QUANTS_PER_ITERATION
#define K_QUANTS_PER_ITERATION 2
#else
static_assert(K_QUANTS_PER_ITERATION == 1 || K_QUANTS_PER_ITERATION == 2, "K_QUANTS_PER_ITERATION must be 1 or 2");
#endif

static __global__ void add_f32(const float * x, const float * y, float * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = x[i] + y[i];
}

static __global__ void mul_f32(const float * x, const float * y, float * dst, const int kx, const int ky) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= kx) {
        return;
    }
    dst[i] = x[i] * y[i%ky];
}

static __global__ void silu_f32(const float * x, float * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = x[i] / (1.0f + expf(-x[i]));
}

static __global__ void rms_norm_f32(const float * x, float * dst, const int ncols) {
    const int row = blockIdx.x*blockDim.y + threadIdx.y;
    const int tid = threadIdx.x;

    const float eps = 1e-6;

    float tmp = 0.0f; // partial sum for thread in warp

    for (int i = 0; i < ncols; i += WARP_SIZE) {
        const int col = i + tid;
        const float xi = x[row*ncols + col];
        tmp += xi * xi;
    }

    // sum up partial sums
    __syncthreads();
#pragma unroll
    for (int mask = 16; mask > 0; mask >>= 1) {
        tmp += __shfl_xor_sync(0xffffffff, tmp, mask, 32);
    }

    const float mean = tmp / ncols;
    const float scale = 1.0f / sqrtf(mean + eps);

    for (int i = 0; i < ncols; i += WARP_SIZE) {
        const int col = i + tid;
        dst[row*ncols + col] = scale * x[row*ncols + col];
    }
}

static __device__ __forceinline__ void dequantize_q4_0(const void * vx, const int ib, const int iqs, dfloat2 & v){
    const block_q4_0 * x = (const block_q4_0 *) vx;

    const dfloat d = x[ib].d;

    const int vui = x[ib].qs[iqs];

    v.x = vui & 0xF;
    v.y = vui >> 4;

#ifdef GGML_CUDA_DMMV_F16
    v = __hsub2(v, {8.0f, 8.0f});
    v = __hmul2(v, {d, d});
#else
    v.x = (v.x - 8.0f) * d;
    v.y = (v.y - 8.0f) * d;
#endif // GGML_CUDA_DMMV_F16
}

static __device__ __forceinline__ void dequantize_q4_1(const void * vx, const int ib, const int iqs, dfloat2 & v){
    const block_q4_1 * x = (const block_q4_1 *) vx;

    const dfloat d = x[ib].d;
    const dfloat m = x[ib].m;

    const int vui = x[ib].qs[iqs];

    v.x = vui & 0xF;
    v.y = vui >> 4;

#ifdef GGML_CUDA_DMMV_F16
    v = __hmul2(v, {d, d});
    v = __hadd2(v, {m, m});
#else
    v.x = (v.x * d) + m;
    v.y = (v.y * d) + m;
#endif // GGML_CUDA_DMMV_F16
}

static __device__ __forceinline__ void dequantize_q5_0(const void * vx, const int ib, const int iqs, dfloat2 & v){
    const block_q5_0 * x = (const block_q5_0 *) vx;

    const dfloat d = x[ib].d;

    uint32_t qh;
    memcpy(&qh, x[ib].qh, sizeof(qh));

    const int xh_0 = ((qh >> (iqs +  0)) << 4) & 0x10;
    const int xh_1 = ((qh >> (iqs + 12))     ) & 0x10;

    v.x = ((x[ib].qs[iqs] & 0xf) | xh_0);
    v.y = ((x[ib].qs[iqs] >>  4) | xh_1);

#ifdef GGML_CUDA_DMMV_F16
    v = __hsub2(v, {16.0f, 16.0f});
    v = __hmul2(v, {d, d});
#else
    v.x = (v.x - 16.0f) * d;
    v.y = (v.y - 16.0f) * d;
#endif // GGML_CUDA_DMMV_F16
}

static __device__ __forceinline__ void dequantize_q5_1(const void * vx, const int ib, const int iqs, dfloat2 & v){
    const block_q5_1 * x = (const block_q5_1 *) vx;

    const dfloat d = x[ib].d;
    const dfloat m = x[ib].m;

    uint32_t qh;
    memcpy(&qh, x[ib].qh, sizeof(qh));

    const int xh_0 = ((qh >> (iqs +  0)) << 4) & 0x10;
    const int xh_1 = ((qh >> (iqs + 12))     ) & 0x10;

    v.x = ((x[ib].qs[iqs] & 0xf) | xh_0);
    v.y = ((x[ib].qs[iqs] >>  4) | xh_1);

#ifdef GGML_CUDA_DMMV_F16
    v = __hmul2(v, {d, d});
    v = __hadd2(v, {m, m});
#else
    v.x = (v.x * d) + m;
    v.y = (v.y * d) + m;
#endif // GGML_CUDA_DMMV_F16
}

static __device__ __forceinline__ void dequantize_q8_0(const void * vx, const int ib, const int iqs, dfloat2 & v){
    const block_q8_0 * x = (const block_q8_0 *) vx;

    const dfloat d = x[ib].d;

    v.x = x[ib].qs[iqs + 0];
    v.y = x[ib].qs[iqs + 1];

#ifdef GGML_CUDA_DMMV_F16
    v = __hmul2(v, {d, d});
#else
    v.x *= d;
    v.y *= d;
#endif // GGML_CUDA_DMMV_F16
}

//================================== k-quants

static __global__ void dequantize_block_q2_K(const void * vx, float * yy) {

    const int i   = blockIdx.x;
    const int tid = threadIdx.x;
    const int n   = tid/32;
    const int l   = tid - 32*n;
    const int is  = 8*n + l/16;

    const block_q2_K * x = (const block_q2_K *) vx;

    const uint8_t q = x[i].qs[32*n + l];
    float * y = yy + i*QK_K + 128*n;

    float dall = x[i].d;
    float dmin = x[i].dmin;
    y[l+ 0] = dall * (x[i].scales[is+0] & 0xF) * ((q >> 0) & 3) - dmin * (x[i].scales[is+0] >> 4);
    y[l+32] = dall * (x[i].scales[is+2] & 0xF) * ((q >> 2) & 3) - dmin * (x[i].scales[is+2] >> 4);
    y[l+64] = dall * (x[i].scales[is+4] & 0xF) * ((q >> 4) & 3) - dmin * (x[i].scales[is+4] >> 4);
    y[l+96] = dall * (x[i].scales[is+6] & 0xF) * ((q >> 6) & 3) - dmin * (x[i].scales[is+6] >> 4);

}

static __global__ void dequantize_block_q3_K(const void * vx, float * yy) {

    int r = threadIdx.x/4;
    int i = blockIdx.x;
    int tid = r/2;
    int is0 = r%2;
    int l0 = 16*is0 + 4*(threadIdx.x%4);
    int n = tid / 4;
    int j = tid - 4*n;

    const block_q3_K * x = (const block_q3_K *) vx;

    uint8_t m = 1 << (4*n + j);
    int is = 8*n + 2*j + is0;
    int shift = 2*j;

    int8_t us = is <  4 ? (x[i].scales[is-0] & 0xF) | (((x[i].scales[is+8] >> 0) & 3) << 4) :
                is <  8 ? (x[i].scales[is-0] & 0xF) | (((x[i].scales[is+4] >> 2) & 3) << 4) :
                is < 12 ? (x[i].scales[is-8] >>  4) | (((x[i].scales[is+0] >> 4) & 3) << 4) :
                          (x[i].scales[is-8] >>  4) | (((x[i].scales[is-4] >> 6) & 3) << 4);
    float d_all = x[i].d;
    float dl = d_all * (us - 32);

    float * y = yy + i*QK_K + 128*n + 32*j;
    const uint8_t * q = x[i].qs + 32*n;
    const uint8_t * hm = x[i].hmask;

    for (int l = l0; l < l0+4; ++l) y[l] = dl * ((int8_t)((q[l] >> shift) & 3) - ((hm[l] & m) ? 0 : 4));

}

static inline __device__ void get_scale_min_k4(int j, const uint8_t * q, uint8_t & d, uint8_t & m) {
    if (j < 4) {
        d = q[j] & 63; m = q[j + 4] & 63;
    } else {
        d = (q[j+4] & 0xF) | ((q[j-4] >> 6) << 4);
        m = (q[j+4] >>  4) | ((q[j-0] >> 6) << 4);
    }
}

static __global__ void dequantize_block_q4_K(const void * vx, float * yy) {
    const block_q4_K * x = (const block_q4_K *) vx;

    const int i = blockIdx.x;

    //// assume 64 threads - this is very slightly better than the one below
    //const int tid = threadIdx.x;
    //const int il  = tid/16;
    //const int ir  = tid%16;
    //const int is  = 2*il;
    //const int n   = 2;

    // assume 32 threads
    const int tid = threadIdx.x;
    const int il  = tid/8;
    const int ir  = tid%8;
    const int is  = 2*il;
    const int n   = 4;

    float * y = yy + i*QK_K + 64*il + n*ir;

    const float dall = x[i].d;
    const float dmin = x[i].dmin;

    const uint8_t * q = x[i].qs + 32*il + n*ir;

    uint8_t sc, m;
    get_scale_min_k4(is + 0, x[i].scales, sc, m);
    const float d1 = dall * sc; const float m1 = dmin * m;
    get_scale_min_k4(is + 1, x[i].scales, sc, m);
    const float d2 = dall * sc; const float m2 = dmin * m;
    for (int l = 0; l < n; ++l) {
        y[l + 0] = d1 * (q[l] & 0xF) - m1;
        y[l +32] = d2 * (q[l] >>  4) - m2;
    }
}

static __global__ void dequantize_block_q5_K(const void * vx, float * yy) {
    const block_q5_K * x = (const block_q5_K *) vx;

    const int i = blockIdx.x;

    // assume 64 threads - this is very slightly better than the one below
    const int tid = threadIdx.x;
    const int il  = tid/16;   // il is in 0...3
    const int ir  = tid%16;   // ir is in 0...15
    const int is  = 2*il;     // is is in 0...6

    float * y = yy + i*QK_K + 64*il + 2*ir;

    const float dall = x[i].d;
    const float dmin = x[i].dmin;

    const uint8_t * ql = x[i].qs + 32*il + 2*ir;
    const uint8_t * qh = x[i].qh + 2*ir;

    uint8_t sc, m;
    get_scale_min_k4(is + 0, x[i].scales, sc, m);
    const float d1 = dall * sc; const float m1 = dmin * m;
    get_scale_min_k4(is + 1, x[i].scales, sc, m);
    const float d2 = dall * sc; const float m2 = dmin * m;

    uint8_t   hm  = 1 << (2*il);
    y[ 0] = d1 * ((ql[ 0] & 0xF) + (qh[ 0] & hm ? 16 : 0)) - m1;
    y[ 1] = d1 * ((ql[ 1] & 0xF) + (qh[ 1] & hm ? 16 : 0)) - m1;
    hm <<= 1;
    y[32] = d2 * ((ql[ 0] >>  4) + (qh[ 0] & hm ? 16 : 0)) - m2;
    y[33] = d2 * ((ql[ 1] >>  4) + (qh[ 1] & hm ? 16 : 0)) - m2;
}

static __global__ void dequantize_block_q6_K(const void * vx, float * yy) {
    const block_q6_K * x = (const block_q6_K *) vx;

    const int i = blockIdx.x;

    // assume 64 threads - this is very slightly better than the one below
    const int tid = threadIdx.x;
    const int ip  = tid/32;   // ip is 0 or 1
    const int il  = tid - 32*ip; // 0...32
    const int is  = 8*ip + il/16;

    float * y = yy + i*QK_K + 128*ip + il;

    const float d = x[i].d;

    const uint8_t * ql = x[i].ql + 64*ip + il;
    const uint8_t   qh = x[i].qh[32*ip + il];
    const int8_t  * sc = x[i].scales + is;

    y[ 0] = d * sc[0] * ((int8_t)((ql[ 0] & 0xF) | (((qh >> 0) & 3) << 4)) - 32);
    y[32] = d * sc[2] * ((int8_t)((ql[32] & 0xF) | (((qh >> 2) & 3) << 4)) - 32);
    y[64] = d * sc[4] * ((int8_t)((ql[ 0]  >> 4) | (((qh >> 4) & 3) << 4)) - 32);
    y[96] = d * sc[6] * ((int8_t)((ql[32]  >> 4) | (((qh >> 6) & 3) << 4)) - 32);
}

static __global__ void dequantize_mul_mat_vec_q2_k(const void * vx, const float * yy, float * dst, const int ncols, int nrows) {

    static_assert(16%K_QUANTS_PER_ITERATION == 0, "16 must be divisible by K_QUANTS_PER_ITERATION");

    const int row = blockIdx.y*blockDim.y + threadIdx.y;
    if (row > nrows) return;

    const int num_blocks_per_row = ncols / QK_K;
    const int ib0 = row*num_blocks_per_row;

    const block_q2_K * x = (const block_q2_K *)vx + ib0;

    const int tid = threadIdx.x/K_QUANTS_PER_ITERATION;  // 0...31 or 0...15
    const int ix  = threadIdx.x%K_QUANTS_PER_ITERATION;  // 0 or 0,1

    const int step = 16/K_QUANTS_PER_ITERATION;

    const int im = tid/step;                             // 0 or 1. 0 computes 0..., 1 computes 128...
    const int in = tid - step*im;                        // 0...15 or 0...7

    const int l0 = K_QUANTS_PER_ITERATION*in;            // 0...15 or 0...14 in steps of 2
    const int q_offset = 32*im + l0;
    const int s_offset = 8*im;
    const int y_offset = 128*im + l0;

    float tmp = 0; // partial sum for thread in warp

    uint32_t aux[4];
    const uint8_t * d = (const uint8_t *)aux;
    const uint8_t * m = (const uint8_t *)(aux + 2);

    for (int i = ix; i < num_blocks_per_row; i += K_QUANTS_PER_ITERATION) {

        const float   * y = yy + i * QK_K + y_offset;
        const uint8_t * q = x[i].qs + q_offset;

        const float dall = x[i].d;
        const float dmin = x[i].dmin;

        const uint32_t * a = (const uint32_t *)(x[i].scales + s_offset);
        aux[0] = a[0] & 0x0f0f0f0f;
        aux[1] = a[1] & 0x0f0f0f0f;
        aux[2] = (a[0] >> 4) & 0x0f0f0f0f;
        aux[3] = (a[1] >> 4) & 0x0f0f0f0f;

        float sum1 = 0, sum2 = 0;
        for (int l = 0; l < K_QUANTS_PER_ITERATION; ++l) {
            sum1 += y[l+ 0] * d[0] * ((q[l+ 0] >> 0) & 3)
                  + y[l+32] * d[2] * ((q[l+ 0] >> 2) & 3)
                  + y[l+64] * d[4] * ((q[l+ 0] >> 4) & 3)
                  + y[l+96] * d[6] * ((q[l+ 0] >> 6) & 3)
                  + y[l+16] * d[1] * ((q[l+16] >> 0) & 3)
                  + y[l+48] * d[3] * ((q[l+16] >> 2) & 3)
                  + y[l+80] * d[5] * ((q[l+16] >> 4) & 3)
                  +y[l+112] * d[7] * ((q[l+16] >> 6) & 3);
            sum2 += y[l+ 0] * m[0] + y[l+32] * m[2] + y[l+64] * m[4] + y[ l+96] * m[6]
                  + y[l+16] * m[1] + y[l+48] * m[3] + y[l+80] * m[5] + y[l+112] * m[7];

        }
        tmp += dall * sum1 - dmin * sum2;

    }

    // sum up partial sums and write back result
    __syncthreads();
#pragma unroll
    for (int mask = 16; mask > 0; mask >>= 1) {
        tmp += __shfl_xor_sync(0xffffffff, tmp, mask, 32);
    }

    if (tid == 0) {
        dst[row] = tmp;
    }
}

static __global__ void dequantize_mul_mat_vec_q3_k(const void * vx, const float * yy, float * dst, const int ncols, int nrows) {

    const uint16_t kmask1 = 0x0303;
    const uint16_t kmask2 = 0x0f0f;

    const int row = blockIdx.y*blockDim.y + threadIdx.y;
    if (row > nrows) return;

    const int num_blocks_per_row = ncols / QK_K;
    const int ib0 = row*num_blocks_per_row;

    const block_q3_K * x = (const block_q3_K *)vx + ib0;

    const int tid = threadIdx.x/K_QUANTS_PER_ITERATION;  // 0...31 or 0...16
    const int ix  = threadIdx.x%K_QUANTS_PER_ITERATION;  // 0 or 0,1

    const int n  = K_QUANTS_PER_ITERATION;               // iterations in the inner loop
    const int step = 16/K_QUANTS_PER_ITERATION;
    const int im = tid/step;                             // 0 or 1. 0 computes 0..., 1 computes 128...
    const int in = tid - step*im;                        // 0....15 or 0...7

    const uint8_t m = 1 << (4*im);

    const int l0 = n*in;                                 // 0...15 or 0...14 in steps of 2
    const int q_offset =  32*im + l0;
    const int y_offset = 128*im + l0;

    uint16_t utmp[4];
    const int8_t * s = (const int8_t *)utmp;

    const uint16_t s_shift = 4*im;

    float tmp = 0; // partial sum for thread in warp

    for (int i = ix; i < num_blocks_per_row; i += K_QUANTS_PER_ITERATION) {

        const float   * y  = yy + i * QK_K + y_offset;
        const uint8_t * q = x[i].qs + q_offset;
        const uint8_t * h = x[i].hmask + l0;

        const uint16_t * a = (const uint16_t *)x[i].scales;
        utmp[0] = ((a[0] >> s_shift) & kmask2) | (((a[4] >> (s_shift + 0)) & kmask1) << 4);
        utmp[1] = ((a[1] >> s_shift) & kmask2) | (((a[5] >> (s_shift + 0)) & kmask1) << 4);
        utmp[2] = ((a[2] >> s_shift) & kmask2) | (((a[4] >> (s_shift + 2)) & kmask1) << 4);
        utmp[3] = ((a[3] >> s_shift) & kmask2) | (((a[5] >> (s_shift + 2)) & kmask1) << 4);

        const float d = x[i].d;

        float sum = 0;
        for (int l = 0; l < n; ++l) {
            sum += y[l+ 0] * (s[0] - 32) * (((q[l] >> 0) & 3) - (h[l] & (m << 0) ? 0 : 4))
                 + y[l+32] * (s[2] - 32) * (((q[l] >> 2) & 3) - (h[l] & (m << 1) ? 0 : 4))
                 + y[l+64] * (s[4] - 32) * (((q[l] >> 4) & 3) - (h[l] & (m << 2) ? 0 : 4))
                 + y[l+96] * (s[6] - 32) * (((q[l] >> 6) & 3) - (h[l] & (m << 3) ? 0 : 4));
            sum += y[l+16] * (s[1] - 32) * (((q[l+16] >> 0) & 3) - (h[l+16] & (m << 0) ? 0 : 4))
                 + y[l+48] * (s[3] - 32) * (((q[l+16] >> 2) & 3) - (h[l+16] & (m << 1) ? 0 : 4))
                 + y[l+80] * (s[5] - 32) * (((q[l+16] >> 4) & 3) - (h[l+16] & (m << 2) ? 0 : 4))
                + y[l+112] * (s[7] - 32) * (((q[l+16] >> 6) & 3) - (h[l+16] & (m << 3) ? 0 : 4));
        }
        tmp += d * sum;

    }

    // sum up partial sums and write back result
    __syncthreads();
#pragma unroll
    for (int mask = 16; mask > 0; mask >>= 1) {
        tmp += __shfl_xor_sync(0xffffffff, tmp, mask, 32);
    }

    if (tid == 0) {
        dst[row] = tmp;
    }
}

static __global__ void dequantize_mul_mat_vec_q4_k(const void * vx, const float * yy, float * dst, const int ncols, int nrows) {

    const uint16_t kmask1 = 0x3f3f;
    const uint16_t kmask2 = 0x0f0f;
    const uint16_t kmask3 = 0xc0c0;

    const int row = blockIdx.y*blockDim.y + threadIdx.y;
    if (row > nrows) return;
    const int num_blocks_per_row = ncols / QK_K;
    const int ib0 = row*num_blocks_per_row;

    const int tid = threadIdx.x/K_QUANTS_PER_ITERATION;  // 0...31 or 0...16
    const int ix  = threadIdx.x%K_QUANTS_PER_ITERATION;  // 0 or 0,1

    const int step = 8/K_QUANTS_PER_ITERATION;           // 8 or 4

    const int il  = tid/step;                            // 0...3
    const int ir  = tid - step*il;                       // 0...7 or 0...3
    const int n   = 2 * K_QUANTS_PER_ITERATION;          // 2 or 4

    const int im = il/2;  // 0 or 1. 0 computes 0,32 + 128,160, 1 computes 64,96 + 192,224
    const int in = il%2;

    const int l0 = n*(2*ir + in);
    const int q_offset = 32*im + l0;
    const int y_offset = 64*im + l0;

    uint16_t aux[4];
    const uint8_t * sc = (const uint8_t *)aux;

    const block_q4_K * x = (const block_q4_K *)vx + ib0;

    float tmp = 0; // partial sum for thread in warp

    for (int i = ix; i < num_blocks_per_row; i += K_QUANTS_PER_ITERATION) {

        const uint8_t * q1 = x[i].qs + q_offset;
        const uint8_t * q2 = q1 + 64;
        const float   * y1 = yy + i*QK_K + y_offset;
        const float   * y2 = y1 + 128;

        const float dall = x[i].d;
        const float dmin = x[i].dmin;

        const uint16_t * a = (const uint16_t *)x[i].scales;
        aux[0] = a[im+0] & kmask1;
        aux[1] = a[im+2] & kmask1;
        aux[2] = ((a[im+4] >> 0) & kmask2) | ((a[im+0] & kmask3) >> 2);
        aux[3] = ((a[im+4] >> 4) & kmask2) | ((a[im+2] & kmask3) >> 2);

        float4 s = {0.f, 0.f, 0.f, 0.f};
        float smin = 0;
        for (int l = 0; l < n; ++l) {
            s.x += y1[l] * (q1[l] & 0xF); s.y += y1[l+32] * (q1[l] >> 4);
            s.z += y2[l] * (q2[l] & 0xF); s.w += y2[l+32] * (q2[l] >> 4);
            smin += y1[l] * sc[2] + y1[l+32] * sc[3] + y2[l] * sc[6] + y2[l+32] * sc[7];
        }
        tmp += dall * (s.x * sc[0] + s.y * sc[1] + s.z * sc[4] + s.w * sc[5]) - dmin * smin;

    }

    // sum up partial sums and write back result
    __syncthreads();
#pragma unroll
    for (int mask = 16; mask > 0; mask >>= 1) {
        tmp += __shfl_xor_sync(0xffffffff, tmp, mask, 32);
    }

    if (tid == 0) {
        dst[row] = tmp;
    }
}

static __global__ void dequantize_mul_mat_vec_q5_k(const void * vx, const float * yy, float * dst, const int ncols) {

    const uint16_t kmask1 = 0x3f3f;
    const uint16_t kmask2 = 0x0f0f;
    const uint16_t kmask3 = 0xc0c0;

    //const int row = blockIdx.x*blockDim.y + threadIdx.y;
    const int row = blockIdx.x;
    const int num_blocks_per_row = ncols / QK_K;
    const int ib0 = row*num_blocks_per_row;

    const int tid = threadIdx.x/2;  // 0...15
    const int ix  = threadIdx.x%2;

    const int il  = tid/4;     // 0...3
    const int ir  = tid - 4*il;// 0...3
    const int n   = 2;

    const int im = il/2;  // 0 or 1. 0 computes 0,32 + 128,160, 1 computes 64,96 + 192,224
    const int in = il%2;

    const int l0 = n*(2*ir + in);
    const int q_offset = 32*im + l0;
    const int y_offset = 64*im + l0;

    const uint8_t hm1  = 1 << (2*im);
    const uint8_t hm2  = hm1 << 4;

    uint16_t aux[4];
    const uint8_t * sc = (const uint8_t *)aux;

    const block_q5_K * x = (const block_q5_K *)vx + ib0;

    float tmp = 0; // partial sum for thread in warp

    for (int i = ix; i < num_blocks_per_row; i += 2) {

        const uint8_t * ql1 = x[i].qs + q_offset;
        const uint8_t * ql2 = ql1 + 64;
        const uint8_t * qh  = x[i].qh + l0;
        const float   * y1  = yy + i*QK_K + y_offset;
        const float   * y2  = y1 + 128;

        const float dall = x[i].d;
        const float dmin = x[i].dmin;

        const uint16_t * a = (const uint16_t *)x[i].scales;
        aux[0] = a[im+0] & kmask1;
        aux[1] = a[im+2] & kmask1;
        aux[2] = ((a[im+4] >> 0) & kmask2) | ((a[im+0] & kmask3) >> 2);
        aux[3] = ((a[im+4] >> 4) & kmask2) | ((a[im+2] & kmask3) >> 2);

        float4 sum = {0.f, 0.f, 0.f, 0.f};
        float smin = 0;
        for (int l = 0; l < n; ++l) {
            sum.x += y1[l+ 0] * ((ql1[l+ 0] & 0xF) + (qh[l+ 0] & (hm1 << 0) ? 16 : 0))
                   + y1[l+16] * ((ql1[l+16] & 0xF) + (qh[l+16] & (hm1 << 0) ? 16 : 0));
            sum.y += y1[l+32] * ((ql1[l+ 0] >>  4) + (qh[l+ 0] & (hm1 << 1) ? 16 : 0))
                   + y1[l+48] * ((ql1[l+16] >>  4) + (qh[l+16] & (hm1 << 1) ? 16 : 0));
            sum.z += y2[l+ 0] * ((ql2[l+ 0] & 0xF) + (qh[l+ 0] & (hm2 << 0) ? 16 : 0))
                   + y2[l+16] * ((ql2[l+16] & 0xF) + (qh[l+16] & (hm2 << 0) ? 16 : 0));
            sum.w += y2[l+32] * ((ql2[l+ 0] >>  4) + (qh[l+ 0] & (hm2 << 1) ? 16 : 0))
                   + y2[l+48] * ((ql2[l+16] >>  4) + (qh[l+16] & (hm2 << 1) ? 16 : 0));
            smin += (y1[l] + y1[l+16]) * sc[2] + (y1[l+32] + y1[l+48]) * sc[3]
                  + (y2[l] + y2[l+16]) * sc[6] + (y2[l+32] + y2[l+48]) * sc[7];
        }
        tmp += dall * (sum.x * sc[0] + sum.y * sc[1] + sum.z * sc[4] + sum.w * sc[5]) - dmin * smin;

    }

    // sum up partial sums and write back result
    __syncthreads();
#pragma unroll
    for (int mask = 16; mask > 0; mask >>= 1) {
        tmp += __shfl_xor_sync(0xffffffff, tmp, mask, 32);
    }

    if (tid == 0) {
        dst[row] = tmp;
    }
}

static __global__ void dequantize_mul_mat_vec_q6_k(const void * vx, const float * yy, float * dst, const int ncols, int nrows) {

    static_assert(16%K_QUANTS_PER_ITERATION == 0, "16 must be divisible by K_QUANTS_PER_ITERATION");

    const int row = blockIdx.y*blockDim.y + threadIdx.y;
    if (row > nrows) return;

    const int num_blocks_per_row = ncols / QK_K;
    const int ib0 = row*num_blocks_per_row;

    const block_q6_K * x = (const block_q6_K *)vx + ib0;

    const int tid = threadIdx.x/K_QUANTS_PER_ITERATION;  // 0...31 or 0...16
    const int ix  = threadIdx.x%K_QUANTS_PER_ITERATION;  // 0 or 0, 1

    const int step = 16/K_QUANTS_PER_ITERATION;          // 16 or 8

    const int im = tid/step;                             // 0 or 1. 0 computes 0..., 1 computes 128...
    const int in = tid - step*im;                        // 0...15 or 0...7

#if K_QUANTS_PER_ITERATION == 1
    const int l0 = K_QUANTS_PER_ITERATION*in;            // 0...15
    const int is = 0;
#else
    const int l0 = 4 * in;                               // 0, 4, 8, ..., 28
    const int is = in / 4;
#endif
    const int ql_offset = 64*im + l0;
    const int qh_offset = 32*im + l0;
    const int s_offset  =  8*im + is;
    const int y_offset = 128*im + l0;

    float tmp = 0; // partial sum for thread in warp

    for (int i = ix; i < num_blocks_per_row; i += K_QUANTS_PER_ITERATION) {

        const float   * y  = yy + i * QK_K + y_offset;
        const uint8_t * ql = x[i].ql + ql_offset;
        const uint8_t * qh = x[i].qh + qh_offset;
        const int8_t  * s  = x[i].scales + s_offset;

        const float d = x[i].d;

#if K_QUANTS_PER_ITERATION == 1
        float sum = y[ 0] * s[0] * d * ((int8_t)((ql[ 0] & 0xF) | ((qh[ 0] & 0x03) << 4)) - 32)
                  + y[16] * s[1] * d * ((int8_t)((ql[16] & 0xF) | ((qh[16] & 0x03) << 4)) - 32)
                  + y[32] * s[2] * d * ((int8_t)((ql[32] & 0xF) | ((qh[ 0] & 0x0c) << 2)) - 32)
                  + y[48] * s[3] * d * ((int8_t)((ql[48] & 0xF) | ((qh[16] & 0x0c) << 2)) - 32)
                  + y[64] * s[4] * d * ((int8_t)((ql[ 0]  >> 4) | ((qh[ 0] & 0x30) >> 0)) - 32)
                  + y[80] * s[5] * d * ((int8_t)((ql[16]  >> 4) | ((qh[16] & 0x30) >> 0)) - 32)
                  + y[96] * s[6] * d * ((int8_t)((ql[32]  >> 4) | ((qh[ 0] & 0xc0) >> 2)) - 32)
                  +y[112] * s[7] * d * ((int8_t)((ql[48]  >> 4) | ((qh[16] & 0xc0) >> 2)) - 32);
        tmp += sum;
#else
        float sum = 0;
        for (int l = 0; l < 4; ++l) {
            sum += y[l+ 0] * s[0] * d * ((int8_t)((ql[l+ 0] & 0xF) | (((qh[l] >> 0) & 3) << 4)) - 32)
                 + y[l+32] * s[2] * d * ((int8_t)((ql[l+32] & 0xF) | (((qh[l] >> 2) & 3) << 4)) - 32)
                 + y[l+64] * s[4] * d * ((int8_t)((ql[l+ 0]  >> 4) | (((qh[l] >> 4) & 3) << 4)) - 32)
                 + y[l+96] * s[6] * d * ((int8_t)((ql[l+32]  >> 4) | (((qh[l] >> 6) & 3) << 4)) - 32);
        }
        tmp += sum;
#endif

    }

    // sum up partial sums and write back result
    __syncthreads();
#pragma unroll
    for (int mask = 16; mask > 0; mask >>= 1) {
        tmp += __shfl_xor_sync(0xffffffff, tmp, mask, 32);
    }

    if (tid == 0) {
        dst[row] = tmp;
    }
}

static __device__ void convert_f16(const void * vx, const int ib, const int iqs, dfloat2 & v){
    const half * x = (const half *) vx;

    // automatic half -> float type cast if dfloat == float
    v.x = x[ib + iqs + 0];
    v.y = x[ib + iqs + 1];
}

template <int qk, int qr, dequantize_kernel_t dequantize_kernel>
static __global__ void dequantize_block(const void * vx, float * y, const int k) {
    const int i = blockDim.x*blockIdx.x + 2*threadIdx.x;

    if (i >= k) {
        return;
    }

    const int ib = i/qk; // block index
    const int iqs = (i%qk)/qr; // quant index
    const int iybs = i - i%qk; // y block start index
    const int y_offset = qr == 1 ? 1 : qk/2;

    // dequantize
    dfloat2 v;
    dequantize_kernel(vx, ib, iqs, v);

    y[iybs + iqs + 0]        = v.x;
    y[iybs + iqs + y_offset] = v.y;
}

template <int qk, int qr, dequantize_kernel_t dequantize_kernel>
static __global__ void dequantize_mul_mat_vec(const void * vx, const dfloat * y, float * dst, const int ncols, const int nrows) {
    // qk = quantized weights per x block
    // qr = number of quantized weights per data value in x block
    const int row = blockIdx.y*blockDim.y + threadIdx.y;

    if (row >= nrows) {
        return;
    }

    const int tid = threadIdx.x;

    const int iter_stride = 2*GGML_CUDA_DMMV_X;
    const int vals_per_iter = iter_stride / WARP_SIZE; // num quantized vals per thread and i iter
    const int y_offset = qr == 1 ? 1 : qk/2;

// partial sum for each thread
#ifdef GGML_CUDA_DMMV_F16
    half2 tmp = {0.0f, 0.0f}; // two sums for f16 to take advantage of half2 intrinsics
#else
    float tmp = 0.0f;
#endif // GGML_CUDA_DMMV_F16

    for (int i = 0; i < ncols; i += iter_stride) {
        const int col = i + vals_per_iter*tid;
        const int ib = (row*ncols + col)/qk; // x block index
        const int iqs = (col%qk)/qr; // x quant index
        const int iybs = col - col%qk; // y block start index

// processing >2 values per i iter is faster for fast GPUs
#pragma unroll
        for (int j = 0; j < vals_per_iter; j += 2) {
            // process 2 vals per j iter

            // dequantize
            // for qr = 2 the iqs needs to increase by 1 per j iter because 2 weights per data val
            dfloat2 v;
            dequantize_kernel(vx, ib, iqs + j/qr, v);

            // matrix multiplication
            // for qr = 2 the y index needs to increase by 1 per j iter because of y_offset = qk/2
#ifdef GGML_CUDA_DMMV_F16
            tmp += __hmul2(v, {
                y[iybs + iqs + j/qr + 0],
                y[iybs + iqs + j/qr + y_offset]
            });
#else
            tmp += v.x * y[iybs + iqs + j/qr + 0];
            tmp += v.y * y[iybs + iqs + j/qr + y_offset];
#endif // GGML_CUDA_DMMV_F16
        }
    }

    // sum up partial sums and write back result
    __syncthreads();
#pragma unroll
    for (int mask = 16; mask > 0; mask >>= 1) {
        tmp += __shfl_xor_sync(0xffffffff, tmp, mask, 32);
    }

    if (tid == 0) {
#ifdef GGML_CUDA_DMMV_F16
        dst[row] = tmp.x + tmp.y;
#else
        dst[row] = tmp;
#endif // GGML_CUDA_DMMV_F16
    }
}

static __global__ void mul_mat_p021_f16_f32(const void * vx, const float * y, float * dst, const int ncols_x, const int nrows_x, const int nchannels_x) {
    const half * x = (half *) vx;

    const int row_x = blockDim.y*blockIdx.y + threadIdx.y;
    const int channel = blockDim.z*blockIdx.z + threadIdx.z;

    const int nrows_y = ncols_x;
    const int nrows_dst = nrows_x;
    const int row_dst = row_x;

    float tmp = 0.0f;

    for (int col_x0 = 0; col_x0 < ncols_x; col_x0 += blockDim.x) {
        const int col_x = col_x0 + threadIdx.x;

        if (col_x >= ncols_x) {
            break;
        }

        // x is transposed and permuted
        const int ix = row_x*nchannels_x*ncols_x + channel*ncols_x + col_x;
        const float xi = __half2float(x[ix]);

        const int row_y = col_x;


        // y is not transposed but permuted
        const int iy = channel*nrows_y + row_y;

        tmp += xi * y[iy];
    }

    // dst is not transposed and not permuted
    const int idst = channel*nrows_dst + row_dst;

    // sum up partial sums and write back result
    __syncthreads();
#pragma unroll
    for (int mask = 16; mask > 0; mask >>= 1) {
        tmp += __shfl_xor_sync(0xffffffff, tmp, mask, 32);
    }

    if (threadIdx.x == 0) {
        dst[idst] = tmp;
    }
}

static __global__ void mul_mat_vec_nc_f16_f32( // nc == non-contiguous
    const void * vx, const float * y, float * dst, const int ncols_x, const int nrows_x,
    const int row_stride_x, const int nchannels_x, const int channel_stride_x) {

    const half * x = (half *) vx;

    const int row_x = blockDim.y*blockIdx.y + threadIdx.y;
    const int channel = blockDim.z*blockIdx.z + threadIdx.z;

    const int nrows_y = ncols_x;
    const int nrows_dst = nrows_x;
    const int row_dst = row_x;

    const int idst = channel*nrows_dst + row_dst;

    float tmp = 0.0f;

    for (int col_x0 = 0; col_x0 < ncols_x; col_x0 += blockDim.x) {
        const int col_x = col_x0 + threadIdx.x;

        if (col_x >= ncols_x) {
            break;
        }

        const int ix = channel*channel_stride_x + row_x*row_stride_x + col_x;
        const float xi = __half2float(x[ix]);

        const int row_y = col_x;

        const int iy = channel*nrows_y + row_y;

        tmp += xi * y[iy];
    }

    // sum up partial sums and write back result
    __syncthreads();
#pragma unroll
    for (int mask = 16; mask > 0; mask >>= 1) {
        tmp += __shfl_xor_sync(0xffffffff, tmp, mask, 32);
    }

    if (threadIdx.x == 0) {
        dst[idst] = tmp;
    }
}

static __device__ void cpy_1_f32_f32(const char * cxi, char * cdsti) {
    const float * xi = (float *) cxi;
    float * dsti = (float *) cdsti;

    *dsti = *xi;
}

static __device__ void cpy_1_f32_f16(const char * cxi, char * cdsti) {
    const float * xi = (float *) cxi;
    half * dsti = (half *) cdsti;

    *dsti = __float2half(*xi);
}

template <cpy_kernel_t cpy_1>
static __global__ void cpy_f32_f16(const char * cx, char * cdst, const int ne,
                                   const int ne00, const int ne01, const int nb00, const int nb01, const int nb02,
                                   const int ne10, const int ne11, const int nb10, const int nb11, const int nb12) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= ne) {
        return;
    }

    // determine indices i02/i12, i01/i11, i00/i10 as a function of index i of flattened tensor
    // then combine those indices with the corresponding byte offsets to get the total offsets
    const int i02 = i / (ne00*ne01);
    const int i01 = (i - i02*ne01*ne00) / ne00;
    const int i00 = i - i02*ne01*ne00 - i01*ne00;
    const int x_offset = i00*nb00 + i01*nb01 + i02*nb02;

    const int i12 = i / (ne10*ne11);
    const int i11 = (i - i12*ne10*ne11) / ne10;
    const int i10 = i - i12*ne10*ne11 - i11*ne10;
    const int dst_offset = i10*nb10 + i11*nb11 + i12*nb12;

    cpy_1(cx + x_offset, cdst + dst_offset);
}

// rope == RoPE == rotary positional embedding
static __global__ void rope_f32(const float * x, float * dst, const int ncols, const float p, const float theta_scale) {
    const int col = 2*(blockDim.x*blockIdx.x + threadIdx.x);

    if (col >= ncols) {
        return;
    }

    const int row = blockDim.y*blockIdx.y + threadIdx.y;
    const int i = row*ncols + col;

    const float theta = p*powf(theta_scale, col/2);
    const float sin_theta = sinf(theta);
    const float cos_theta = cosf(theta);

    const float x0 = x[i + 0];
    const float x1 = x[i + 1];

    dst[i + 0] = x0*cos_theta - x1*sin_theta;
    dst[i + 1] = x0*sin_theta + x1*cos_theta;
}

static __global__ void diag_mask_inf_f32(const float * x, float * dst, const int ncols, const int rows_per_channel, const int n_past) {
    const int col = blockDim.x*blockIdx.x + threadIdx.x;
    const int row = blockDim.y*blockIdx.y + threadIdx.y;

    if (col >= ncols) {
        return;
    }

    const int i = row*ncols + col;
    // dst[i] = col > n_past + row ? -INFINITY : x[i];
    dst[i] = x[i] - (col > n_past + row % rows_per_channel) * INT_MAX; // equivalent within rounding error but slightly faster on GPU
}

// the CUDA soft max implementation differs from the CPU implementation
// instead of doubles floats are used
// values are also not normalized to the maximum value by subtracting it in the exponential function
// theoretically these changes could cause problems with rounding error and arithmetic overflow but for LLaMa it seems to be fine
static __global__ void soft_max_f32(const float * x, float * dst, const int ncols) {
    const int row = blockDim.y*blockIdx.y + threadIdx.y;
    const int block_size = blockDim.x;
    const int tid = threadIdx.x;

    float tmp = 0.0;

    for (int block_start = 0; block_start < ncols; block_start += block_size) {
        const int col = block_start + tid;

        if (col >= ncols) {
            break;
        }

        const int i = row*ncols + col;
        const float val = expf(x[i]);
        tmp += val;
        dst[i] = val;
    }

    // sum up partial sums
    __syncthreads();
#pragma unroll
    for (int mask = 16; mask > 0; mask >>= 1) {
        tmp += __shfl_xor_sync(0xffffffff, tmp, mask, 32);
    }

    for (int block_start = 0; block_start < ncols; block_start += block_size) {
        const int col = block_start + tid;

        if (col >= ncols) {
            break;
        }

        const int i = row*ncols + col;
        dst[i] /= tmp;
    }
}

static __global__ void scale_f32(const float * x, float * dst, const float scale, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }

    dst[i] = scale * x[i];
}

static void add_f32_cuda(const float * x, const float * y, float * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_ADD_BLOCK_SIZE - 1) / CUDA_ADD_BLOCK_SIZE;
    add_f32<<<num_blocks, CUDA_ADD_BLOCK_SIZE, 0, stream>>>(x, y, dst, k);
}

static void mul_f32_cuda(const float * x, const float * y, float * dst, const int kx, const int ky, hipStream_t stream) {
    const int num_blocks = (kx + CUDA_MUL_BLOCK_SIZE - 1) / CUDA_MUL_BLOCK_SIZE;
    mul_f32<<<num_blocks, CUDA_MUL_BLOCK_SIZE, 0, stream>>>(x, y, dst, kx, ky);
}

static void silu_f32_cuda(const float * x, float * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_SILU_BLOCK_SIZE - 1) / CUDA_SILU_BLOCK_SIZE;
    silu_f32<<<num_blocks, CUDA_SILU_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

static void rms_norm_f32_cuda(const float * x, float * dst, const int ncols, const int nrows, hipStream_t stream) {
    GGML_ASSERT(ncols % WARP_SIZE == 0);
    const dim3 block_dims(WARP_SIZE, 1, 1);
    rms_norm_f32<<<nrows, block_dims, 0, stream>>>(x, dst, ncols);
}

static void dequantize_row_q4_0_cuda(const void * vx, float * y, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_DEQUANTIZE_BLOCK_SIZE - 1) / CUDA_DEQUANTIZE_BLOCK_SIZE;
    dequantize_block<QK4_0, QR4_0, dequantize_q4_0><<<num_blocks, CUDA_DEQUANTIZE_BLOCK_SIZE, 0, stream>>>(vx, y, k);
}

static void dequantize_row_q4_1_cuda(const void * vx, float * y, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_DEQUANTIZE_BLOCK_SIZE - 1) / CUDA_DEQUANTIZE_BLOCK_SIZE;
    dequantize_block<QK4_1, QR4_1, dequantize_q4_1><<<num_blocks, CUDA_DEQUANTIZE_BLOCK_SIZE, 0, stream>>>(vx, y, k);
}

static void dequantize_row_q5_0_cuda(const void * vx, float * y, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_DEQUANTIZE_BLOCK_SIZE - 1) / CUDA_DEQUANTIZE_BLOCK_SIZE;
    dequantize_block<QK5_0, QR5_0, dequantize_q5_0><<<num_blocks, CUDA_DEQUANTIZE_BLOCK_SIZE, 0, stream>>>(vx, y, k);
}

static void dequantize_row_q5_1_cuda(const void * vx, float * y, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_DEQUANTIZE_BLOCK_SIZE - 1) / CUDA_DEQUANTIZE_BLOCK_SIZE;
    dequantize_block<QK5_1, QR5_1, dequantize_q5_1><<<num_blocks, CUDA_DEQUANTIZE_BLOCK_SIZE, 0, stream>>>(vx, y, k);
}

static void dequantize_row_q8_0_cuda(const void * vx, float * y, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_DEQUANTIZE_BLOCK_SIZE - 1) / CUDA_DEQUANTIZE_BLOCK_SIZE;
    dequantize_block<QK8_0, QR8_0, dequantize_q8_0><<<num_blocks, CUDA_DEQUANTIZE_BLOCK_SIZE, 0, stream>>>(vx, y, k);
}

static void dequantize_row_q2_K_cuda(const void * vx, float * y, const int k, hipStream_t stream) {
    const int nb = k / QK_K;
    dequantize_block_q2_K<<<nb, 64, 0, stream>>>(vx, y);
}

static void dequantize_row_q3_K_cuda(const void * vx, float * y, const int k, hipStream_t stream) {
    const int nb = k / QK_K;
    dequantize_block_q3_K<<<nb, 64, 0, stream>>>(vx, y);
}

static void dequantize_row_q4_K_cuda(const void * vx, float * y, const int k, hipStream_t stream) {
    const int nb = k / QK_K;
    dequantize_block_q4_K<<<nb, 32, 0, stream>>>(vx, y);
}

static void dequantize_row_q5_K_cuda(const void * vx, float * y, const int k, hipStream_t stream) {
    const int nb = k / QK_K;
    dequantize_block_q5_K<<<nb, 64, 0, stream>>>(vx, y);
}

static void dequantize_row_q6_K_cuda(const void * vx, float * y, const int k, hipStream_t stream) {
    const int nb = k / QK_K;
    dequantize_block_q6_K<<<nb, 64, 0, stream>>>(vx, y);
}

static void dequantize_mul_mat_vec_q4_0_cuda(const void * vx, const dfloat * y, float * dst, const int ncols, const int nrows, hipStream_t stream) {
    GGML_ASSERT(ncols % GGML_CUDA_DMMV_X == 0);
    const int block_num_y = (nrows + GGML_CUDA_DMMV_Y - 1) / GGML_CUDA_DMMV_Y;
    const dim3 block_nums(1, block_num_y, 1);
    const dim3 block_dims(WARP_SIZE, GGML_CUDA_DMMV_Y, 1);
    dequantize_mul_mat_vec<QK4_0, QR4_0, dequantize_q4_0>
        <<<block_nums, block_dims, 0, stream>>>(vx, y, dst, ncols, nrows);
}

static void dequantize_mul_mat_vec_q4_1_cuda(const void * vx, const dfloat * y, float * dst, const int ncols, const int nrows, hipStream_t stream) {
    GGML_ASSERT(ncols % GGML_CUDA_DMMV_X == 0);
    const int block_num_y = (nrows + GGML_CUDA_DMMV_Y - 1) / GGML_CUDA_DMMV_Y;
    const dim3 block_nums(1, block_num_y, 1);
    const dim3 block_dims(WARP_SIZE, GGML_CUDA_DMMV_Y, 1);
    dequantize_mul_mat_vec<QK4_1, QR4_1, dequantize_q4_1>
        <<<block_nums, block_dims, 0, stream>>>(vx, y, dst, ncols, nrows);
}

static void dequantize_mul_mat_vec_q5_0_cuda(const void * vx, const dfloat * y, float * dst, const int ncols, const int nrows, hipStream_t stream) {
    GGML_ASSERT(ncols % GGML_CUDA_DMMV_X == 0);
    const int block_num_y = (nrows + GGML_CUDA_DMMV_Y - 1) / GGML_CUDA_DMMV_Y;
    const dim3 block_nums(1, block_num_y, 1);
    const dim3 block_dims(WARP_SIZE, GGML_CUDA_DMMV_Y, 1);
    dequantize_mul_mat_vec<QK5_0, QR5_0, dequantize_q5_0>
        <<<block_nums, block_dims, 0, stream>>>(vx, y, dst, ncols, nrows);
}

static void dequantize_mul_mat_vec_q5_1_cuda(const void * vx, const dfloat * y, float * dst, const int ncols, const int nrows, hipStream_t stream) {
    GGML_ASSERT(ncols % GGML_CUDA_DMMV_X == 0);
    const int block_num_y = (nrows + GGML_CUDA_DMMV_Y - 1) / GGML_CUDA_DMMV_Y;
    const dim3 block_nums(1, block_num_y, 1);
    const dim3 block_dims(WARP_SIZE, GGML_CUDA_DMMV_Y, 1);
    dequantize_mul_mat_vec<QK5_1, QR5_1, dequantize_q5_1>
        <<<block_nums, block_dims, 0, stream>>>(vx, y, dst, ncols, nrows);
}

static void dequantize_mul_mat_vec_q8_0_cuda(const void * vx, const dfloat * y, float * dst, const int ncols, const int nrows, hipStream_t stream) {
    GGML_ASSERT(ncols % GGML_CUDA_DMMV_X == 0);
    const int block_num_y = (nrows + GGML_CUDA_DMMV_Y - 1) / GGML_CUDA_DMMV_Y;
    const dim3 block_nums(1, block_num_y, 1);
    const dim3 block_dims(WARP_SIZE, GGML_CUDA_DMMV_Y, 1);
    dequantize_mul_mat_vec<QK8_0, QR8_0, dequantize_q8_0>
        <<<block_nums, block_dims, 0, stream>>>(vx, y, dst, ncols, nrows);
}

static void dequantize_mul_mat_vec_q2_K_cuda(const void * vx, const float * y, float * dst, const int ncols, const int nrows, hipStream_t stream) {
    GGML_ASSERT(ncols % QK_K == 0);
    const int ny = 2; // very slightly faster than 1 even when K_QUANTS_PER_ITERATION = 2
    const int block_num_y = (nrows + ny - 1) / ny;
    const dim3 block_nums(1, block_num_y, 1);
    const dim3 block_dims(32, ny, 1);
    dequantize_mul_mat_vec_q2_k<<<block_nums, block_dims, 0, stream>>>(vx, y, dst, ncols, nrows);
}

static void dequantize_mul_mat_vec_q3_K_cuda(const void * vx, const float * y, float * dst, const int ncols, const int nrows, hipStream_t stream) {
    GGML_ASSERT(ncols % QK_K == 0);
    const int ny = 2 / K_QUANTS_PER_ITERATION;
    const int block_num_y = (nrows + ny - 1) / ny;
    const dim3 block_nums(1, block_num_y, 1);
    const dim3 block_dims(32, ny, 1);
    dequantize_mul_mat_vec_q3_k<<<block_nums, block_dims, 0, stream>>>(vx, y, dst, ncols, nrows);
}

static void dequantize_mul_mat_vec_q4_K_cuda(const void * vx, const float * y, float * dst, const int ncols, const int nrows, hipStream_t stream) {
    GGML_ASSERT(ncols % QK_K == 0);
    const int ny = 2 / K_QUANTS_PER_ITERATION;
    const int block_num_y = (nrows + ny - 1) / ny;
    const dim3 block_nums(1, block_num_y, 1);
    const dim3 block_dims(32, ny, 1);
    dequantize_mul_mat_vec_q4_k<<<block_nums, block_dims, 0, stream>>>(vx, y, dst, ncols, nrows);
}

static void dequantize_mul_mat_vec_q5_K_cuda(const void * vx, const float * y, float * dst, const int ncols, const int nrows, hipStream_t stream) {
    GGML_ASSERT(ncols % QK_K == 0);
    const dim3 block_dims(32, 1, 1);
    dequantize_mul_mat_vec_q5_k<<<nrows, block_dims, 0, stream>>>(vx, y, dst, ncols);
}

static void dequantize_mul_mat_vec_q6_K_cuda(const void * vx, const float * y, float * dst, const int ncols, const int nrows, hipStream_t stream) {
    GGML_ASSERT(ncols % QK_K == 0);
    const int ny = 2 / K_QUANTS_PER_ITERATION;
    const int block_num_y = (nrows + ny - 1) / ny;
    const dim3 block_nums(1, block_num_y, 1);
    const dim3 block_dims(32, ny, 1);
    dequantize_mul_mat_vec_q6_k<<<block_nums, block_dims, 0, stream>>>(vx, y, dst, ncols, nrows);
}

static void convert_fp16_to_fp32_cuda(const void * vx, float * y, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_DEQUANTIZE_BLOCK_SIZE - 1) / CUDA_DEQUANTIZE_BLOCK_SIZE;
    dequantize_block<1, 1, convert_f16><<<num_blocks, CUDA_DEQUANTIZE_BLOCK_SIZE, 0, stream>>>(vx, y, k);
}

static void convert_mul_mat_vec_f16_cuda(const void * vx, const dfloat * y, float * dst, const int ncols, const int nrows, hipStream_t stream) {
    GGML_ASSERT(ncols % GGML_CUDA_DMMV_X == 0);
    const int block_num_y = (nrows + GGML_CUDA_DMMV_Y - 1) / GGML_CUDA_DMMV_Y;
    const dim3 block_nums(1, block_num_y, 1);
    const dim3 block_dims(WARP_SIZE, GGML_CUDA_DMMV_Y, 1);
    dequantize_mul_mat_vec<1, 1, convert_f16>
        <<<block_nums, block_dims, 0, stream>>>(vx, y, dst, ncols, nrows);
}

static to_fp32_cuda_t ggml_get_to_fp32_cuda(ggml_type type) {
    switch (type) {
        case GGML_TYPE_Q4_0:
            return dequantize_row_q4_0_cuda;
        case GGML_TYPE_Q4_1:
            return dequantize_row_q4_1_cuda;
        case GGML_TYPE_Q5_0:
            return dequantize_row_q5_0_cuda;
        case GGML_TYPE_Q5_1:
            return dequantize_row_q5_1_cuda;
        case GGML_TYPE_Q8_0:
            return dequantize_row_q8_0_cuda;
        case GGML_TYPE_Q2_K:
            return dequantize_row_q2_K_cuda;
        case GGML_TYPE_Q3_K:
            return dequantize_row_q3_K_cuda;
        case GGML_TYPE_Q4_K:
            return dequantize_row_q4_K_cuda;
        case GGML_TYPE_Q5_K:
            return dequantize_row_q5_K_cuda;
        case GGML_TYPE_Q6_K:
            return dequantize_row_q6_K_cuda;
        case GGML_TYPE_F16:
            return convert_fp16_to_fp32_cuda;
        default:
            return nullptr;
    }
}

static void ggml_mul_mat_p021_f16_f32_cuda(const void * vx, const float * y, float * dst, const int ncols_x, const int nrows_x, const int nchannels_x, hipStream_t stream) {
    const dim3 block_nums(1, nrows_x, nchannels_x);
    const dim3 block_dims(WARP_SIZE, 1, 1);
    mul_mat_p021_f16_f32<<<block_nums, block_dims, 0, stream>>>(vx, y, dst, ncols_x, nrows_x, nchannels_x);
}

static void ggml_mul_mat_vec_nc_f16_f32_cuda(
    const void * vx, const float * y, float * dst, const int ncols_x, const int nrows_x, const int row_stride_x,
    const int nchannels_x, const int channel_stride_x, hipStream_t stream) {

    const dim3 block_nums(1, nrows_x, nchannels_x);
    const dim3 block_dims(WARP_SIZE, 1, 1);
    mul_mat_vec_nc_f16_f32<<<block_nums, block_dims, 0, stream>>>
        (vx, y, dst, ncols_x, nrows_x, row_stride_x, nchannels_x, channel_stride_x);
}

static void ggml_cpy_f32_f32_cuda(
    const char * cx, char * cdst, const int ne,
    const int ne00, const int ne01, const int nb00, const int nb01, const int nb02,
    const int ne10, const int ne11, const int nb10, const int nb11, const int nb12, hipStream_t stream) {

    const int num_blocks = (ne + CUDA_CPY_BLOCK_SIZE - 1) / CUDA_CPY_BLOCK_SIZE;
    cpy_f32_f16<cpy_1_f32_f32><<<num_blocks, CUDA_CPY_BLOCK_SIZE, 0, stream>>>
        (cx, cdst, ne, ne00, ne01, nb00, nb01, nb02, ne10, ne11, nb10, nb11, nb12);
}

static void ggml_cpy_f32_f16_cuda(
    const char * cx, char * cdst, const int ne,
    const int ne00, const int ne01, const int nb00, const int nb01, const int nb02,
    const int ne10, const int ne11, const int nb10, const int nb11, const int nb12, hipStream_t stream) {

    const int num_blocks = (ne + CUDA_CPY_BLOCK_SIZE - 1) / CUDA_CPY_BLOCK_SIZE;
    cpy_f32_f16<cpy_1_f32_f16><<<num_blocks, CUDA_CPY_BLOCK_SIZE, 0, stream>>>
        (cx, cdst, ne, ne00, ne01, nb00, nb01, nb02, ne10, ne11, nb10, nb11, nb12);
}

static void scale_f32_cuda(const float * x, float * dst, const float scale, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_SCALE_BLOCK_SIZE - 1) / CUDA_SCALE_BLOCK_SIZE;
    scale_f32<<<num_blocks, CUDA_SCALE_BLOCK_SIZE, 0, stream>>>(x, dst, scale, k);
}

static void rope_f32_cuda(const float * x, float * dst, const int ncols, const int nrows, const float p, const float theta_scale, hipStream_t stream) {
    GGML_ASSERT(nrows % 2 == 0);
    const dim3 block_dims(2*CUDA_ROPE_BLOCK_SIZE, 1, 1);
    const int num_blocks_x = (ncols + 2*CUDA_ROPE_BLOCK_SIZE - 1) / (2*CUDA_ROPE_BLOCK_SIZE);
    const dim3 block_nums(num_blocks_x, nrows, 1);
    rope_f32<<<block_nums, block_dims, 0, stream>>>(x, dst, ncols, p, theta_scale);
}

static void diag_mask_inf_f32_cuda(const float * x, float * dst, const int ncols_x, const int nrows_x, const int rows_per_channel, const int n_past, hipStream_t stream) {
    const dim3 block_dims(CUDA_DIAG_MASK_INF_BLOCK_SIZE, 1, 1);
    const int block_num_x = (ncols_x + CUDA_DIAG_MASK_INF_BLOCK_SIZE - 1) / CUDA_DIAG_MASK_INF_BLOCK_SIZE;
    const dim3 block_nums(block_num_x, nrows_x, 1);
    diag_mask_inf_f32<<<block_nums, block_dims, 0, stream>>>(x, dst, ncols_x, rows_per_channel, n_past);
}

static void soft_max_f32_cuda(const float * x, float * dst, const int ncols_x, const int nrows_x, hipStream_t stream) {
    const dim3 block_dims(WARP_SIZE, 1, 1);
    const dim3 block_nums(1, nrows_x, 1);
    soft_max_f32<<<block_nums, block_dims, 0, stream>>>(x, dst, ncols_x);
}

// buffer pool for cuda
#define MAX_CUDA_BUFFERS 256

struct scoped_spin_lock {
    std::atomic_flag& lock;
    scoped_spin_lock(std::atomic_flag& lock) : lock(lock) {
        while (lock.test_and_set(std::memory_order_acquire)) {
            ; // spin
        }
    }
    ~scoped_spin_lock() {
        lock.clear(std::memory_order_release);
    }
    scoped_spin_lock(const scoped_spin_lock&) = delete;
    scoped_spin_lock& operator=(const scoped_spin_lock&) = delete;
};

struct cuda_buffer {
    void * ptr = nullptr;
    size_t size = 0;
};

static cuda_buffer g_cuda_buffer_pool[GGML_CUDA_MAX_DEVICES][MAX_CUDA_BUFFERS];
static std::atomic_flag g_cuda_pool_lock = ATOMIC_FLAG_INIT;

static void * ggml_cuda_pool_malloc(size_t size, size_t * actual_size) {
    scoped_spin_lock lock(g_cuda_pool_lock);
    int id;
    CUDA_CHECK(hipGetDevice(&id));

    for (int i = 0; i < MAX_CUDA_BUFFERS; ++i) {
        cuda_buffer& b = g_cuda_buffer_pool[id][i];
        if (b.size >= size && b.ptr != nullptr) {
            void * ptr = b.ptr;
            *actual_size = b.size;
            b.ptr = nullptr;
            b.size = 0;
            return ptr;
        }
    }
    void * ptr;
    CUDA_CHECK(hipMalloc((void **) &ptr, size));
    *actual_size = size;
    return ptr;
}

static void ggml_cuda_pool_free(void * ptr, size_t size) {
    scoped_spin_lock lock(g_cuda_pool_lock);
    int id;
    CUDA_CHECK(hipGetDevice(&id));

    for (int i = 0; i < MAX_CUDA_BUFFERS; ++i) {
        cuda_buffer& b = g_cuda_buffer_pool[id][i];
        if (b.ptr == nullptr) {
            b.ptr = ptr;
            b.size = size;
            return;
        }
    }
    fprintf(stderr, "WARNING: cuda buffer pool full, increase MAX_CUDA_BUFFERS\n");
    CUDA_CHECK(hipFree(ptr));
}


static void * g_scratch_buffer = nullptr;
static size_t g_scratch_size = 1024*1024*1024; // 1 GB by default
static size_t g_scratch_offset = 0;

static int g_device_count = -1;
static int g_main_device = 0;
static float g_tensor_split[GGML_CUDA_MAX_DEVICES] = {0};

static hipblasHandle_t g_cublas_handles[GGML_CUDA_MAX_DEVICES] = {nullptr};

static hipStream_t g_cudaStreams_main[GGML_CUDA_MAX_DEVICES] = { nullptr };

void ggml_init_cublas() {
    static bool initialized = false;

    if (!initialized) {
        CUDA_CHECK(hipGetDeviceCount(&g_device_count));
        GGML_ASSERT(g_device_count <= GGML_CUDA_MAX_DEVICES);
        int64_t total_vram = 0;
        fprintf(stderr, "%s: found %d CUDA devices:\n", __func__, g_device_count);
        for (int id = 0; id < g_device_count; ++id) {
            hipDeviceProp_t prop;
            CUDA_CHECK(hipGetDeviceProperties(&prop, id));
            fprintf(stderr, "  Device %d: %s\n", id, prop.name);
            g_tensor_split[id] = total_vram;
            total_vram += prop.totalGlobalMem;
        }
        for (int id = 0; id < g_device_count; ++id) {
            g_tensor_split[id] /= total_vram;
        }

        for (int id = 0; id < g_device_count; ++id) {
            CUDA_CHECK(hipSetDevice(id));

            // create main stream
            CUDA_CHECK(hipStreamCreateWithFlags(&g_cudaStreams_main[id], hipStreamNonBlocking));

            // create cublas handle
            CUBLAS_CHECK(hipblasCreate(&g_cublas_handles[id]));
            CUBLAS_CHECK(hipblasSetMathMode(g_cublas_handles[id], HIPBLAS_TF32_TENSOR_OP_MATH));
        }

        // configure logging to stdout
        // CUBLAS_CHECK(cublasLoggerConfigure(1, 1, 0, nullptr));

        initialized = true;
    }
}

void ggml_cuda_set_tensor_split(const float * tensor_split) {
    bool all_zero = true;
    for (int i = 0; i < g_device_count; ++i) {
        if (tensor_split[i] != 0.0f) {
            all_zero = false;
            break;
        }
    }
    if (all_zero) {
        return;
    }
    float split_sum = 0.0f;
    for (int i = 0; i < g_device_count; ++i) {
        g_tensor_split[i] = split_sum;
        split_sum += tensor_split[i];
    }
    for (int i = 0; i < g_device_count; ++i) {
        g_tensor_split[i] /= split_sum;
    }
}

void * ggml_cuda_host_malloc(size_t size) {
    if (getenv("GGML_CUDA_NO_PINNED") != nullptr) {
        return nullptr;
    }

    void * ptr = nullptr;
    hipError_t err = hipHostMalloc((void **) &ptr, size);
    if (err != hipSuccess) {
        // The allocation error can be bypassed. A null ptr will assigned out of this function.
        // This can fixed the OOM error in WSL.
        hipGetLastError();
        fprintf(stderr, "WARNING: failed to allocate %.2f MB of pinned memory: %s\n",
            size/1024.0/1024.0, hipGetErrorString(err));
        return nullptr;
    }

    return ptr;
}

void ggml_cuda_host_free(void * ptr) {
    CUDA_CHECK(hipHostFree(ptr));
}

static hipError_t ggml_cuda_cpy_tensor_2d(
    void * dst, const struct ggml_tensor * src, int64_t i3, int64_t i2, int64_t i1_low, int64_t i1_high, hipStream_t stream) {

    hipMemcpyKind kind;
    char * src_ptr;
    if (src->backend == GGML_BACKEND_CPU) {
        kind = hipMemcpyHostToDevice;
        src_ptr = (char *) src->data;
    } else if (src->backend == GGML_BACKEND_GPU) {
        kind = hipMemcpyDeviceToDevice;
        struct ggml_tensor_extra_gpu * extra = (ggml_tensor_extra_gpu *) src->extra;
        int id;
        CUDA_CHECK(hipGetDevice(&id));
        src_ptr = (char *) extra->data_device[id];
    } else {
        GGML_ASSERT(false);
    }
    char * dst_ptr = (char *) dst;

    const int64_t ne0 = src->ne[0];
    const int64_t nb0 = src->nb[0];
    const int64_t nb1 = src->nb[1];
    const int64_t nb2 = src->nb[2];
    const int64_t nb3 = src->nb[3];
    const enum ggml_type type = src->type;
    const int64_t ts = ggml_type_size(type);
    const int64_t bs = ggml_blck_size(type);
    int64_t i1_diff = i1_high - i1_low;

    const char * x = src_ptr + i1_low*nb1 + i2*nb2 + i3*nb3;
    if (nb0 == ts && nb1 == ts*ne0/bs) {
        return hipMemcpyAsync(dst_ptr, x, i1_diff*nb1, kind, stream);
    } else if (nb0 == ts) {
        return hipMemcpy2DAsync(dst_ptr, ts*ne0/bs, x, nb1, ts*ne0/bs, i1_diff, kind, stream);
    } else {
        for (int64_t i1 = 0; i1 < i1_diff; i1++) {
            const void * rx = (const void *) ((const char *) x + i1*nb1);
            void * rd = (void *) (dst_ptr + i1*ts*ne0/bs);
            // pretend the row is a matrix with cols=1
            hipError_t r = hipMemcpy2DAsync(rd, ts/bs, rx, nb0, ts/bs, ne0, kind, stream);
            if (r != hipSuccess) return r;
        }
        return hipSuccess;
    }
}

inline void ggml_cuda_op_add(
    const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst, char * src0_ddq_i,
    float * src0_ddf_i, float * src1_ddf_i, float * dst_ddf_i, int64_t i02, int64_t i01_low, int64_t i01_high, int i1,
    hipStream_t & cudaStream_main){

    GGML_ASSERT(src0_ddf_i != nullptr);
    GGML_ASSERT(src1_ddf_i != nullptr);
    GGML_ASSERT(dst_ddf_i != nullptr);

    const int64_t ne0 = src0->ne[0];
    const int64_t i01_diff = i01_high - i01_low;

    // compute
    add_f32_cuda(src0_ddf_i, src1_ddf_i, dst_ddf_i, ne0*i01_diff, cudaStream_main);
    CUDA_CHECK(hipGetLastError());

    (void) src1;
    (void) dst;
    (void) src0_ddq_i;
    (void) i02;
    (void) i1;
}

inline void ggml_cuda_op_mul(
    const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst, char * src0_ddq_i,
    float * src0_ddf_i, float * src1_ddf_i, float * dst_ddf_i, int64_t i02, int64_t i01_low, int64_t i01_high, int i1,
    hipStream_t & cudaStream_main){

    GGML_ASSERT(src0_ddf_i != nullptr);
    GGML_ASSERT(src1_ddf_i != nullptr);
    GGML_ASSERT(dst_ddf_i != nullptr);

    const int64_t ne00 = src0->ne[0];

    const int64_t ne10 = src1->ne[0];
    const int64_t ne11 = src1->ne[1];

    for (int64_t i01 = i01_low; i01 < i01_high; i01++) {
        const int64_t i11 = i1*ne11 + i01%ne11; // broadcast src1 across src0

        float * src0_ddf_i01 = src0_ddf_i + i01*ne00;
        float * src1_ddf_i01 = src1_ddf_i + i11*ne10;
        float * dst_ddf_i01 = dst_ddf_i + i01*ne00;

        // compute
        mul_f32_cuda(src0_ddf_i01, src1_ddf_i01, dst_ddf_i01, ne00, ne10, cudaStream_main);
        CUDA_CHECK(hipGetLastError());
    }

    (void) dst;
    (void) src0_ddq_i;
    (void) i02;
}

inline void ggml_cuda_op_silu(
    const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst, char * src0_ddq_i,
    float * src0_ddf_i, float * src1_ddf_i, float * dst_ddf_i, int64_t i02, int64_t i01_low, int64_t i01_high, int i1,
    hipStream_t & cudaStream_main){

    GGML_ASSERT(src0_ddf_i != nullptr);
    GGML_ASSERT(dst_ddf_i != nullptr);

    const int64_t ne00 = src0->ne[0];
    const int64_t i01_diff = i01_high - i01_low;

    // compute
    silu_f32_cuda(src0_ddf_i, dst_ddf_i, ne00*i01_diff, cudaStream_main);
    CUDA_CHECK(hipGetLastError());

    (void) src1;
    (void) dst;
    (void) src0_ddq_i;
    (void) src1_ddf_i;
    (void) i02;
    (void) i1;
}

inline void ggml_cuda_op_rms_norm(
    const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst, char * src0_ddq_i,
    float * src0_ddf_i, float * src1_ddf_i, float * dst_ddf_i, int64_t i02, int64_t i01_low, int64_t i01_high, int i1,
    hipStream_t & cudaStream_main){

    GGML_ASSERT(src0_ddf_i != nullptr);
    GGML_ASSERT(dst_ddf_i != nullptr);

    const int64_t ne00 = src0->ne[0];
    const int64_t i01_diff = i01_high - i01_low;

    // compute
    rms_norm_f32_cuda(src0_ddf_i, dst_ddf_i, ne00, i01_diff, cudaStream_main);
    CUDA_CHECK(hipGetLastError());

    (void) src1;
    (void) dst;
    (void) src0_ddq_i;
    (void) src1_ddf_i;
    (void) i02;
    (void) i1;
}

inline void ggml_cuda_op_dequantize_mul_mat_vec(
    const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst, char * src0_ddq_i,
    float * src0_ddf_i, float * src1_ddf_i, float * dst_ddf_i, int64_t i02, int64_t i01_low, int64_t i01_high, int i1,
    hipStream_t & cudaStream_main){

    GGML_ASSERT(src0_ddq_i != nullptr);
    GGML_ASSERT(src1_ddf_i != nullptr);
    GGML_ASSERT(dst_ddf_i != nullptr);

    const int64_t ne00 = src0->ne[0];
    const int64_t nrows = i01_high - i01_low;

// on some GPUs it is faster to convert src1 to half and to use half precision intrinsics
#ifdef GGML_CUDA_DMMV_F16
    size_t ash;
    dfloat * src1_dfloat = nullptr; // dfloat == half

    bool src1_convert_f16 = src0->type == GGML_TYPE_Q4_0 || src0->type == GGML_TYPE_Q4_1 ||
        src0->type == GGML_TYPE_Q5_0 || src0->type == GGML_TYPE_Q5_1 ||
        src0->type == GGML_TYPE_Q8_0 || src0->type == GGML_TYPE_F16;

    if (src1_convert_f16) {
        src1_dfloat = (half *) ggml_cuda_pool_malloc(ne00*sizeof(half), &ash);
        ggml_cpy_f32_f16_cuda((char *) src1_ddf_i, (char *) src1_dfloat, ne00,
                                ne00, 1, sizeof(float), 0, 0,
                                ne00, 1, sizeof(half),  0, 0, cudaStream_main);
    }
#else
    dfloat * src1_dfloat = src1_ddf_i; // dfloat == float, no conversion
#endif // GGML_CUDA_DMMV_F16

    switch (src0->type) {
        case GGML_TYPE_Q4_0:
            dequantize_mul_mat_vec_q4_0_cuda(src0_ddq_i, src1_dfloat, dst_ddf_i, ne00, nrows, cudaStream_main);
            break;
        case GGML_TYPE_Q4_1:
            dequantize_mul_mat_vec_q4_1_cuda(src0_ddq_i, src1_dfloat, dst_ddf_i, ne00, nrows, cudaStream_main);
            break;
        case GGML_TYPE_Q5_0:
            dequantize_mul_mat_vec_q5_0_cuda(src0_ddq_i, src1_dfloat, dst_ddf_i, ne00, nrows, cudaStream_main);
            break;
        case GGML_TYPE_Q5_1:
            dequantize_mul_mat_vec_q5_1_cuda(src0_ddq_i, src1_dfloat, dst_ddf_i, ne00, nrows, cudaStream_main);
            break;
        case GGML_TYPE_Q8_0:
            dequantize_mul_mat_vec_q8_0_cuda(src0_ddq_i, src1_dfloat, dst_ddf_i, ne00, nrows, cudaStream_main);
            break;
        case GGML_TYPE_Q2_K:
            dequantize_mul_mat_vec_q2_K_cuda(src0_ddq_i, src1_ddf_i, dst_ddf_i, ne00, nrows, cudaStream_main);
            break;
        case GGML_TYPE_Q3_K:
            dequantize_mul_mat_vec_q3_K_cuda(src0_ddq_i, src1_ddf_i, dst_ddf_i, ne00, nrows, cudaStream_main);
            break;
        case GGML_TYPE_Q4_K:
            dequantize_mul_mat_vec_q4_K_cuda(src0_ddq_i, src1_ddf_i, dst_ddf_i, ne00, nrows, cudaStream_main);
            break;
        case GGML_TYPE_Q5_K:
            dequantize_mul_mat_vec_q5_K_cuda(src0_ddq_i, src1_ddf_i, dst_ddf_i, ne00, nrows, cudaStream_main);
            break;
        case GGML_TYPE_Q6_K:
            dequantize_mul_mat_vec_q6_K_cuda(src0_ddq_i, src1_ddf_i, dst_ddf_i, ne00, nrows, cudaStream_main);
            break;
        case GGML_TYPE_F16:
            convert_mul_mat_vec_f16_cuda(src0_ddq_i, src1_dfloat, dst_ddf_i, ne00, nrows, cudaStream_main);
            break;
        default:
            GGML_ASSERT(false);
            break;
    }
    CUDA_CHECK(hipGetLastError());

#ifdef GGML_CUDA_DMMV_F16
    if (src1_convert_f16) {
        ggml_cuda_pool_free(src1_dfloat, ash);
    }
#endif // GGML_CUDA_DMMV_F16

    (void) src1;
    (void) dst;
    (void) src0_ddf_i;
    (void) i02;
    (void) i1;
}

inline void ggml_cuda_op_mul_mat_cublas(
    const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst, char * src0_ddq_i,
    float * src0_ddf_i, float * src1_ddf_i, float * dst_ddf_i, int64_t i02, int64_t i01_low, int64_t i01_high, int i1,
    hipStream_t & cudaStream_main){

    GGML_ASSERT(src0_ddf_i != nullptr);
    GGML_ASSERT(src1_ddf_i != nullptr);
    GGML_ASSERT(dst_ddf_i != nullptr);

    const float alpha = 1.0f;
    const float beta = 0.0f;

    const int64_t ne00 = src0->ne[0];

    const int64_t ne10 = src1->ne[0];
    const int64_t ne11 = src1->ne[1];

    const int64_t ne0 = dst->ne[0];
    const int64_t i01_diff = i01_high - i01_low;

    int id;
    CUDA_CHECK(hipGetDevice(&id));

    // the main device has a larger memory buffer to hold the results from all GPUs
    // ldc == nrows of the matrix that cuBLAS writes into
    int ldc = dst->backend == GGML_BACKEND_GPU && id == g_main_device ? ne0 : i01_diff;

    CUBLAS_CHECK(hipblasSetStream(g_cublas_handles[id], cudaStream_main));
    CUBLAS_CHECK(
        hipblasSgemm(g_cublas_handles[id], HIPBLAS_OP_T, HIPBLAS_OP_N,
                i01_diff, ne11, ne10,
                &alpha, src0_ddf_i, ne00,
                        src1_ddf_i, ne10,
                &beta,  dst_ddf_i,  ldc));

    (void) dst;
    (void) src0_ddq_i;
    (void) i02;
    (void) i1;
}

inline void ggml_cuda_op_rope(
    const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst, char * src0_ddq_i,
    float * src0_ddf_i, float * src1_ddf_i, float * dst_ddf_i, int64_t i02, int64_t i01_low, int64_t i01_high, int i1,
    hipStream_t & cudaStream_main){

    GGML_ASSERT(src0_ddf_i != nullptr);
    GGML_ASSERT(dst_ddf_i != nullptr);

    const int64_t ne00 = src0->ne[0];
    const int64_t i01_diff = i01_high - i01_low;

    const int n_past = ((int32_t *) src1->data)[0];
    const int n_dims = ((int32_t *) src1->data)[1];
    const int mode   = ((int32_t *) src1->data)[2];
    GGML_ASSERT(mode == 0);

    const float theta_scale = powf(10000.0, -2.0f/n_dims);
    const float p = ((mode & 1) == 0 ? n_past + i02 : i02);

    // compute
    rope_f32_cuda(src0_ddf_i, dst_ddf_i, ne00, i01_diff, p, theta_scale, cudaStream_main);
    CUDA_CHECK(hipGetLastError());

    (void) dst;
    (void) src0_ddq_i;
    (void) src1_ddf_i;
    (void) i1;
}

inline void ggml_cuda_op_diag_mask_inf(
    const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst, char * src0_ddq_i,
    float * src0_ddf_i, float * src1_ddf_i, float * dst_ddf_i, int64_t i02, int64_t i01_low, int64_t i01_high, int i1,
    hipStream_t & cudaStream_main){

    GGML_ASSERT(src0_ddf_i != nullptr);
    GGML_ASSERT(dst_ddf_i != nullptr);

    const int64_t ne00 = src0->ne[0];
    const int64_t ne01 = src0->ne[1];
    const int64_t i01_diff = i01_high - i01_low;

    const int n_past = ((int32_t *) src1->data)[0];

    // compute
    diag_mask_inf_f32_cuda(src0_ddf_i, dst_ddf_i, ne00, i01_diff, ne01, n_past, cudaStream_main);
    CUDA_CHECK(hipGetLastError());

    (void) dst;
    (void) src0_ddq_i;
    (void) src1_ddf_i;
    (void) i02;
    (void) i1;
}

inline void ggml_cuda_op_soft_max(
    const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst, char * src0_ddq_i,
    float * src0_ddf_i, float * src1_ddf_i, float * dst_ddf_i, int64_t i02, int64_t i01_low, int64_t i01_high, int i1,
    hipStream_t & cudaStream_main){

    GGML_ASSERT(src0_ddf_i != nullptr);
    GGML_ASSERT(dst_ddf_i != nullptr);

    const int64_t ne00 = src0->ne[0];
    const int64_t i01_diff = i01_high - i01_low;

    // compute
    soft_max_f32_cuda(src0_ddf_i, dst_ddf_i, ne00, i01_diff, cudaStream_main);
    CUDA_CHECK(hipGetLastError());

    (void) src1;
    (void) dst;
    (void) src0_ddq_i;
    (void) src1_ddf_i;
    (void) i02;
    (void) i1;
}

inline void ggml_cuda_op_scale(
    const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst, char * src0_ddq_i,
    float * src0_ddf_i, float * src1_ddf_i, float * dst_ddf_i, int64_t i02, int64_t i01_low, int64_t i01_high, int i1,
    hipStream_t & cudaStream_main){

    GGML_ASSERT(src0_ddf_i != nullptr);
    GGML_ASSERT(dst_ddf_i != nullptr);

    const float scale = ((float *) src1->data)[0];

    const int64_t ne00 = src0->ne[0];
    const int64_t i01_diff = i01_high - i01_low;

    // compute
    scale_f32_cuda(src0_ddf_i, dst_ddf_i, scale, ne00*i01_diff, cudaStream_main);
    CUDA_CHECK(hipGetLastError());

    (void) src1;
    (void) dst;
    (void) src0_ddq_i;
    (void) src1_ddf_i;
    (void) i02;
    (void) i1;
}

static void ggml_cuda_op(const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst,
                         ggml_cuda_op_t op, bool src0_needs_f32, bool flatten_rows) {
    const int64_t ne00 = src0->ne[0];
    const int64_t ne01 = src0->ne[1];
    const int64_t ne02 = src0->ne[2];
    const int64_t ne03 = src0->ne[3];
    const int64_t nrows0 = ggml_nrows(src0);

    const bool use_src1 = src1 != nullptr;
    const int64_t ne10 = use_src1 ? src1->ne[0] : 1;
    const int64_t ne11 = use_src1 ? src1->ne[1] : 1;
    const int64_t ne12 = use_src1 ? src1->ne[2] : 1;
    const int64_t ne13 = use_src1 ? src1->ne[3] : 1;

    const int64_t ne0 = dst->ne[0];
    const int64_t ne1 = dst->ne[1];

    const int nb2  = dst->nb[2];
    const int nb3  = dst->nb[3];

    GGML_ASSERT(dst->backend != GGML_BACKEND_GPU_SPLIT);
    GGML_ASSERT(!use_src1 || src1->backend != GGML_BACKEND_GPU_SPLIT);

    // strides for iteration over dims 3 and 2
    const int64_t num_iters = flatten_rows ? 1 : ne02 * ne03;
    const int64_t stride_mod = flatten_rows ? ne02 * ne03 : 1;
    const int64_t src0_stride = ne00 * ne01 * stride_mod;
    const int64_t src1_stride = ne10 * ne11 * stride_mod;
    const int64_t dst_stride = ne0 * ne1 * stride_mod;

    const size_t src0_ts = ggml_type_size(src0->type);
    const size_t src0_bs = ggml_blck_size(src0->type);

    struct ggml_tensor_extra_gpu * src0_extra =            (ggml_tensor_extra_gpu *) src0->extra;
    struct ggml_tensor_extra_gpu * src1_extra = use_src1 ? (ggml_tensor_extra_gpu *) src1->extra : nullptr;
    struct ggml_tensor_extra_gpu * dst_extra  =            (ggml_tensor_extra_gpu *) dst->extra;

    const bool src0_on_device = src0->backend == GGML_BACKEND_GPU || src0->backend == GGML_BACKEND_GPU_SPLIT;
    const bool src0_is_contiguous = ggml_is_contiguous(src0);
    const bool src0_is_f32 = src0->type == GGML_TYPE_F32;

    const bool src1_is_contiguous = use_src1 && ggml_is_contiguous(src1);
    const bool src1_stays_on_host = use_src1 && (
        dst->op == GGML_OP_SCALE || dst->op == GGML_OP_DIAG_MASK_INF || dst->op == GGML_OP_ROPE);

    const bool split = src0->backend == GGML_BACKEND_GPU_SPLIT;

    const to_fp32_cuda_t to_fp32_cuda = ggml_get_to_fp32_cuda(src0->type);

    // dd = data device
    char  * src0_ddq[GGML_CUDA_MAX_DEVICES] = {nullptr}; // quantized
    float * src0_ddf[GGML_CUDA_MAX_DEVICES] = {nullptr}; // float
    float * src1_ddf[GGML_CUDA_MAX_DEVICES] = {nullptr};
    float *  dst_ddf[GGML_CUDA_MAX_DEVICES] = {nullptr};

    // asq = actual size quantized, asf = actual size float
    size_t src0_asq[GGML_CUDA_MAX_DEVICES] = {0};
    size_t src0_asf[GGML_CUDA_MAX_DEVICES] = {0};
    size_t src1_asf[GGML_CUDA_MAX_DEVICES] = {0};
    size_t  dst_asf[GGML_CUDA_MAX_DEVICES] = {0};

    // if multiple GPUs are used they need to wait for the main GPU to finish
    if (split && g_device_count > 1) {
        CUDA_CHECK(hipSetDevice(g_main_device));
        CUDA_CHECK(hipDeviceSynchronize());
    }

    for (int id = 0; id < g_device_count; ++id) {
        if (!split && id != g_main_device) {
            continue;
        }

        const bool src1_on_device = use_src1 && src1->backend == GGML_BACKEND_GPU && id == g_main_device;
        const bool dst_on_device = dst->backend == GGML_BACKEND_GPU && id == g_main_device;

        int64_t row_low, row_high;
        if (split) {
            row_low = id == 0 ? 0 : nrows0*g_tensor_split[id];
            row_high = id == g_device_count - 1 ? nrows0 : nrows0*g_tensor_split[id + 1];
        } else {
            row_low = 0;
            row_high = nrows0;
        }
        if (row_low == row_high) {
            continue;
        }

        int64_t row_diff = row_high - row_low;

        hipSetDevice(id);

        if (src0_on_device && src0_is_contiguous) {
            if (src0_is_f32) {
                src0_ddf[id] = (float *) src0_extra->data_device[id];
            } else {
                src0_ddq[id] = (char *) src0_extra->data_device[id];
            }
        } else {
            if (src0_is_f32) {
                src0_ddf[id] = (float *) ggml_cuda_pool_malloc(row_diff*ne00 * sizeof(float), &src0_asf[id]);
            } else {
                src0_ddq[id] = (char *) ggml_cuda_pool_malloc(row_diff*ne00 * src0_ts/src0_bs, &src0_asq[id]);
            }
        }

        if (src0_needs_f32 && !src0_is_f32) {
            src0_ddf[id] = (float *) ggml_cuda_pool_malloc(row_diff*ne00 * sizeof(float), &src0_asf[id]);
        }

        if (use_src1 && !src1_stays_on_host) {
            if (src1_on_device && src1_is_contiguous) {
                src1_ddf[id] = (float *) src1_extra->data_device[id];
            } else {
                src1_ddf[id] = (float *) ggml_cuda_pool_malloc(num_iters*src1_stride * sizeof(float), &src1_asf[id]);
            }
        }
        if (dst_on_device) {
            dst_ddf[id] = (float *) dst_extra->data_device[id];
        } else {
            size_t size_dst_ddf = split ? row_diff*ne1 * sizeof(float) : num_iters*dst_stride * sizeof(float);
            dst_ddf[id] = (float *) ggml_cuda_pool_malloc(size_dst_ddf, &dst_asf[id]);
        }

        const int64_t i03_max = flatten_rows ? 1 : ne03;
        const int64_t i02_max = flatten_rows ? 1 : ne02;
        const int64_t rows_per_iter = flatten_rows ? nrows0 : ne01;

        for (int64_t i03 = 0; i03 < i03_max; i03++) {
            const int64_t i13 = i03 % ne13;
            for (int64_t i02 = 0; i02 < i02_max; i02++) {
                const int64_t i12 = i02 % ne12;

                const int64_t i0 = i03*ne02 + i02;

                // i0 values that contain the lower/upper rows for a split tensor when using multiple GPUs
                const int64_t i0_offset_low = row_low/rows_per_iter;
                const int64_t i0_offset_high = row_high/rows_per_iter;

                int64_t i01_low = 0;
                int64_t i01_high = rows_per_iter;
                if (split) {
                    if (i0 < i0_offset_low || i0 > i0_offset_high) {
                        continue;
                    }
                    if (i0 == i0_offset_low) {
                        i01_low = row_low % rows_per_iter;
                    }
                    if (i0 == i0_offset_high) {
                        i01_high = row_high % rows_per_iter;
                    }
                }

                // There is possibly a bug in the Windows nvcc compiler regarding instruction reordering or optimizing out local variables.
                // Removing the first assert or changing the order of the arguments causes the second assert to fail.
                // Removing both asserts results in i01_high becoming 0 which in turn results in garbage output.
                // The root cause seems to be a problem with i0_offset_high becoming 0 when it should always be >0 (for single GPU).
                GGML_ASSERT(i01_low == 0 || g_device_count > 1);
                GGML_ASSERT(i01_high == rows_per_iter || g_device_count > 1);

                const int64_t i01_diff = i01_high - i01_low;
                if (i01_diff == 0) {
                    continue;
                }
                const int64_t i11 = i13*ne12 + i12;

                hipStream_t cudaStream_main = g_cudaStreams_main[id];

                // for split tensors the data begins at i0 == i0_offset_low
                char  * src0_ddq_i = src0_ddq[id] + (i0 - i0_offset_low)*src0_stride*src0_ts/src0_bs;
                float * src0_ddf_i = src0_ddf[id] + (i0 - i0_offset_low)*src0_stride;
                float * src1_ddf_i = src1_ddf[id] + i11*src1_stride;
                float * dst_ddf_i  =  dst_ddf[id] + (i0 - i0_offset_low)*dst_stride;

                // for split tensors the data pointer needs to be rounded down
                // to the bin edge for i03, i02 bins beyond the first
                if (i0 - i0_offset_low > 0) {
                    GGML_ASSERT(!flatten_rows);
                    src0_ddq_i -= (row_low % ne01)*ne00 * src0_ts/src0_bs;
                    src0_ddf_i -= (row_low % ne01)*ne00;
                    dst_ddf_i  -= (row_low % ne0)*ne1;
                }

                // the main device memory buffer can be on VRAM scratch, with space for all partial results
                // in that case an offset on dst_ddf_i is needed
                if (dst->backend == GGML_BACKEND_GPU && id == g_main_device) {
                    dst_ddf_i += i01_low; // offset is 0 if no tensor split
                }

                // copy src0, src1 to device if necessary
                if (use_src1 && !src1_stays_on_host) {
                    if (src1->backend == GGML_BACKEND_CPU) {
                        GGML_ASSERT(!flatten_rows || nrows0 == ggml_nrows(src1));
                        int64_t nrows1 = flatten_rows ? nrows0 : ne11;
                        CUDA_CHECK(ggml_cuda_cpy_tensor_2d(src1_ddf_i, src1, i03, i02, 0, nrows1, cudaStream_main));
                    } else if (src1->backend == GGML_BACKEND_GPU && src1_is_contiguous) {
                        if (id != g_main_device) {
                            GGML_ASSERT(!flatten_rows);
                            float * src1_ddf_i_source = (float *) src1_extra->data_device[g_main_device];
                            src1_ddf_i_source += i11*src1_stride;
                            CUDA_CHECK(hipMemcpyAsync(src1_ddf_i, src1_ddf_i_source, src1_stride*sizeof(float),
                                                    hipMemcpyDeviceToDevice, cudaStream_main));
                        }
                    } else if (src1_on_device && !src1_is_contiguous) {
                        GGML_ASSERT(!split);
                        CUDA_CHECK(ggml_cuda_cpy_tensor_2d(src1_ddf_i, src1, i03, i02, 0, ne11, cudaStream_main));
                    } else {
                        GGML_ASSERT(false);
                    }
                }

                if (!src0_on_device || !src0_is_contiguous) {
                    if (src0_is_f32) {
                        CUDA_CHECK(ggml_cuda_cpy_tensor_2d(src0_ddf_i, src0, i03, i02, i01_low, i01_high, cudaStream_main));
                    } else {
                        CUDA_CHECK(ggml_cuda_cpy_tensor_2d(src0_ddq_i, src0, i03, i02, i01_low, i01_high, cudaStream_main));
                    }
                }

                // convert src0 to f32 if it is necessary for the ggml_cuda_op
                if (src0_needs_f32 && !src0_is_f32) {
                    to_fp32_cuda(src0_ddq_i, src0_ddf_i, i01_diff*ne00, cudaStream_main);
                    CUDA_CHECK(hipGetLastError());
                }

                // do the computation
                op(src0, src1, dst, src0_ddq_i, src0_ddf_i, src1_ddf_i, dst_ddf_i, i02, i01_low, i01_high, i11, cudaStream_main);

                // copy dst to host or other device if necessary
                if (!dst_on_device) {
                    void * dst_off_device;
                    hipMemcpyKind kind;
                    if (dst->backend == GGML_BACKEND_CPU) {
                        dst_off_device = dst->data;
                        kind = hipMemcpyDeviceToHost;
                    } else if (dst->backend == GGML_BACKEND_GPU) {
                        dst_off_device = dst_extra->data_device[g_main_device];
                        kind = hipMemcpyDeviceToDevice;
                    } else {
                        GGML_ASSERT(false);
                    }
                    if (split) {
                        // src0 = weight matrix is saved as a transposed matrix for better memory layout.
                        // dst is NOT transposed.
                        // The outputs of cuBLAS matrix matrix multiplications can therefore NOT simply be concatenated for >1 GPU.
                        // Instead they need to be copied to the correct slice in ne0 = dst row index.
                        // If dst is a vector with ne0 == 1 then you don't have to do this but it still produces correct results.
                        for (int64_t j = 0; j < ne1; ++j) {
                            float * dhf_dst_i = (float *) ((char *) dst_off_device + (j*ne0 + i01_low)*sizeof(float) + i02*nb2 + i03*nb3);
                            CUDA_CHECK(hipMemcpyAsync(dhf_dst_i, dst_ddf_i + j*i01_diff, i01_diff*sizeof(float), kind, cudaStream_main));
                        }
                    } else {
                        float * dhf_dst_i = (float *) ((char *) dst_off_device + i02*nb2 + i03*nb3);
                        CUDA_CHECK(hipMemcpyAsync(dhf_dst_i, dst_ddf_i, dst_stride*sizeof(float), kind, cudaStream_main));
                    }
                }
            }
        }
    }

    // wait until each device is finished, then free their buffers
    for (int id = 0; id < g_device_count; ++id) {
        if (src0_asq[id] == 0 && src0_asf[id] == 0 && src1_asf[id] == 0 && dst_asf[id] == 0) {
            continue;
        }

        CUDA_CHECK(hipSetDevice(id));
        CUDA_CHECK(hipDeviceSynchronize());

        if (src0_asq[id] > 0) {
            ggml_cuda_pool_free(src0_ddq[id], src0_asq[id]);
        }
        if (src0_asf[id] > 0) {
            ggml_cuda_pool_free(src0_ddf[id], src0_asf[id]);
        }
        if (src1_asf[id] > 0) {
            ggml_cuda_pool_free(src1_ddf[id], src1_asf[id]);
        }
        if (dst_asf[id] > 0) {
            ggml_cuda_pool_free(dst_ddf[id], dst_asf[id]);
        }
    }
}

void ggml_cuda_add(const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst) {
    GGML_ASSERT(src0->type == GGML_TYPE_F32 && src1->type == GGML_TYPE_F32 && dst->type == GGML_TYPE_F32);
    ggml_cuda_op(src0, src1, dst, ggml_cuda_op_add, true, true);
}

void ggml_cuda_mul(const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst) {
    GGML_ASSERT(src0->type == GGML_TYPE_F32 && src1->type == GGML_TYPE_F32 && dst->type == GGML_TYPE_F32);
    ggml_cuda_op(src0, src1, dst, ggml_cuda_op_mul, true, false); // TODO ggml_cuda_op needs modification for flatten
}

void ggml_cuda_silu(const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst) {
    GGML_ASSERT(src0->type == GGML_TYPE_F32 && dst->type == GGML_TYPE_F32);
    ggml_cuda_op(src0, src1, dst, ggml_cuda_op_silu, true, true);
}

void ggml_cuda_rms_norm(const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst) {
    GGML_ASSERT(src0->type == GGML_TYPE_F32 && dst->type == GGML_TYPE_F32);
    ggml_cuda_op(src0, src1, dst, ggml_cuda_op_rms_norm, true, true);
}

bool ggml_cuda_can_mul_mat(const struct ggml_tensor * src0, const struct ggml_tensor * src1, struct ggml_tensor * dst) {
    const int64_t ne10 = src1->ne[0];

    const int64_t ne0 = dst->ne[0];
    const int64_t ne1 = dst->ne[1];

    // TODO: find the optimal values for these
    if ((src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16 || ggml_is_quantized(src0->type)) &&
        src1->type == GGML_TYPE_F32 &&
        dst->type == GGML_TYPE_F32 &&
        (ne0 >= 32 && ne1 >= 32 && ne10 >= 32)) {
        return true;
    }

    return false;
}

void ggml_cuda_mul_mat_vec_p021(const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst){
    GGML_ASSERT(ggml_is_permuted(src0) && ggml_is_permuted(src1));
    GGML_ASSERT(src0->backend != GGML_BACKEND_GPU_SPLIT);
    GGML_ASSERT(src0->nb[0] <= src0->nb[1] && src0->nb[2] <= src0->nb[3]); // 0213 permutation
    GGML_ASSERT(src1->nb[0] <= src1->nb[1] && src1->nb[2] <= src1->nb[3]); // 0213 permutation
    GGML_ASSERT(src0->type == GGML_TYPE_F16);
    GGML_ASSERT(src1->type == GGML_TYPE_F32);

    const int64_t ne00 = src0->ne[0];
    const int64_t ne01 = src0->ne[1];
    const int64_t ne02 = src0->ne[2];

    CUDA_CHECK(hipSetDevice(g_main_device));
    hipStream_t cudaStream_main = g_cudaStreams_main[g_main_device];

    struct ggml_tensor_extra_gpu * src0_extra = (ggml_tensor_extra_gpu *) src0->extra;
    void * src0_ddq = src0_extra->data_device[g_main_device];

    struct ggml_tensor_extra_gpu * src1_extra = (ggml_tensor_extra_gpu *) src1->extra;
    float * src1_ddf = (float *) src1_extra->data_device[g_main_device];

    struct ggml_tensor_extra_gpu * dst_extra = (ggml_tensor_extra_gpu *) dst->extra;
    float * dst_ddf = (float *) dst_extra->data_device[g_main_device];

    ggml_mul_mat_p021_f16_f32_cuda(src0_ddq, src1_ddf, dst_ddf, ne00, ne01, ne02, cudaStream_main);
}

void ggml_cuda_mul_mat_vec_nc(const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst){
    GGML_ASSERT(!ggml_is_contiguous(src0) && ggml_is_contiguous(src1));
    GGML_ASSERT(!ggml_is_permuted(src0));
    GGML_ASSERT(src0->backend != GGML_BACKEND_GPU_SPLIT);
    GGML_ASSERT(src0->type == GGML_TYPE_F16);
    GGML_ASSERT(src1->type == GGML_TYPE_F32);

    const int64_t ne00 = src0->ne[0];
    const int64_t ne01 = src0->ne[1];
    const int64_t ne02 = src0->ne[2];

    const int64_t nb01 = src0->nb[1];
    const int64_t nb02 = src0->nb[2];

    CUDA_CHECK(hipSetDevice(g_main_device));
    hipStream_t cudaStream_main = g_cudaStreams_main[g_main_device];

    struct ggml_tensor_extra_gpu * src0_extra = (ggml_tensor_extra_gpu *) src0->extra;
    void * src0_ddq = src0_extra->data_device[g_main_device];

    struct ggml_tensor_extra_gpu * src1_extra = (ggml_tensor_extra_gpu *) src1->extra;
    float * src1_ddf = (float *) src1_extra->data_device[g_main_device];

    struct ggml_tensor_extra_gpu * dst_extra = (ggml_tensor_extra_gpu *) dst->extra;
    float * dst_ddf = (float *) dst_extra->data_device[g_main_device];

    const int row_stride_x = nb01 / sizeof(half);
    const int channel_stride_x = nb02 / sizeof(half);

    ggml_mul_mat_vec_nc_f16_f32_cuda(src0_ddq, src1_ddf, dst_ddf, ne00, ne01, row_stride_x, ne02, channel_stride_x, cudaStream_main);
}

void ggml_cuda_mul_mat(const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst) {
    bool all_on_device = (src0->backend == GGML_BACKEND_GPU || src0->backend == GGML_BACKEND_GPU_SPLIT) &&
        src1->backend == GGML_BACKEND_GPU && dst->backend == GGML_BACKEND_GPU;

    if (all_on_device && ggml_is_permuted(src0) && ggml_is_permuted(src1) && src1->ne[1] == 1) {
        ggml_cuda_mul_mat_vec_p021(src0, src1, dst);
    } else if (all_on_device && !ggml_is_contiguous(src0) && ggml_is_contiguous(src1) && src1->ne[1] == 1) {
        ggml_cuda_mul_mat_vec_nc(src0, src1, dst);
    }else if (src0->type == GGML_TYPE_F32) {
        ggml_cuda_op(src0, src1, dst, ggml_cuda_op_mul_mat_cublas, true, false);
    } else if (ggml_is_quantized(src0->type) || src0->type == GGML_TYPE_F16) {
        if (src1->ne[1] == 1 && src0->ne[0] % GGML_CUDA_DMMV_X == 0 && src0->ne[1] % GGML_CUDA_DMMV_Y == 0) {
            ggml_cuda_op(src0, src1, dst, ggml_cuda_op_dequantize_mul_mat_vec, false, false);
        } else {
            ggml_cuda_op(src0, src1, dst, ggml_cuda_op_mul_mat_cublas, true, false);
        }
    } else {
        GGML_ASSERT(false);
    }
}

void ggml_cuda_scale(const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst) {
    GGML_ASSERT(src0->type == GGML_TYPE_F32 && dst->type == GGML_TYPE_F32);
    ggml_cuda_op(src0, src1, dst, ggml_cuda_op_scale, true, true);
}

void ggml_cuda_cpy(const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst) {
    const int64_t ne = ggml_nelements(src0);
    GGML_ASSERT(ne == ggml_nelements(src1));

    GGML_ASSERT(src0->backend == GGML_BACKEND_GPU);
    GGML_ASSERT(src1->backend == GGML_BACKEND_GPU);

    GGML_ASSERT(ggml_nbytes(src0) <= INT_MAX);
    GGML_ASSERT(ggml_nbytes(src1) <= INT_MAX);

    const int64_t ne00 = src0->ne[0];
    const int64_t ne01 = src0->ne[1];
    GGML_ASSERT(src0->ne[3] == 1);

    const int64_t nb00 = src0->nb[0];
    const int64_t nb01 = src0->nb[1];
    const int64_t nb02 = src0->nb[2];

    const int64_t ne10 = src1->ne[0];
    const int64_t ne11 = src1->ne[1];
    GGML_ASSERT(src1->ne[3] == 1);

    const int64_t nb10 = src1->nb[0];
    const int64_t nb11 = src1->nb[1];
    const int64_t nb12 = src1->nb[2];

    CUDA_CHECK(hipSetDevice(g_main_device));
    hipStream_t cudaStream_main = g_cudaStreams_main[g_main_device];

    const struct ggml_tensor_extra_gpu * src0_extra = (ggml_tensor_extra_gpu *) src0->extra;
    const struct ggml_tensor_extra_gpu * src1_extra = (ggml_tensor_extra_gpu *) src1->extra;

    char * src0_ddc = (char *) src0_extra->data_device[g_main_device];
    char * src1_ddc = (char *) src1_extra->data_device[g_main_device];

    if (src0->type == GGML_TYPE_F32 && src1->type == GGML_TYPE_F32) {
        ggml_cpy_f32_f32_cuda(src0_ddc, src1_ddc, ne, ne00, ne01, nb00, nb01, nb02,
                              ne10, ne11, nb10, nb11, nb12, cudaStream_main);
    } else if (src0->type == GGML_TYPE_F32 && src1->type == GGML_TYPE_F16) {
        ggml_cpy_f32_f16_cuda(src0_ddc, src1_ddc, ne, ne00, ne01, nb00, nb01, nb02,
                              ne10, ne11, nb10, nb11, nb12, cudaStream_main);
    } else {
        GGML_ASSERT(false);
    }

    (void) dst;
}

void ggml_cuda_diag_mask_inf(const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst) {
    GGML_ASSERT(src0->type == GGML_TYPE_F32 && dst->type == GGML_TYPE_F32);
    ggml_cuda_op(src0, src1, dst, ggml_cuda_op_diag_mask_inf, true, true);
}

void ggml_cuda_soft_max(const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst) {
    GGML_ASSERT(src0->type == GGML_TYPE_F32 && dst->type == GGML_TYPE_F32);
    ggml_cuda_op(src0, src1, dst, ggml_cuda_op_soft_max, true, true);
}

void ggml_cuda_rope(const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst) {
    GGML_ASSERT(src0->type == GGML_TYPE_F32 && dst->type == GGML_TYPE_F32);
    ggml_cuda_op(src0, src1, dst, ggml_cuda_op_rope, true, false); // FIXME flatten changes results
}

void ggml_cuda_nop(const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst) {
    (void) src0;
    (void) src1;
    (void) dst;
}

void ggml_cuda_transform_tensor(void * data, struct ggml_tensor * tensor) {
    int nrows = ggml_nrows(tensor);
    const size_t nb1 = tensor->nb[1];
    ggml_backend backend = tensor->backend;
    struct ggml_tensor_extra_gpu * extra = new struct ggml_tensor_extra_gpu;
    memset(extra, 0, sizeof(*extra));

    for (int id = 0; id < g_device_count; ++id) {
        if (backend == GGML_BACKEND_GPU && id != g_main_device) {
            continue;
        }

        hipSetDevice(id);

        int row_low, row_high;
        if (backend == GGML_BACKEND_GPU) {
            row_low = 0;
            row_high = nrows;
        } else if (backend == GGML_BACKEND_GPU_SPLIT) {
            row_low = id == 0 ? 0 : nrows*g_tensor_split[id];
            row_high = id == g_device_count - 1 ? nrows : nrows*g_tensor_split[id + 1];
        } else {
            GGML_ASSERT(false);
        }
        if (row_low == row_high) {
            continue;
        }

        int64_t nrows_split = row_high - row_low;

        const size_t offset_split = row_low*nb1;
        const size_t size = ggml_nbytes_split(tensor, nrows_split);

        void * buf;
        CUDA_CHECK(hipMalloc(&buf, size));
        void * buf_host = (char*)data + offset_split;

        hipMemcpy(buf, buf_host, size, hipMemcpyHostToDevice);

        extra->data_device[id] = buf;
    }

    tensor->extra = extra;
}

void ggml_cuda_free_data(struct ggml_tensor * tensor) {
    if (tensor->backend != GGML_BACKEND_GPU && tensor->backend != GGML_BACKEND_GPU_SPLIT) {
        return;
    }

    ggml_tensor_extra_gpu * extra = (ggml_tensor_extra_gpu *) tensor->extra;

    for (int id = 0; id < g_device_count; ++id) {
        if (extra->data_device[id] == nullptr) {
            continue;
        }

        CUDA_CHECK(hipSetDevice(id));
        CUDA_CHECK(hipFree(extra->data_device[id]));
    }

    delete extra;
}

void ggml_cuda_assign_buffers_impl(struct ggml_tensor * tensor, bool scratch) {
    if (scratch && g_scratch_size == 0) {
        return;
    }

    // recursively assign CUDA buffers until a compute tensor is found
    if (tensor->src0 != nullptr && tensor->src0->backend == GGML_BACKEND_CPU) {
        const ggml_op src0_op = tensor->src0->op;
        if (src0_op == GGML_OP_RESHAPE || src0_op == GGML_OP_TRANSPOSE || src0_op == GGML_OP_VIEW) {
            ggml_cuda_assign_buffers_impl(tensor->src0, scratch);
        }
    }
    if (tensor->op == GGML_OP_CPY && tensor->src1->backend == GGML_BACKEND_CPU) {
        ggml_cuda_assign_buffers_impl(tensor->src1, scratch);
    }

    tensor->backend = GGML_BACKEND_GPU;
    struct ggml_tensor_extra_gpu * extra = new ggml_tensor_extra_gpu;

    const bool inplace = (tensor->src0 != nullptr && tensor->src0->data == tensor->data) ||
        tensor->op == GGML_OP_VIEW;
    const size_t size = ggml_nbytes(tensor);

    CUDA_CHECK(hipSetDevice(g_main_device));
    if (inplace && tensor->src0->backend == GGML_BACKEND_GPU) {
        struct ggml_tensor_extra_gpu * src0_extra = (ggml_tensor_extra_gpu * ) tensor->src0->extra;
        char * src0_ddc = (char *) src0_extra->data_device[g_main_device];
        size_t offset = 0;
        if (tensor->op == GGML_OP_VIEW) {
            memcpy(&offset, tensor->opt[0]->data, sizeof(size_t));
        }
        extra->data_device[g_main_device] = src0_ddc + offset;
    } else if (tensor->op == GGML_OP_CPY) {
        struct ggml_tensor_extra_gpu * src1_extra = (ggml_tensor_extra_gpu * ) tensor->src1->extra;
        void * src1_ddv = src1_extra->data_device[g_main_device];
        extra->data_device[g_main_device] = src1_ddv;
    } else if (scratch) {
        GGML_ASSERT(size <= g_scratch_size);
        if (g_scratch_offset + size > g_scratch_size) {
            g_scratch_offset = 0;
        }

        char * data = (char *) g_scratch_buffer;
        if (data == nullptr) {
            CUDA_CHECK(hipMalloc(&data, g_scratch_size));
            g_scratch_buffer = data;
        }
        extra->data_device[g_main_device] = data + g_scratch_offset;

        g_scratch_offset += size;

        GGML_ASSERT(g_scratch_offset <= g_scratch_size);
    } else { // allocate new buffers outside of scratch
        void * data;
        CUDA_CHECK(hipMalloc(&data, size));
        CUDA_CHECK(hipMemset(data, 0, size));
        extra->data_device[g_main_device] = data;
    }

    tensor->extra = extra;
}

void ggml_cuda_assign_buffers(struct ggml_tensor * tensor) {
    ggml_cuda_assign_buffers_impl(tensor, true);
}

void ggml_cuda_assign_buffers_no_scratch(struct ggml_tensor * tensor) {
    ggml_cuda_assign_buffers_impl(tensor, false);
}

void ggml_cuda_set_main_device(int main_device) {
    if (main_device >= g_device_count) {
        fprintf(stderr, "warning: cannot set main_device=%d because there are only %d devices. Using device %d instead.\n",
                main_device, g_device_count, g_main_device);
        return;
    }
    g_main_device = main_device;
    if (g_device_count > 1) {
        hipDeviceProp_t prop;
        CUDA_CHECK(hipGetDeviceProperties(&prop, g_main_device));
        fprintf(stderr, "%s: using device %d (%s) as main device\n", __func__, g_main_device, prop.name);
    }
}

void ggml_cuda_set_scratch_size(size_t scratch_size) {
    g_scratch_size = scratch_size;
}

void ggml_cuda_free_scratch() {
    if (g_scratch_buffer == nullptr) {
        return;
    }

    CUDA_CHECK(hipFree(g_scratch_buffer));
    g_scratch_buffer = nullptr;
}

bool ggml_cuda_compute_forward(struct ggml_compute_params * params, struct ggml_tensor * tensor){
    ggml_cuda_func_t func;
    const bool any_on_device = tensor->backend == GGML_BACKEND_GPU
        || tensor->src0->backend == GGML_BACKEND_GPU || tensor->src0->backend == GGML_BACKEND_GPU_SPLIT
        || (tensor->src1 != nullptr && tensor->src1->backend == GGML_BACKEND_GPU);

    switch (tensor->op) {
        case GGML_OP_ADD:
            if (!any_on_device) {
                return false;
            }
            func = ggml_cuda_add;
            break;
        case GGML_OP_MUL:
            if (!any_on_device) {
                return false;
            }
            func = ggml_cuda_mul;
            break;
        case GGML_OP_SILU:
            if (!any_on_device) {
                return false;
            }
            func = ggml_cuda_silu;
            break;
        case GGML_OP_RMS_NORM:
            if (!any_on_device) {
                return false;
            }
            func = ggml_cuda_rms_norm;
            break;
        case GGML_OP_MUL_MAT:
            if (!any_on_device && !ggml_cuda_can_mul_mat(tensor->src0, tensor->src1, tensor)) {
                return false;
            }
            func = ggml_cuda_mul_mat;
            break;
        case GGML_OP_SCALE:
            if (!any_on_device) {
                return false;
            }
            func = ggml_cuda_scale;
            break;
        case GGML_OP_CPY:
            if (!any_on_device) {
                return false;
            }
            func = ggml_cuda_cpy;
            break;
        case GGML_OP_RESHAPE:
        case GGML_OP_VIEW:
        case GGML_OP_PERMUTE:
        case GGML_OP_TRANSPOSE:
            if (!any_on_device) {
                return false;
            }
            func = ggml_cuda_nop;
            break;
        case GGML_OP_DIAG_MASK_INF:
            if (!any_on_device) {
                return false;
            }
            func = ggml_cuda_diag_mask_inf;
            break;
        case GGML_OP_SOFT_MAX:
            if (!any_on_device) {
                return false;
            }
            func = ggml_cuda_soft_max;
            break;
        case GGML_OP_ROPE:
            if (!any_on_device) {
                return false;
            }
            func = ggml_cuda_rope;
            break;
        default:
            return false;
    }

    if (params->ith != 0) {
        return true;
    }
    if (params->type == GGML_TASK_INIT || params->type == GGML_TASK_FINALIZE) {
        return true;
    }
    func(tensor->src0, tensor->src1, tensor);
    return true;
}

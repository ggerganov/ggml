#include "hip/hip_runtime.h"
#include <cstddef>
#include <cstdint>
#include <stdint.h>
#include <stdio.h>
#include <atomic>

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_fp16.h>

#include "ggml-cuda.h"
#include "ggml.h"

static_assert(sizeof(half) == sizeof(ggml_fp16_t), "wrong fp16 size");

#define CUDA_CHECK(err)                                                                 \
    do {                                                                                \
        hipError_t err_ = (err);                                                       \
        if (err_ != hipSuccess) {                                                      \
            fprintf(stderr, "CUDA error %d at %s:%d: %s\n", err_, __FILE__, __LINE__,   \
                hipGetErrorString(err_));                                              \
            exit(1);                                                                    \
        }                                                                               \
    } while (0)

#define CUBLAS_CHECK(err)                                                               \
    do {                                                                                \
        hipblasStatus_t err_ = (err);                                                    \
        if (err_ != HIPBLAS_STATUS_SUCCESS) {                                            \
            fprintf(stderr, "cuBLAS error %d at %s:%d\n", err_, __FILE__, __LINE__);    \
            exit(1);                                                                    \
        }                                                                               \
    } while (0)

typedef void (*dequantize_kernel_t)(const void * vx, const int ib, const int iqs, float & v0, float & v1);
typedef void (*to_fp32_cuda_t)(const void * x, float * y, int k, hipStream_t stream);
typedef void (*dequantize_mul_mat_vec_cuda_t)(const void * vx, const float * y, float * dst, const int ncols, const int nrows, hipStream_t stream);

// QK = number of values after dequantization
// QR = QK / number of values before dequantization

#define QK4_0 32
#define QR4_0 2
typedef struct {
    float   d;              // delta
    uint8_t qs[QK4_0 / 2];  // nibbles / quants
} block_q4_0;
static_assert(sizeof(block_q4_0) == sizeof(float) + QK4_0 / 2, "wrong q4_0 block size/padding");

#define QK4_1 32
#define QR4_1 2
typedef struct {
    float   d;              // delta
    float   m;              // min
    uint8_t qs[QK4_1 / 2];  // nibbles / quants
} block_q4_1;
static_assert(sizeof(block_q4_1) == sizeof(float) * 2 + QK4_1 / 2, "wrong q4_1 block size/padding");

#define QK5_0 32
#define QR5_0 2
typedef struct {
    half d;                 // delta
    uint8_t qh[4];          // 5-th bit of quants
    uint8_t qs[QK5_0 / 2];  // nibbles / quants
} block_q5_0;
static_assert(sizeof(block_q5_0) == sizeof(ggml_fp16_t) + sizeof(uint32_t) + QK5_0 / 2, "wrong q5_0 block size/padding");

#define QK5_1 32
#define QR5_1 2
typedef struct {
    half d;                 // delta
    half m;                 // min
    uint8_t qh[4];          // 5-th bit of quants
    uint8_t qs[QK5_1 / 2];  // nibbles / quants
} block_q5_1;
static_assert(sizeof(block_q5_1) == 2 * sizeof(ggml_fp16_t) + sizeof(uint32_t) + QK5_1 / 2, "wrong q5_1 block size/padding");

#define QK8_0 32
#define QR8_0 1
typedef struct {
    float   d;              // delta
    int8_t  qs[QK8_0];      // quants
} block_q8_0;
static_assert(sizeof(block_q8_0) == sizeof(float) + QK8_0, "wrong q8_0 block size/padding");

#define CUDA_DMMV_BLOCK_SIZE 32

static __device__ void dequantize_q4_0(const void * vx, const int ib, const int iqs, float & v0, float & v1){
    const block_q4_0 * x = (const block_q4_0 *) vx;

    const float d = x[ib].d;

    const uint8_t vui = x[ib].qs[iqs];

    const int8_t vi0 = vui & 0xF;
    const int8_t vi1 = vui >> 4;

    v0 = (vi0 - 8)*d;
    v1 = (vi1 - 8)*d;
}

static __device__ void dequantize_q4_1(const void * vx, const int ib, const int iqs, float & v0, float & v1){
    const block_q4_1 * x = (const block_q4_1 *) vx;

    const float d = x[ib].d;
    const float m = x[ib].m;

    const uint8_t vui = x[ib].qs[iqs];

    const int8_t vi0 = vui & 0xF;
    const int8_t vi1 = vui >> 4;

    v0 = vi0*d + m;
    v1 = vi1*d + m;
}

static __device__ void dequantize_q5_0(const void * vx, const int ib, const int iqs, float & v0, float & v1){
    const block_q5_0 * x = (const block_q5_0 *) vx;

    const float d = x[ib].d;

    uint32_t qh;
    memcpy(&qh, x[ib].qh, sizeof(qh));

    const uint8_t xh_0 = ((qh >> (iqs +  0)) << 4) & 0x10;
    const uint8_t xh_1 = ((qh >> (iqs + 12))     ) & 0x10;

    const int32_t x0 = ((x[ib].qs[iqs] & 0xf) | xh_0) - 16;
    const int32_t x1 = ((x[ib].qs[iqs] >>  4) | xh_1) - 16;

    v0 = x0*d;
    v1 = x1*d;
}

static __device__ void dequantize_q5_1(const void * vx, const int ib, const int iqs, float & v0, float & v1){
    const block_q5_1 * x = (const block_q5_1 *) vx;

    const float d = x[ib].d;
    const float m = x[ib].m;

    uint32_t qh;
    memcpy(&qh, x[ib].qh, sizeof(qh));

    const uint8_t xh_0 = ((qh >> (iqs +  0)) << 4) & 0x10;
    const uint8_t xh_1 = ((qh >> (iqs + 12))     ) & 0x10;

    const int32_t x0 = ((x[ib].qs[iqs] & 0xf) | xh_0);
    const int32_t x1 = ((x[ib].qs[iqs] >>  4) | xh_1);

    v0 = x0*d + m;
    v1 = x1*d + m;
}

static __device__ void dequantize_q8_0(const void * vx, const int ib, const int iqs, float & v0, float & v1){
    const block_q8_0 * x = (const block_q8_0 *) vx;

    const float d = x[ib].d;

    const int8_t vi0 = x[ib].qs[iqs + 0];
    const int8_t vi1 = x[ib].qs[iqs + 1];

    v0 = vi0*d;
    v1 = vi1*d;
}

static __device__ void convert_f16(const void * vx, const int ib, const int iqs, float & v0, float & v1){
    const half * x = (const half *) vx;

    v0 = __half2float(x[ib + 0]);
    v1 = __half2float(x[ib + 1]);
}

static __global__ void dequantize_block_q4_0(const void * vx, float * y) {
    static const int qk = QK4_0;

    const block_q4_0 * x = (const block_q4_0 *) vx;

    const int i = blockIdx.x;

    const float d = x[i].d;

    for (int j = 0; j < qk/2; ++j) {
        const int x0 = (x[i].qs[j] & 0xf) - 8;
        const int x1 = (x[i].qs[j] >>  4) - 8;

        y[i*qk + j + 0   ] = x0*d;
        y[i*qk + j + qk/2] = x1*d;
    }
}

static __global__ void dequantize_block_q4_1(const void * vx, float * y) {
    static const int qk = QK4_1;

    const block_q4_1 * x = (const block_q4_1 *) vx;

    const int i = blockIdx.x;

    const float d = x[i].d;
    const float m = x[i].m;

    for (int j = 0; j < qk/2; ++j) {
        const int x0 = (x[i].qs[j] & 0xf);
        const int x1 = (x[i].qs[j] >>  4);

        y[i*qk + j + 0   ] = x0*d + m;
        y[i*qk + j + qk/2] = x1*d + m;
    }
}

static __global__ void dequantize_block_q5_0(const void * vx, float * y) {
    static const int qk = QK5_0;

    const block_q5_0 * x = (const block_q5_0 *) vx;

    const int i = blockIdx.x;

    const float d = x[i].d;

    uint32_t qh;
    memcpy(&qh, x[i].qh, sizeof(qh));

    for (int j = 0; j < qk/2; ++j) {
        const uint8_t xh_0 = ((qh >> (j +  0)) << 4) & 0x10;
        const uint8_t xh_1 = ((qh >> (j + 12))     ) & 0x10;

        const int32_t x0 = ((x[i].qs[j] & 0xf) | xh_0) - 16;
        const int32_t x1 = ((x[i].qs[j] >>  4) | xh_1) - 16;

        y[i*qk + j + 0   ] = x0*d;
        y[i*qk + j + qk/2] = x1*d;
    }
}

static __global__ void dequantize_block_q5_1(const void * vx, float * y) {
    static const int qk = QK5_1;

    const block_q5_1 * x = (const block_q5_1 *) vx;

    const int i = blockIdx.x;

    const float d = x[i].d;
    const float m = x[i].m;

    uint32_t qh;
    memcpy(&qh, x[i].qh, sizeof(qh));

    for (int j = 0; j < qk/2; ++j) {
        const uint8_t xh_0 = ((qh >> (j +  0)) << 4) & 0x10;
        const uint8_t xh_1 = ((qh >> (j + 12))     ) & 0x10;

        const int x0 = (x[i].qs[j] & 0xf) | xh_0;
        const int x1 = (x[i].qs[j] >>  4) | xh_1;

        y[i*qk + j + 0   ] = x0*d + m;
        y[i*qk + j + qk/2] = x1*d + m;
    }
}

static __global__ void dequantize_block_q8_0(const void * vx, float * y) {
    static const int qk = QK8_0;

    const block_q8_0 * x = (const block_q8_0 *) vx;

    const int i = blockIdx.x;

    const float d = x[i].d;

    for (int j = 0; j < qk; ++j) {
        y[i*qk + j] = x[i].qs[j]*d;
    }
}

template <int block_size, int qk, int qr, dequantize_kernel_t dequantize_kernel>
static __global__ void dequantize_mul_mat_vec(const void * vx, const float * y, float * dst, const int ncols) {
    const int row = blockIdx.x;
    const int tid = threadIdx.x;

    const int y_offset = qr == 1 ? 1 : qk/2;

    __shared__ float tmp[block_size]; // separate sum for each thread
    tmp[tid] = 0;

    for (int i = 0; i < ncols/block_size; i += 2) {
        const int col = i*block_size + 2*tid;
        const int ib = (row*ncols + col)/qk; // block index
        const int iqs = (col%qk)/qr; // quant index
        const int iybs = col - col%qk; // y block start index

        // dequantize
        float v0, v1;
        dequantize_kernel(vx, ib, iqs, v0, v1);

        // matrix multiplication
        tmp[tid] += v0 * y[iybs + iqs + 0];
        tmp[tid] += v1 * y[iybs + iqs + y_offset];
    }

    // sum up partial sums and write back result
    __syncthreads();
    for (int s=block_size/2; s>0; s>>=1) {
        if (tid < s) {
            tmp[tid] += tmp[tid + s];
        }
        __syncthreads();
    }
    if (tid == 0) {
        dst[row] = tmp[0];
    }
}

static void dequantize_row_q4_0_cuda(const void * vx, float * y, int k, hipStream_t stream) {
    const int nb = k / QK4_0;
    dequantize_block_q4_0<<<nb, 1, 0, stream>>>(vx, y);
}

static void dequantize_row_q4_1_cuda(const void * vx, float * y, int k, hipStream_t stream) {
    const int nb = k / QK4_1;
    dequantize_block_q4_1<<<nb, 1, 0, stream>>>(vx, y);
}

static void dequantize_row_q5_0_cuda(const void * vx, float * y, int k, hipStream_t stream) {
    const int nb = k / QK5_0;
    dequantize_block_q5_0<<<nb, 1, 0, stream>>>(vx, y);
}

static void dequantize_row_q5_1_cuda(const void * vx, float * y, int k, hipStream_t stream) {
    const int nb = k / QK5_1;
    dequantize_block_q5_1<<<nb, 1, 0, stream>>>(vx, y);
}

static void dequantize_row_q8_0_cuda(const void * vx, float * y, int k, hipStream_t stream) {
    const int nb = k / QK8_0;
    dequantize_block_q8_0<<<nb, 1, 0, stream>>>(vx, y);
}

static void dequantize_mul_mat_vec_q4_0_cuda(const void * vx, const float * y, float * dst, const int ncols, const int nrows, hipStream_t stream) {
    GGML_ASSERT(ncols % CUDA_DMMV_BLOCK_SIZE == 0);
    dequantize_mul_mat_vec<CUDA_DMMV_BLOCK_SIZE, QK4_0, QR4_0, dequantize_q4_0>
        <<<nrows, CUDA_DMMV_BLOCK_SIZE, 0, stream>>>(vx, y, dst, ncols);
}

static void dequantize_mul_mat_vec_q4_1_cuda(const void * vx, const float * y, float * dst, const int ncols, const int nrows, hipStream_t stream) {
    GGML_ASSERT(ncols % CUDA_DMMV_BLOCK_SIZE == 0);
    dequantize_mul_mat_vec<CUDA_DMMV_BLOCK_SIZE, QK4_1, QR4_1, dequantize_q4_1>
        <<<nrows, CUDA_DMMV_BLOCK_SIZE, 0, stream>>>(vx, y, dst, ncols);
}

static void dequantize_mul_mat_vec_q5_0_cuda(const void * vx, const float * y, float * dst, const int ncols, const int nrows, hipStream_t stream) {
    GGML_ASSERT(ncols % CUDA_DMMV_BLOCK_SIZE == 0);
    dequantize_mul_mat_vec<CUDA_DMMV_BLOCK_SIZE, QK5_0, QR5_0, dequantize_q5_0>
        <<<nrows, CUDA_DMMV_BLOCK_SIZE, 0, stream>>>(vx, y, dst, ncols);
}

static void dequantize_mul_mat_vec_q5_1_cuda(const void * vx, const float * y, float * dst, const int ncols, const int nrows, hipStream_t stream) {
    GGML_ASSERT(ncols % CUDA_DMMV_BLOCK_SIZE == 0);
    dequantize_mul_mat_vec<CUDA_DMMV_BLOCK_SIZE, QK5_1, QR5_1, dequantize_q5_1>
        <<<nrows, CUDA_DMMV_BLOCK_SIZE, 0, stream>>>(vx, y, dst, ncols);
}

static void dequantize_mul_mat_vec_q8_0_cuda(const void * vx, const float * y, float * dst, const int ncols, const int nrows, hipStream_t stream) {
    GGML_ASSERT(ncols % CUDA_DMMV_BLOCK_SIZE == 0);
    dequantize_mul_mat_vec<CUDA_DMMV_BLOCK_SIZE, QK8_0, QR8_0, dequantize_q8_0>
        <<<nrows, CUDA_DMMV_BLOCK_SIZE, 0, stream>>>(vx, y, dst, ncols);
}

// TODO: optimize
static __global__ void convert_fp16_to_fp32(const void * vx, float * y) {
    const half * x = (const half *) vx;

    const int i = blockIdx.x;

    y[i] = __half2float(x[i]);
}

static void convert_fp16_to_fp32_cuda(const void * x, float * y, int k, hipStream_t stream) {
    convert_fp16_to_fp32<<<k, 1, 0, stream>>>(x, y);
}

static void convert_mul_mat_vec_f16_cuda(const void * vx, const float * y, float * dst, const int ncols, const int nrows, hipStream_t stream) {
    GGML_ASSERT(ncols % CUDA_DMMV_BLOCK_SIZE == 0);
    dequantize_mul_mat_vec<CUDA_DMMV_BLOCK_SIZE, 32, 1, convert_f16>
        <<<nrows, CUDA_DMMV_BLOCK_SIZE, 0, stream>>>(vx, y, dst, ncols);
}

static to_fp32_cuda_t ggml_get_to_fp32_cuda(ggml_type type) {
    switch (type) {
        case GGML_TYPE_Q4_0:
            return dequantize_row_q4_0_cuda;
        case GGML_TYPE_Q4_1:
            return dequantize_row_q4_1_cuda;
        case GGML_TYPE_Q5_0:
            return dequantize_row_q5_0_cuda;
        case GGML_TYPE_Q5_1:
            return dequantize_row_q5_1_cuda;
        case GGML_TYPE_Q8_0:
            return dequantize_row_q8_0_cuda;
        case GGML_TYPE_F16:
            return convert_fp16_to_fp32_cuda;
        default:
            return nullptr;
    }
}

static dequantize_mul_mat_vec_cuda_t ggml_get_dequantize_mul_mat_vec_cuda(ggml_type type) {
    switch (type) {
        case GGML_TYPE_Q4_0:
            return dequantize_mul_mat_vec_q4_0_cuda;
        case GGML_TYPE_Q4_1:
            return dequantize_mul_mat_vec_q4_1_cuda;
        case GGML_TYPE_Q5_0:
            return dequantize_mul_mat_vec_q5_0_cuda;
        case GGML_TYPE_Q5_1:
            return dequantize_mul_mat_vec_q5_1_cuda;
        case GGML_TYPE_Q8_0:
            return dequantize_mul_mat_vec_q8_0_cuda;
        case GGML_TYPE_F16:
            return convert_mul_mat_vec_f16_cuda;
        default:
            return nullptr;
    }
}

// buffer pool for cuda
#define MAX_CUDA_BUFFERS 256

struct scoped_spin_lock {
    std::atomic_flag& lock;
    scoped_spin_lock(std::atomic_flag& lock) : lock(lock) {
        while (lock.test_and_set(std::memory_order_acquire)) {
            ; // spin
        }
    }
    ~scoped_spin_lock() {
        lock.clear(std::memory_order_release);
    }
    scoped_spin_lock(const scoped_spin_lock&) = delete;
    scoped_spin_lock& operator=(const scoped_spin_lock&) = delete;
};

struct cuda_buffer {
    void * ptr = nullptr;
    size_t size = 0;
};

static cuda_buffer g_cuda_buffer_pool[MAX_CUDA_BUFFERS];
static std::atomic_flag g_cuda_pool_lock = ATOMIC_FLAG_INIT;

static void * ggml_cuda_pool_malloc(size_t size, size_t * actual_size) {
    scoped_spin_lock lock(g_cuda_pool_lock);

    for (int i = 0; i < MAX_CUDA_BUFFERS; ++i) {
        cuda_buffer& b = g_cuda_buffer_pool[i];
        if (b.size >= size && b.ptr != nullptr) {
            void * ptr = b.ptr;
            *actual_size = b.size;
            b.ptr = nullptr;
            b.size = 0;
            return ptr;
        }
    }
    void * ptr;
    CUDA_CHECK(hipMalloc((void **) &ptr, size));
    *actual_size = size;
    return ptr;
}

static void ggml_cuda_pool_free(void * ptr, size_t size) {
    scoped_spin_lock lock(g_cuda_pool_lock);

    for (int i = 0; i < MAX_CUDA_BUFFERS; ++i) {
        cuda_buffer& b = g_cuda_buffer_pool[i];
        if (b.ptr == nullptr) {
            b.ptr = ptr;
            b.size = size;
            return;
        }
    }
    fprintf(stderr, "WARNING: cuda buffer pool full, increase MAX_CUDA_BUFFERS\n");
    CUDA_CHECK(hipFree(ptr));
}

#define GGML_CUDA_MAX_STREAMS 8 // Set this to 1 for reproducible matrix multiplication.
#define GGML_CUDA_MAX_EVENTS 64
static hipblasHandle_t g_cublasH = nullptr;
static hipStream_t g_cudaStreams[GGML_CUDA_MAX_STREAMS] = { nullptr };
static hipStream_t g_cudaStreams2[GGML_CUDA_MAX_STREAMS] = { nullptr };
static hipEvent_t g_cudaEvents[GGML_CUDA_MAX_EVENTS] = { nullptr };

void ggml_init_cublas() {
    if (g_cublasH == nullptr) {
        // create streams
        for (int i = 0; i < GGML_CUDA_MAX_STREAMS; ++i) {
            CUDA_CHECK(hipStreamCreateWithFlags(&g_cudaStreams[i], hipStreamNonBlocking));
            CUDA_CHECK(hipStreamCreateWithFlags(&g_cudaStreams2[i], hipStreamNonBlocking));
        }
        // create events
        for (int i = 0; i < GGML_CUDA_MAX_EVENTS; ++i) {
            CUDA_CHECK(hipEventCreateWithFlags(&g_cudaEvents[i], hipEventDisableTiming));
        }

        // create cublas handle
        CUBLAS_CHECK(hipblasCreate(&g_cublasH));
        CUBLAS_CHECK(hipblasSetMathMode(g_cublasH, HIPBLAS_TF32_TENSOR_OP_MATH));

        // configure logging to stdout
        // CUBLAS_CHECK(cublasLoggerConfigure(1, 1, 0, nullptr));
    }
}

void * ggml_cuda_host_malloc(size_t size) {
    if (getenv("GGML_CUDA_NO_PINNED") != nullptr) {
        return nullptr;
    }

    void * ptr = nullptr;
    hipError_t err = hipHostMalloc((void **) &ptr, size);
    if (err != hipSuccess) {
        fprintf(stderr, "WARNING: failed to allocate %.2f MB of pinned memory: %s\n",
            size/1024.0/1024.0, hipGetErrorString(err));
        return nullptr;
    }

    return ptr;
}

void ggml_cuda_host_free(void * ptr) {
    CUDA_CHECK(hipHostFree(ptr));
}

static hipError_t ggml_cuda_h2d_tensor_2d(void * dst, const struct ggml_tensor * src, uint64_t i3, uint64_t i2, hipStream_t stream) {
    const uint64_t ne0 = src->ne[0];
    const uint64_t ne1 = src->ne[1];
    const uint64_t nb0 = src->nb[0];
    const uint64_t nb1 = src->nb[1];
    const uint64_t nb2 = src->nb[2];
    const uint64_t nb3 = src->nb[3];
    const enum ggml_type type = src->type;
    const size_t ts = ggml_type_size(type);
    const size_t bs = ggml_blck_size(type);

    const void * x = (const void *) ((const char *) src->data + i2*nb2 + i3*nb3);
    if (nb0 == ts && nb1 == ts*ne0/bs) {
        return hipMemcpyAsync(dst, x, ne1*nb1, hipMemcpyHostToDevice, stream);
    } else if (nb0 == ts) {
        return hipMemcpy2DAsync(dst, ts*ne0/bs, x, nb1, ts*ne0/bs, ne1, hipMemcpyHostToDevice, stream);
    } else {
        for (uint64_t i1 = 0; i1 < ne1; i1++) {
            const void * rx = (const void *) ((const char *) x + i1*nb1);
            void * rd = (void *) ((char *) dst + i1*ts*ne0/bs);
            // pretend the row is a matrix with cols=1
            hipError_t r = hipMemcpy2DAsync(rd, ts/bs, rx, nb0, ts/bs, ne0, hipMemcpyHostToDevice, stream);
            if (r != hipSuccess) return r;
        }
        return hipSuccess;
    }
}

static void ggml_cuda_mul_mat_f32(const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst) {
    const int64_t ne00 = src0->ne[0];
    const int64_t ne01 = src0->ne[1];
    const int64_t ne02 = src0->ne[2];
    const int64_t ne03 = src0->ne[3];

    const int64_t ne10 = src1->ne[0];
    const int64_t ne11 = src1->ne[1];

    const int nb2  = dst->nb[2];
    const int nb3  = dst->nb[3];

    const float alpha = 1.0f;
    const float beta = 0.0f;
    const int x_ne = ne01 * ne00;
    const int y_ne = ne11 * ne10;
    const int d_ne = ne11 * ne01;
    const int n_mm = ne03 * ne02;

    size_t x_size, y_size, d_size;
    float * d_X = (float *) ggml_cuda_pool_malloc(n_mm * sizeof(float) * x_ne, &x_size);
    float * d_Y = (float *) ggml_cuda_pool_malloc(n_mm * sizeof(float) * y_ne, &y_size);
    float * d_D = (float *) ggml_cuda_pool_malloc(n_mm * sizeof(float) * d_ne, &d_size);

    for (int64_t i03 = 0; i03 < ne03; i03++) {
        for (int64_t i02 = 0; i02 < ne02; i02++) {
            int i = i03*ne02 + i02;
            hipStream_t cudaStream = g_cudaStreams[i % GGML_CUDA_MAX_STREAMS];

            float * c_X = d_X + i * x_ne;
            float * c_Y = d_Y + i * y_ne;
            float * c_D = d_D + i * d_ne;

            // copy data to device
            CUDA_CHECK(ggml_cuda_h2d_tensor_2d(c_X, src0, i03, i02, cudaStream));
            CUDA_CHECK(ggml_cuda_h2d_tensor_2d(c_Y, src1, i03, i02, cudaStream));

            // compute
            CUBLAS_CHECK(hipblasSetStream(g_cublasH, cudaStream));
            CUBLAS_CHECK(
                hipblasSgemm(g_cublasH, HIPBLAS_OP_T, HIPBLAS_OP_N,
                        ne01, ne11, ne10,
                        &alpha, c_X, ne00,
                                c_Y, ne10,
                        &beta,  c_D, ne01));

            // copy dst to host
            float * d = (float *) ((char *) dst->data + i02*nb2 + i03*nb3);
            CUDA_CHECK(hipMemcpyAsync(d, c_D, sizeof(float) * d_ne, hipMemcpyDeviceToHost, cudaStream));
        }
    }

    CUDA_CHECK(hipDeviceSynchronize());
    ggml_cuda_pool_free(d_X, x_size);
    ggml_cuda_pool_free(d_Y, y_size);
    ggml_cuda_pool_free(d_D, d_size);
}

static void ggml_cuda_mul_mat_f16(const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst, void * wdata, size_t /* wsize */) {
    const int64_t ne00 = src0->ne[0];
    const int64_t ne01 = src0->ne[1];
    const int64_t ne02 = src0->ne[2];
    const int64_t ne03 = src0->ne[3];

    const int64_t ne10 = src1->ne[0];
    const int64_t ne11 = src1->ne[1];

    const int nb10 = src1->nb[0];
    const int nb11 = src1->nb[1];
    const int nb12 = src1->nb[2];
    const int nb13 = src1->nb[3];

    const int nb2  = dst->nb[2];
    const int nb3  = dst->nb[3];

    const float alpha = 1.0f;
    const float beta = 0.0f;
    const int x_ne = ne01 * ne00;
    const int y_ne = ne11 * ne10;
    const int d_ne = ne11 * ne01;
    const int n_mm = ne03 * ne02;

    size_t x_size, y_size, d_size;
    half  * d_X =  (half *) ggml_cuda_pool_malloc(n_mm * sizeof(half) * x_ne, &x_size);
    half  * d_Y =  (half *) ggml_cuda_pool_malloc(n_mm * sizeof(half) * y_ne, &y_size);
    float * d_D = (float *) ggml_cuda_pool_malloc(n_mm * sizeof(float) * d_ne, &d_size);

    bool src1_cont_rows = nb10 == sizeof(float);
    bool src1_cont_cols = (size_t)nb11 == ne11*sizeof(float);

    for (int64_t i03 = 0; i03 < ne03; i03++) {
        for (int64_t i02 = 0; i02 < ne02; i02++) {
            int i = i03*ne02 + i02;
            hipStream_t cudaStream = g_cudaStreams[i % GGML_CUDA_MAX_STREAMS];

            half  * c_X = d_X + i * x_ne;
            half  * c_Y = d_Y + i * y_ne;
            float * c_D = d_D + i * d_ne;

            // copy src0 to device
            CUDA_CHECK(ggml_cuda_h2d_tensor_2d(c_X, src0, i03, i02, cudaStream));

            // convert src1 to fp16
            // TODO: use multiple threads
            ggml_fp16_t * const tmp = (ggml_fp16_t *) wdata + (ne11 * ne10) * (i03 * ne02 + i02);
            char * src1i = (char *) src1->data + i03*nb13 + i02*nb12;
            if (src1_cont_rows) {
                if (src1_cont_cols) {
                    ggml_fp32_to_fp16_row((float *) src1i, tmp, ne10*ne11);
                }
                else {
                    for (int64_t i01 = 0; i01 < ne11; i01++) {
                        ggml_fp32_to_fp16_row((float *) (src1i + i01*nb11), tmp + i01*ne10, ne10);
                    }
                }
            }
            else {
                for (int64_t i01 = 0; i01 < ne11; i01++) {
                    for (int64_t i00 = 0; i00 < ne10; i00++) {
                        // very slow due to no inlining
                        tmp[i01*ne10 + i00] = ggml_fp32_to_fp16(*(float *) (src1i + i01*nb11 + i00*nb10));
                    }
                }
            }

            // copy src1 to device
            CUDA_CHECK(hipMemcpyAsync(c_Y, tmp, sizeof(half) * y_ne, hipMemcpyHostToDevice, cudaStream));

            // compute
            CUBLAS_CHECK(hipblasSetStream(g_cublasH, cudaStream));
            CUBLAS_CHECK(
                hipblasGemmEx(g_cublasH, HIPBLAS_OP_T, HIPBLAS_OP_N,
                        ne01, ne11, ne10,
                        &alpha, c_X, HIP_R_16F, ne00,
                                c_Y, HIP_R_16F, ne10,
                        &beta,  c_D, HIP_R_32F, ne01,
                        HIPBLAS_COMPUTE_32F_FAST_16F,
                        HIPBLAS_GEMM_DEFAULT));

            // copy dst to host
            float * d = (float *) ((char *) dst->data + i02*nb2 + i03*nb3);
            CUDA_CHECK(hipMemcpyAsync(d, c_D, sizeof(float) * d_ne, hipMemcpyDeviceToHost, cudaStream));
        }
    }

    CUDA_CHECK(hipDeviceSynchronize());
    ggml_cuda_pool_free(d_X, x_size);
    ggml_cuda_pool_free(d_Y, y_size);
    ggml_cuda_pool_free(d_D, d_size);
}

static void ggml_cuda_mul_mat_q_f32(const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst) {
    const int64_t ne00 = src0->ne[0];
    const int64_t ne01 = src0->ne[1];
    const int64_t ne02 = src0->ne[2];
    const int64_t ne03 = src0->ne[3];

    const int64_t ne10 = src1->ne[0];
    const int64_t ne11 = src1->ne[1];

    const int nb2  = dst->nb[2];
    const int nb3  = dst->nb[3];
    const ggml_type type = src0->type;
    const bool mul_mat_vec = ne11 == 1;

    const float alpha = 1.0f;
    const float beta = 0.0f;
    const int x_ne = ne01 * ne00;
    const int y_ne = ne11 * ne10;
    const int d_ne = ne11 * ne01;
    const int n_mm = ne03 * ne02;
    const size_t q_sz = ggml_type_size(type) * x_ne / ggml_blck_size(type);

    size_t x_size, y_size, d_size, q_size;
    float * d_X = nullptr;
    if (!mul_mat_vec) {
        d_X = (float *) ggml_cuda_pool_malloc(n_mm * sizeof(float) * x_ne, &x_size);
    }
    float * d_Y = (float *) ggml_cuda_pool_malloc(n_mm * sizeof(float) * y_ne, &y_size);
    float * d_D = (float *) ggml_cuda_pool_malloc(n_mm * sizeof(float) * d_ne, &d_size);
    char  * d_Q = (char  *) ggml_cuda_pool_malloc(n_mm * q_sz, &q_size);

    const to_fp32_cuda_t to_fp32_cuda = ggml_get_to_fp32_cuda(type);
    dequantize_mul_mat_vec_cuda_t dmmv = ggml_get_dequantize_mul_mat_vec_cuda(type);
    GGML_ASSERT(to_fp32_cuda != nullptr);

    for (int64_t i03 = 0; i03 < ne03; i03++) {
        for (int64_t i02 = 0; i02 < ne02; i02++) {
            int i = i03*ne02 + i02;
            hipStream_t cudaStream = g_cudaStreams[i % GGML_CUDA_MAX_STREAMS];
            hipStream_t cudaStream2 = g_cudaStreams2[i % GGML_CUDA_MAX_STREAMS];
            hipEvent_t  cudaEvent = g_cudaEvents[i % GGML_CUDA_MAX_EVENTS];

            float * c_Y = d_Y + i * y_ne;
            float * c_D = d_D + i * d_ne;
            char  * c_Q = d_Q + i * q_sz;

            // copy src0 to device if necessary
            if (src0->backend == GGML_BACKEND_CPU) {
                CUDA_CHECK(ggml_cuda_h2d_tensor_2d(c_Q, src0, i03, i02, cudaStream2));
            } else if (src0->backend == GGML_BACKEND_CUDA) {
                c_Q = ((char *) src0->data) + i * q_sz;
            } else {
                GGML_ASSERT(false);
            }
            if (mul_mat_vec) { // specialized dequantize_mul_mat_vec kernel
                CUDA_CHECK(hipEventRecord(cudaEvent, cudaStream2));

                // copy src1 to device
                CUDA_CHECK(ggml_cuda_h2d_tensor_2d(c_Y, src1, i03, i02, cudaStream));

                // wait for data
                CUDA_CHECK(hipStreamWaitEvent(cudaStream, cudaEvent, 0));

                // compute
                dmmv(c_Q, c_Y, c_D, ne00, ne01, cudaStream);
                CUDA_CHECK(hipGetLastError());

            } else { // general dequantization kernel + cuBLAS matrix matrix multiplication
                float * c_X = d_X + i * x_ne;

                // convert src0 to fp32 on device
                to_fp32_cuda(c_Q, c_X, x_ne, cudaStream2);
                CUDA_CHECK(hipGetLastError());
                CUDA_CHECK(hipEventRecord(cudaEvent, cudaStream2));

                // copy src1 to device
                CUDA_CHECK(ggml_cuda_h2d_tensor_2d(c_Y, src1, i03, i02, cudaStream));

                // wait for conversion
                CUDA_CHECK(hipStreamWaitEvent(cudaStream, cudaEvent, 0));

                // compute
                CUBLAS_CHECK(hipblasSetStream(g_cublasH, cudaStream));
                CUBLAS_CHECK(
                    hipblasSgemm(g_cublasH, HIPBLAS_OP_T, HIPBLAS_OP_N,
                            ne01, ne11, ne10,
                            &alpha, c_X, ne00,
                                    c_Y, ne10,
                            &beta,  c_D, ne01));
            }

            // copy dst to host
            float * d = (float *) ((char *) dst->data + i02*nb2 + i03*nb3);
            CUDA_CHECK(hipMemcpyAsync(d, c_D, sizeof(float) * d_ne, hipMemcpyDeviceToHost, cudaStream));
        }
    }

    CUDA_CHECK(hipDeviceSynchronize());
    if (!mul_mat_vec) {
        ggml_cuda_pool_free(d_X, x_size);
    }
    ggml_cuda_pool_free(d_Y, y_size);
    ggml_cuda_pool_free(d_D, d_size);
    ggml_cuda_pool_free(d_Q, q_size);
}

bool ggml_cuda_can_mul_mat(const struct ggml_tensor * src0, const struct ggml_tensor * src1, struct ggml_tensor * dst) {
    const int64_t ne10 = src1->ne[0];

    const int64_t ne0 = dst->ne[0];
    const int64_t ne1 = dst->ne[1];

    // TODO: find the optimal values for these
    if ((src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16 || ggml_is_quantized(src0->type)) &&
        src1->type == GGML_TYPE_F32 &&
        dst->type == GGML_TYPE_F32 &&
        ((ne0 >= 32 && ne1 >= 32 && ne10 >= 32) || src0->backend == GGML_BACKEND_CUDA)) {
        return true;
    }

    return false;
}

bool ggml_cuda_mul_mat_use_f16(const struct ggml_tensor * src0, const struct ggml_tensor * src1, struct ggml_tensor * /* dst */) {
    size_t src0_sz = ggml_nbytes(src0);
    size_t src1_sz = ggml_nbytes(src1);

    // mul_mat_q: src0 is converted to fp32 on device
    size_t mul_mat_q_transfer = src0_sz + src1_sz;

    // mul_mat_f16: src1 is converted to fp16 on cpu
    size_t mul_mat_f16_transfer = src0_sz + sizeof(half) * ggml_nelements(src1);

    // choose the smaller one to transfer to the device
    // TODO: this is not always the best choice due to the overhead of converting to fp16
    return mul_mat_f16_transfer < mul_mat_q_transfer;
}

void ggml_cuda_mul_mat(const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst, void * wdata, size_t wsize) {
    GGML_ASSERT(ggml_cuda_can_mul_mat(src0, src1, dst));

    if (src0->type == GGML_TYPE_F32) {
        ggml_cuda_mul_mat_f32(src0, src1, dst);
    }
    else if (src0->type == GGML_TYPE_F16) {
        if (ggml_cuda_mul_mat_use_f16(src0, src1, dst)) {
            ggml_cuda_mul_mat_f16(src0, src1, dst, wdata, wsize);
        }
        else {
            ggml_cuda_mul_mat_q_f32(src0, src1, dst);
        }
    }
    else if (ggml_is_quantized(src0->type)) {
        ggml_cuda_mul_mat_q_f32(src0, src1, dst);
    }
    else {
        GGML_ASSERT(false);
    }
}

size_t ggml_cuda_mul_mat_get_wsize(const struct ggml_tensor * src0, const struct ggml_tensor * src1, struct ggml_tensor * dst) {
    if (ggml_cuda_mul_mat_use_f16(src0, src1, dst)) {
        return ggml_nelements(src1) * sizeof(ggml_fp16_t);
    }
    else {
        return 0;
    }
}

void ggml_cuda_transform_tensor(ggml_tensor * tensor) {
    const int64_t ne0 = tensor->ne[0];
    const int64_t ne1 = tensor->ne[1];
    const int64_t ne2 = tensor->ne[2];
    const int64_t ne3 = tensor->ne[3];

    const ggml_type type = tensor->type;
    const size_t q_sz = ggml_type_size(type) * ne0 * ne1 * ne2 * ne3 / ggml_blck_size(type);

    size_t q_size;
    char * d_Q = (char *) ggml_cuda_pool_malloc(q_sz, &q_size);

    hipStream_t cudaStream2 = g_cudaStreams2[0];

    // copy tensor to device
    CUDA_CHECK(ggml_cuda_h2d_tensor_2d(d_Q, tensor, 0, 0, cudaStream2));
    CUDA_CHECK(hipDeviceSynchronize());

    tensor->data = d_Q;
    tensor->backend = GGML_BACKEND_CUDA;
}
